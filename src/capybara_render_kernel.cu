#include "hip/hip_runtime.h"
///// Otter: Mandelbrot render kernel using Capybara early Hi/Lo + classic continuation (fills d_iterations). Adds exact cardioid/2-bulb skip and Hi/Lo gating for shallow zooms.
///** Schneefuchs: API unverändert; ASCII-Logs; optionale CUDA-Event-Zeitmessung bei Settings::performanceLogging; keine neuen Dateien.
///** Maus: Zero information loss – Innenpunkte = maxIter; Hi/Lo nur bei feinem Pixelstep; Host/Device sauber getrennt.
///** Datei: src/capybara_render_kernel.cu

#include "pch.hpp"

#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdint.h>

#include "settings.hpp"
#include "luchs_log_host.hpp"
#include "capybara_api.cuh"

// Capybara building blocks
#include "capybara_math.cuh"
#include "capybara_mapping.cuh"
#include "capybara_ziter.cuh"
#include "capybara_integration.cuh"
#include "capybara_pixel_iter.cuh"

// ------------------------------ launch config ---------------------------------
namespace {
    // Balanced config: good occupancy on SM80+ with modest register pressure.
    constexpr int BX = 32;
    constexpr int BY = 8;
    static_assert(BX > 0 && BY > 0, "Block dimensions must be positive");

    // When pixel steps are larger than this, classic double escape-time is sufficient.
    // Avoids Hi/Lo overhead at shallow zooms without any loss of information.
    constexpr double kStepThresh = 1e-12;
}

// --------------------------------- helpers ------------------------------------
static __device__ __forceinline__ uint16_t clamp_u16_from_int(int v) {
    return (v < 0) ? 0u : (v > 65535 ? 65535u : static_cast<uint16_t>(v));
}

// Analytic interior tests (exact): main cardioid and period-2 bulb
static __device__ __forceinline__ bool in_main_cardioid(double2 c) {
    const double x = c.x - 0.25;
    const double y = c.y;
    const double q = x * x + y * y;
    // Inside if q * (q + x) <= 0.25 * y^2
    return q * (q + x) <= 0.25 * (y * y);
}
static __device__ __forceinline__ bool in_period2_bulb(double2 c) {
    const double xr = c.x + 1.0;
    const double yr = c.y;
    // Inside if (x+1)^2 + y^2 <= (1/4)^2
    return (xr * xr + yr * yr) <= (1.0 / 16.0);
}
static __device__ __forceinline__ bool in_cardioid_or_bulb(double2 c) {
    return in_main_cardioid(c) || in_period2_bulb(c);
}

// -------------------------------- render kernel --------------------------------
// Computes iteration counts only. Coloring/heatmap happens elsewhere.
__global__ __launch_bounds__(BX * BY, 2)
void mandelbrotKernel_capybara(
    uint16_t* __restrict__ d_it,
    int w, int h,
    double cx, double cy,
    double stepX, double stepY,
    int maxIter)
{
    const int px = blockIdx.x * blockDim.x + threadIdx.x;
    const int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= w || py >= h) return;

    const int idx = py * w + px;

    // Map pixel -> complex plane (double). Keep it branch-free and deterministic.
    const double x = cx + (static_cast<double>(px) - 0.5 * static_cast<double>(w)) * stepX;
    const double y = cy + (static_cast<double>(py) - 0.5 * static_cast<double>(h)) * stepY;
    const double2 cD = make_double2(x, y);

    // 1) Analytic interior: exact membership → it = maxIter (no iterations needed)
    if (in_cardioid_or_bulb(cD)) {
        d_it[idx] = clamp_u16_from_int(maxIter);
        return;
    }

    // 2) Hi/Lo gating: for coarse pixel steps use classic double escape-time (identical result)
    const double ax = fabs(stepX);
    const double ay = fabs(stepY);
    const double m  = (ax > ay ? ax : ay);
    if (m > kStepThresh) {
        double zx = 0.0, zy = 0.0;
        int it = 0;
        for (; it < maxIter; ++it) {
            const double xx = zx * zx - zy * zy + cD.x;
            const double yy = 2.0 * zx * zy + cD.y;
            zx = xx; zy = yy;
            // Escape when |z|^2 > 4
            if (xx * xx + yy * yy > 4.0) { ++it; break; }
        }
        d_it[idx] = clamp_u16_from_int(it);
        return;
    }

    // 3) Deep zoom path: Capybara early Hi/Lo + classic continuation
    const int iters = capy_compute_iters_from_zero(cx, cy, stepX, stepY, px, py, w, h, maxIter);
    d_it[idx] = clamp_u16_from_int(iters);
}

// ------------------------------- host wrapper ---------------------------------
// Non-throwing numeric RC logs (avoid C4297 under /WX)
#define CAPY_NT_CHECK(call) \
    do { hipError_t _e = (call); if (_e != hipSuccess) { \
        LUCHS_LOG_HOST("[CUDA][CAPY] rc=%d at %s:%d", (int)_e, __FILE__, __LINE__); } } while (0)

extern "C" void launch_mandelbrot_capybara(
    uint16_t* d_it,
    int w, int h,
    double cx, double cy,
    double stepX, double stepY,
    int maxIter,
    hipStream_t stream /*= nullptr*/)
{
    if (!d_it || w <= 0 || h <= 0 || maxIter < 0) {
        if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[CAPY] invalid-args w=%d h=%d maxIter=%d d_it=%p", w, h, maxIter, (void*)d_it);
        }
        return;
    }

    const dim3 block(BX, BY);
    const dim3 grid((w + BX - 1) / BX, (h + BY - 1) / BY);

    if constexpr (Settings::debugLogging || Settings::performanceLogging) {
        LUCHS_LOG_HOST("[CAPY] queued w=%d h=%d grid=%dx%d block=%dx%d maxIter=%d stream=%p",
                       w, h, grid.x, grid.y, block.x, block.y, maxIter, (void*)stream);
    }

    // Optional CUDA event timing (visible when Settings::performanceLogging == true)
    hipEvent_t evStart = nullptr, evStop = nullptr;
    if constexpr (Settings::performanceLogging) {
        (void)hipEventCreateWithFlags(&evStart, hipEventDefault);
        (void)hipEventCreateWithFlags(&evStop,  hipEventDefault);
        (void)hipEventRecord(evStart, stream);
    }

    mandelbrotKernel_capybara<<<grid, block, 0, stream>>>(d_it, w, h, cx, cy, stepX, stepY, maxIter);

    if constexpr (Settings::performanceLogging) {
        (void)hipEventRecord(evStop, stream);
        (void)hipEventSynchronize(evStop);
        float ms = 0.0f;
        (void)hipEventElapsedTime(&ms, evStart, evStop);
        LUCHS_LOG_HOST("[CAPY][time] mand=%.3f ms (w=%d h=%d it=%d)", (double)ms, w, h, maxIter);
        (void)hipEventDestroy(evStart);
        (void)hipEventDestroy(evStop);
    }

    CAPY_NT_CHECK(hipPeekAtLastError());
}
