#include "hip/hip_runtime.h"
// Datei: src/renderer_core.cu
// Zeilen: 66
// 🐭 Maus-Kommentar: Entry-Point fürs Rendering. Keine manuelle TileSize mehr – `setupBuffers()` berechnet aus Zoom & heuristischer Blockgröße implizit die Tile-Anzahl. Schneefuchs sagt: „Wenn das System weiß, was gut für dich ist, dann hör drauf.“

#include "pch.hpp"

#include "renderer_core.hpp"
#include "renderer_window.hpp"
#include "renderer_pipeline.hpp"
#include "renderer_state.hpp"
#include "renderer_loop.hpp"     // 🎯 renderFrame und renderFrame_impl
#include "common.hpp"

Renderer::Renderer(int width, int height)
    : state(width, height) {}

Renderer::~Renderer() {}

void Renderer::initGL() {
    state.window = RendererWindow::createWindow(state.width, state.height, this);
    RendererWindow::setResizeCallback(state.window, this);
    RendererWindow::setKeyCallback(state.window);
    RendererPipeline::init();  // ✅ keine Parameter mehr
}

bool Renderer::shouldClose() const {
    return RendererWindow::shouldClose(state.window);
}

void Renderer::renderFrame(bool autoZoomEnabled) {
    RendererLoop::renderFrame(state, autoZoomEnabled);  // ✅ öffentlich sichtbare Schleife
}

void Renderer::renderFrame_impl(bool autoZoomEnabled) {
    RendererLoop::renderFrame_impl(state, autoZoomEnabled);  // 🔁 interne Schleife bei Bedarf
}

void Renderer::setupBuffers() {
    int totalPixels = state.width * state.height;

    CUDA_CHECK(hipMalloc(&state.d_iterations, totalPixels * sizeof(int)));
    CUDA_CHECK(hipMalloc(&state.d_entropy, totalPixels * sizeof(float)));

    // Dynamische Tile-Größe heuristisch wie im Kernel
    int tileSize = 32;
    if (state.zoom > 30000.0f)
        tileSize = 4;
    else if (state.zoom > 3000.0f)
        tileSize = 8;
    else if (state.zoom > 1000.0f)
        tileSize = 16;
    tileSize = std::max(4, std::min(tileSize, 32));
    state.lastTileSize = tileSize;

    int tilesX = state.width / tileSize;
    int tilesY = state.height / tileSize;
    state.h_entropy.resize(tilesX * tilesY);
}

void Renderer::freeDeviceBuffers() {
    if (state.d_iterations) {
        CUDA_CHECK(hipFree(state.d_iterations));
        state.d_iterations = nullptr;
    }
    if (state.d_entropy) {
        CUDA_CHECK(hipFree(state.d_entropy));
        state.d_entropy = nullptr;
    }
    state.h_entropy.clear();
}

void Renderer::resize(int newW, int newH) {
    state.width = newW;
    state.height = newH;
    std::printf("[INFO] Resized to %d x %d\n", newW, newH);
}
