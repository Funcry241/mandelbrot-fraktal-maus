// Datei: src/memory_utils.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include "memory_utils.hpp"

// Device-Speicher für Complexity-Buffer
extern "C" float* allocComplexityBuffer(int totalTiles) {
    float* d_complexity = nullptr;
    hipError_t err = hipMalloc(&d_complexity, totalTiles * sizeof(float));
    if (err != hipSuccess) {
        std::fprintf(stderr, "hipMalloc für Complexity-Buffer fehlgeschlagen: %s\n", hipGetErrorString(err));
        std::exit(EXIT_FAILURE);
    }
    return d_complexity;
}
