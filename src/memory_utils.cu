// Datei: src/memory_utils.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>   // ✨ Fix: für std::exit()
#include "memory_utils.hpp"

namespace MemoryUtils { // <--- 🐾 Namespace öffnen!

// Device-Speicher für Complexity-Buffer
float* allocComplexityBuffer(int totalTiles) {
    float* d_complexity = nullptr;
    hipError_t err = hipMalloc(&d_complexity, totalTiles * sizeof(float));
    if (err != hipSuccess) {
        std::fprintf(stderr, "hipMalloc für Complexity-Buffer fehlgeschlagen: %s\n", hipGetErrorString(err));
        std::exit(EXIT_FAILURE);
    }
    return d_complexity;
}

} // namespace MemoryUtils  // <--- 🐾 Namespace schließen!
