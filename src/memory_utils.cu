// Datei: src/memory_utils.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>   // ✨ Fix: für std::exit()
#include "memory_utils.hpp"

namespace MemoryUtils {

// Device-Speicher für Complexity-Buffer allokieren
float* allocComplexityBuffer(int totalTiles) {
    float* d_complexity = nullptr;
    hipError_t err = hipMalloc(&d_complexity, totalTiles * sizeof(float));
    if (err != hipSuccess) {
        std::fprintf(stderr, "hipMalloc für Complexity-Buffer fehlgeschlagen: %s\n", hipGetErrorString(err));
        std::exit(EXIT_FAILURE);
    }
    return d_complexity;
}

// Device-Speicher freigeben (idempotent)
void freeComplexityBuffer(float*& d_buffer) {
    if (d_buffer) {
        hipFree(d_buffer);  // Fehler werden bewusst ignoriert
        d_buffer = nullptr;
    }
}

} // namespace MemoryUtils
