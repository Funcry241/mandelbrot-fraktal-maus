#include "hip/hip_runtime.h"
///// Otter: Direkte Mandelbrot-Iteration (ohne Referenz-Orbit), GT-Palette (Cyan→Amber), Smooth-Coloring.
///  Schneefuchs: API & Mapping unverändert (pixelToComplex), deterministisch.
///  Maus: Heatmap-Vertrag bleibt (innen = maxIter).
///  Bonus: Eye-Candy-Animation (sanft, monotone Farbabbildung bleibt erhalten).

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>      // float2, uchar4
#include <vector_functions.h>  // make_float2, make_float3, make_uchar4
#include <cmath>
#include <chrono>

#include "settings.hpp"
#include "luchs_log_host.hpp"
#include "common.hpp"
#include "nacktmull_math.cuh"  // pixelToComplex(...)

// ============================================================================
// Device Utilities
// ============================================================================
__device__ __forceinline__ float  clamp01(float x)                  { return fminf(1.0f, fmaxf(0.0f, x)); }
__device__ __forceinline__ float  mixf(float a, float b, float t)   { return a + t * (b - a); }
__device__ __forceinline__ float3 mix3(float3 a, float3 b, float t) { return make_float3(mixf(a.x,b.x,t), mixf(a.y,b.y,t), mixf(a.z,b.z,t)); }

// Cardioid / Period-2-Bulb (Early-Out)
__device__ __forceinline__ bool insideMainCardioidOrBulb(float x, float y){
    const float x1 = x - 0.25f;
    const float y2 = y * y;
    const float q  = x1*x1 + y2;
    if (q*(q + x1) <= 0.25f*y2) return true; // main cardioid
    const float xp = x + 1.0f;                // period-2 bulb
    if (xp*xp + y2 <= 0.0625f) return true;
    return false;
}

// sRGB <-> Linear
__device__ __forceinline__ float  srgb_to_linear(float c){
    return (c <= 0.04045f) ? (c/12.92f) : powf((c + 0.055f)/1.055f, 2.4f);
}
__device__ __forceinline__ float  linear_to_srgb(float c){
    return (c <= 0.0031308f) ? (12.92f*c) : (1.055f*powf(c, 1.0f/2.4f) - 0.055f);
}
__device__ __forceinline__ float3 srgb_to_linear3(float3 c){
    return make_float3(srgb_to_linear(c.x), srgb_to_linear(c.y), srgb_to_linear(c.z));
}
__device__ __forceinline__ float3 linear_to_srgb3(float3 c){
    return make_float3(linear_to_srgb(c.x), linear_to_srgb(c.y), linear_to_srgb(c.z));
}

// ============================================================================
// GT-Palette (Cyan→Amber), Interpolation im Linearraum + sanfte Animation
// ============================================================================
__device__ __forceinline__ uchar4 gtPalette_u8(float x, bool inSet, float t){
    // Tuning (Eye-Candy, aber konservativ/monoton):
    // - gamma etwas niedriger für mehr Mittenton-Zeichnung
    // - "lift" hebt dunkle Tiefen leicht an (mehr Struktur im Außenbereich)
    // - warmShift driftet sanft über Zeit
    // - ultrafeine dynamische Mikro-Isolinien (sehr geringe Amplitude)
    const float gamma        = 0.86f;
    const float lift         = 0.07f;                         // 0.05..0.10
    const float baseVibr     = 1.04f;                         // Grundvibrance
    const float addVibrMax   = 0.06f;                         // Zusatzvibrance (x-abhängig)
    const float warmDriftAmp = 0.06f;                         // zeitl. Warmdrift ±6%
    const float warmShift    = 1.00f + warmDriftAmp * __sinf(0.30f * t);
    const float stripes      = 0.012f;                        // ultrafein
    const float stripeFreq   = 6.2f;

    if (inSet) return make_uchar4(10, 12, 16, 255);

    // Eingangsshaping
    x = clamp01(powf(clamp01(x), gamma));
    x = clamp01((x + lift) / (1.0f + lift));                  // Low-End anheben

    // Anchors (dezent heller im dunklen Bereich)
    const float  p[8] = { 0.00f, 0.12f, 0.25f, 0.42f, 0.60f, 0.78f, 0.95f, 1.00f };
    const float3 c[8] = {
        make_float3(11/255.f, 14/255.f, 26/255.f), // #0B0E1A (vorher #08090F)
        make_float3(20/255.f, 54/255.f,102/255.f), // #143666 (vorher #112D5F)
        make_float3(22/255.f, 84/255.f,159/255.f), // #16549F
        make_float3(36/255.f,178/255.f,191/255.f), // #24B2BF
        make_float3(255/255.f,210/255.f, 87/255.f),// #FFD257
        make_float3(236/255.f,121/255.f, 44/255.f),// #EC792C
        make_float3(171/255.f, 34/255.f, 61/255.f),// #AB223D
        make_float3(250/255.f,250/255.f,250/255.f) // #FAFAFA
    };

    int j = 0;
    #pragma unroll
    for (int i=0; i<7; ++i) { if (x >= p[i]) j = i; }
    const float span = fmaxf(p[j+1] - p[j], 1e-6f);
    float tseg = clamp01((x - p[j]) / span);
    tseg = tseg*tseg*(3.0f - 2.0f*tseg); // smootherstep

    float3 aLin  = srgb_to_linear3(c[j]);
    float3 bLin  = srgb_to_linear3(c[j+1]);
    float3 rgbLn = mix3(aLin, bLin, tseg);

    // Sanfte „Breathing“-Bewegung: x -> x'
    // Monotonie bleibt erhalten, Amplitude klein.
    {
        const float breath = 0.08f * __sinf(0.80f * t);       // ±0.08
        const float xprime = clamp01(x + breath * x * (1.0f - x));
        // leicht auf Sättigung wirken (x-abhängig)
        const float vibr = baseVibr + addVibrMax * clamp01((xprime - 0.10f) * (1.0f / 0.40f));
        const float luma = 0.2126f*rgbLn.x + 0.7152f*rgbLn.y + 0.0722f*rgbLn.z;
        rgbLn = make_float3(
            luma + (rgbLn.x - luma) * vibr * warmShift,
            luma + (rgbLn.y - luma) * vibr * 1.00f,
            luma + (rgbLn.z - luma) * vibr * (2.0f - warmShift)
        );
    }

    // Ultraf eine dynamische Mikro-Isolinien (nur mittlere Tonwerte, minimal)
    {
        const float mid = 4.0f * x * (1.0f - x);              // 0..1, peak bei x=0.5
        const float amp = stripes * mid;                      // nur Mitte betonen
        if (amp > 0.0f){
            const float phase = 6.2831853f * (x * stripeFreq + 0.10f * t);
            const float s = 0.5f + 0.5f * __sinf(phase);
            const float boost = 1.0f + amp * (s*s*s*s);       // Highlights biasen
            rgbLn.x *= boost; rgbLn.y *= boost; rgbLn.z *= boost;
        }
    }

    const float3 srgb = linear_to_srgb3(make_float3(
        clamp01(rgbLn.x), clamp01(rgbLn.y), clamp01(rgbLn.z)
    ));

    return make_uchar4(
        (unsigned char)(255.0f*clamp01(srgb.x) + 0.5f),
        (unsigned char)(255.0f*clamp01(srgb.y) + 0.5f),
        (unsigned char)(255.0f*clamp01(srgb.z) + 0.5f),
        255
    );
}

// Smooth-Iterations → Farbwert (+ Edge-Glow & Animation)
__device__ __forceinline__ uchar4 gtColor_fromSmoothState(int it, int maxIterations, float zx, float zy, float t){
    const bool inSet = (it >= maxIterations);
    if (inSet) return gtPalette_u8(0.0f, true, t);

    const float mag2 = zx*zx + zy*zy;
    if (mag2 > 1.0000001f && it > 0){
        const float mag = sqrtf(mag2);
        const float l2  = __log2f(__log2f(mag));
        float x = ((float)it - l2) / (float)maxIterations;

        // Edge-Glow: nahe Escape-Grenze leicht boosten, monoton halten
        float edge = clamp01(1.0f - 0.75f * l2);             // 0..1
        x = clamp01(x + 0.15f * edge * (1.0f - x));
        return gtPalette_u8(x, false, t);
    } else {
        float x = clamp01((float)it / (float)maxIterations);
        return gtPalette_u8(x, false, t);
    }
}

// ============================================================================
// Direkter Mandelbrot-Kernel (ohne Referenz-Orbit / Perturbation)
// ============================================================================
__global__ __launch_bounds__(256)
void mandelbrotKernel(
    uchar4* __restrict__ out, int* __restrict__ iterOut,
    int w, int h, float zoom, float2 center, int maxIter, float tSec)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;

    const int idx = y * w + x;

    // Mapping via Projektfunktion (konsistent, keine Verzerrung)
    const float2 c = pixelToComplex(
        (double)x + 0.5, (double)y + 0.5,
        w, h,
        (double)center.x, (double)center.y,
        (double)zoom
    );

    // Early interior exit
    if (insideMainCardioidOrBulb(c.x, c.y)){
        out[idx]     = make_uchar4(10,12,16,255);
        iterOut[idx] = maxIter;   // innen = maxIter (Heatmap-Vertrag)
        return;
    }

    // Direkte Iteration z_{n+1} = z_n^2 + c
    float zx = 0.0f, zy = 0.0f;
    int   it = maxIter;           // default: gilt als "innen"
    const float esc2 = 4.0f;

    #pragma unroll 1
    for (int i=0; i<maxIter; ++i){
        const float x2 = zx*zx, y2 = zy*zy;

        // Escape testen vor dem Update
        if (x2 + y2 > esc2){
            it = i;               // Iteration der Flucht
            break;
        }

        const float xt = x2 - y2 + c.x;
        zy = __fmaf_rn(2.0f*zx, zy, c.y);
        zx = xt;
    }

    out[idx]     = gtColor_fromSmoothState(it, maxIter, zx, zy, tSec);
    iterOut[idx] = it;
}

// ============================================================================
// Öffentliche API (wie im funktionierenden Build, Call-Sites unverändert)
// ============================================================================
extern "C" void launch_mandelbrotHybrid(
    uchar4* out, int* d_it,
    int w, int h, float zoom, float2 offset,
    int maxIter, int /*tile*/)
{
    using clk = std::chrono::high_resolution_clock;
    static clk::time_point t0;
    static bool t0_init = false;
    if (!t0_init){ t0 = clk::now(); t0_init = true; }
    const float tSec = (float)std::chrono::duration<double>(clk::now() - t0).count();

    if (!out || !d_it || w <= 0 || h <= 0 || maxIter <= 0){
        LUCHS_LOG_HOST("[NACKTMULL][ERR] invalid args out=%p it=%p w=%d h=%d itMax=%d",
                       (void*)out, (void*)d_it, w, h, maxIter);
        return;
    }

    const dim3 block(32, 8);
    const dim3 grid((w + block.x - 1) / block.x,
                    (h + block.y - 1) / block.y);

    mandelbrotKernel<<<grid, block>>>(out, d_it, w, h, zoom, offset, maxIter, tSec);

    if constexpr (Settings::performanceLogging){
        hipDeviceSynchronize();
        const double ms = 1e-3 * (double)std::chrono::duration_cast<std::chrono::microseconds>(clk::now() - t0).count();
        LUCHS_LOG_HOST("[PERF] nacktmull direct+anim kern=%.2f ms itMax=%d", ms, maxIter);
    }
}
