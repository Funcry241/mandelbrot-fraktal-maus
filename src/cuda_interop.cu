#include "hip/hip_runtime.h"
///// Otter: Kürzer & robust: Auto-PBO-Register, MapGuard, Events+Stream einmalig, ASCII-Perf-Logs.
/// /// Schneefuchs: Ein Pfad, deterministische State-Übergaben, konsistente Fehlerpfade, keine Redundanz.
/// /// Maus: Keine versteckten Pfade; Host-Pinning für schnelle memcpy; Shader/GL bleiben unberührt.
/// /// Datei: src/cuda_interop.cu

#include "pch.hpp"
#include "luchs_log_host.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "renderer_state.hpp"
#include "hermelin_buffer.hpp"
#include "bear_CudaPBOResource.hpp"

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#include <unordered_map>
#include <vector>
#include <stdexcept>
#include <cstdint>
#include <cstring>
#if !defined(__CUDA_ARCH__)
  #include <chrono>
#endif

// ---- Kernel (extern C) ------------------------------------------------------
extern "C" void launch_mandelbrotHybrid(
    uchar4* out, uint16_t* d_it,
    int w, int h, float zoom, float2 offset,
    int maxIter, int tile
);

namespace CudaInterop {

// ---- TU-lokaler Zustand -----------------------------------------------------
static bear_CudaPBOResource*                     s_pboActive = nullptr;
static std::unordered_map<GLuint, bear_CudaPBOResource*> s_pboMap;

static bool           s_pauseZoom = false;
static bool           s_deviceOk  = false;

static void*          s_hostRegEntropyPtr   = nullptr;  static size_t s_hostRegEntropyBytes   = 0;
static void*          s_hostRegContrastPtr  = nullptr;  static size_t s_hostRegContrastBytes  = 0;

static hipEvent_t    s_evStart = nullptr, s_evStop = nullptr; static bool s_evInit = false;
static hipStream_t   s_copyStrm = nullptr;

// ---- Helpers ----------------------------------------------------------------
static inline void ensureDeviceOnce() {
    if (!s_deviceOk) { CUDA_CHECK(hipSetDevice(0)); s_deviceOk = true; }
}
static inline void ensureEventsOnce() {
    if (s_evInit) return;
    if constexpr (Settings::debugLogging || Settings::performanceLogging) {
        hipEventCreate(&s_evStart);
        hipEventCreate(&s_evStop);
        s_evInit = (s_evStart && s_evStop);
        if constexpr (Settings::debugLogging)
            LUCHS_LOG_HOST("[CUDA] timing events %s", s_evInit ? "created" : "FAILED");
    }
}
static inline void ensureCopyStreamOnce() {
    if (!s_copyStrm) CUDA_CHECK(hipStreamCreateWithFlags(&s_copyStrm, hipStreamNonBlocking));
}
static inline void destroyEventsIfAny() {
    if (!s_evInit) return;
    hipEventDestroy(s_evStart); s_evStart=nullptr;
    hipEventDestroy(s_evStop);  s_evStop =nullptr;
    s_evInit=false;
}
static inline void ensureHostPinned(std::vector<float>& vec, void*& regPtr, size_t& regBytes) {
    const size_t cap = vec.capacity();
    void* ptr = cap ? (void*)vec.data() : nullptr;
    const size_t bytes = cap * sizeof(float);
    if (ptr == regPtr && bytes == regBytes) return;
    if (regPtr) CUDA_CHECK(hipHostUnregister(regPtr));
    if (ptr)    CUDA_CHECK(hipHostRegister(ptr, bytes, hipHostRegisterPortable));
    regPtr = ptr; regBytes = bytes;
    if constexpr (Settings::debugLogging)
        LUCHS_LOG_HOST("[PIN] host-register ptr=%p bytes=%zu", regPtr, regBytes);
}
struct MapGuard {
    bear_CudaPBOResource* r=nullptr;
    void* ptr=nullptr; size_t bytes=0;
    explicit MapGuard(bear_CudaPBOResource* rr):r(rr){ if(r){ ptr=r->mapAndLog(bytes);} }
    ~MapGuard(){ if(r) r->unmap(); }
    MapGuard(const MapGuard&) = delete; MapGuard& operator=(const MapGuard&) = delete;
};

// ---- PBO-Verwaltung ---------------------------------------------------------
void registerAllPBOs(const GLuint* ids, int count) {
    ensureDeviceOnce();
    for (auto &kv : s_pboMap) delete kv.second; s_pboMap.clear(); s_pboActive=nullptr;
    if (!ids || count<=0) return;
    for (int i=0;i<count;++i) {
        if (!ids[i]) continue;
        auto* res = new bear_CudaPBOResource(ids[i]);
        if (res && res->get()) s_pboMap[ids[i]] = res; else delete res;
    }
    for (int i=0;i<count && !s_pboActive;++i){ auto it=s_pboMap.find(ids[i]); if(it!=s_pboMap.end()) s_pboActive=it->second; }
}
void unregisterAllPBOs() {
    for (auto &kv : s_pboMap) delete kv.second; s_pboMap.clear(); s_pboActive=nullptr;
}
void registerPBO(const Hermelin::GLBuffer& pbo) {
    ensureDeviceOnce();
    const GLuint id = pbo.id();
    auto it = s_pboMap.find(id);
    if (it == s_pboMap.end()) {
        auto* res = new bear_CudaPBOResource(id);
        if (res && res->get()) { s_pboMap[id]=res; if constexpr (Settings::debugLogging) LUCHS_LOG_HOST("[CUDA-Interop] auto-registered PBO id=%u", id); }
        else { delete res; LUCHS_LOG_HOST("[FATAL] failed to create CudaPBOResource id=%u", id); return; }
        it = s_pboMap.find(id);
    }
    s_pboActive = it->second;
}
void unregisterPBO() {  // nur aktives PBO lösen + Host-Pins/Event/Stream putzen
    if (s_hostRegEntropyPtr)  { hipHostUnregister(s_hostRegEntropyPtr);  s_hostRegEntropyPtr=nullptr;  s_hostRegEntropyBytes=0; }
    if (s_hostRegContrastPtr) { hipHostUnregister(s_hostRegContrastPtr); s_hostRegContrastPtr=nullptr; s_hostRegContrastBytes=0; }
    destroyEventsIfAny();
    if (s_copyStrm) { hipStreamDestroy(s_copyStrm); s_copyStrm=nullptr; }
    delete s_pboActive; s_pboActive=nullptr;
}

// ---- Hauptpfad --------------------------------------------------------------
void renderCudaFrame(
    Hermelin::CudaDeviceBuffer& d_iterations,
    Hermelin::CudaDeviceBuffer& d_entropy,
    Hermelin::CudaDeviceBuffer& d_contrast,
    int width, int height,
    float zoom, float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    std::vector<float>& h_contrast,
    float2& newOffset, bool& shouldZoom,
    int tileSize, RendererState& state
){
#if !defined(__CUDA_ARCH__)
    const auto t0 = std::chrono::high_resolution_clock::now();
    double mapMs=0.0, mbMs=0.0, entMs=0.0, conMs=0.0;
#endif
    if (!s_pboActive) throw std::runtime_error("[FATAL] CUDA PBO not registered!");
    if (width<=0 || height<=0)  throw std::runtime_error("invalid framebuffer dims");
    if (tileSize<=0) { int was=tileSize; tileSize = Settings::BASE_TILE_SIZE>0 ? Settings::BASE_TILE_SIZE : 16; LUCHS_LOG_HOST("[WARN] tileSize<=0 (%d) -> using %d", was, tileSize); }

    const size_t totalPx = size_t(width)*size_t(height);
    const int tilesX = (width  + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    const size_t itBytes = totalPx * sizeof(uint16_t);
    const size_t enBytes = size_t(numTiles) * sizeof(float);
    const size_t ctBytes = size_t(numTiles) * sizeof(float);

    if (d_iterations.size()<itBytes || d_entropy.size()<enBytes || d_contrast.size()<ctBytes)
        throw std::runtime_error("CudaInterop::renderCudaFrame: device buffers undersized");

#if !defined(__CUDA_ARCH__)
    const auto tMap0 = std::chrono::high_resolution_clock::now();
#endif
    MapGuard map(s_pboActive);
    if (!map.ptr) throw std::runtime_error("pboResource->map() returned null");

#if !defined(__CUDA_ARCH__)
    const auto tMap1 = std::chrono::high_resolution_clock::now();
    mapMs = std::chrono::duration<double, std::milli>(tMap1 - tMap0).count();
#endif
    const size_t needBytes = size_t(width)*size_t(height)*sizeof(uchar4);
    if (map.bytes < needBytes) throw std::runtime_error("PBO byte size mismatch");

    if constexpr (Settings::debugLogging || Settings::performanceLogging) {
        ensureEventsOnce();
        if (s_evInit) hipEventRecord(s_evStart, 0);
    }

    (void)hipGetLastError();
    launch_mandelbrotHybrid(static_cast<uchar4*>(map.ptr),
                            static_cast<uint16_t*>(d_iterations.get()),
                            width,height, zoom, offset, maxIterations, tileSize);
    hipError_t mbErrLaunch = hipGetLastError();
    hipError_t mbErrSync   = hipSuccess;

    if constexpr (Settings::debugLogging || Settings::performanceLogging) {
        if (s_evInit) {
            hipEventRecord(s_evStop, 0);
            mbErrSync = hipEventSynchronize(s_evStop);
            float ms=0.0f;
            if (mbErrSync==hipSuccess) {
                hipEventElapsedTime(&ms, s_evStart, s_evStop);
            #if !defined(__CUDA_ARCH__)
                mbMs = ms;
            #endif
            }
        }
    }
    if (mbErrLaunch != hipSuccess || mbErrSync != hipSuccess)
        throw std::runtime_error("CUDA failure: mandelbrot kernel");

#if !defined(__CUDA_ARCH__)
    const auto tEC0 = std::chrono::high_resolution_clock::now();
#endif
    ::computeCudaEntropyContrast(
        static_cast<const uint16_t*>(d_iterations.get()),
        static_cast<float*>(d_entropy.get()),
        static_cast<float*>(d_contrast.get()),
        width, height, tileSize, maxIterations
    );
#if !defined(__CUDA_ARCH__)
    const auto tEC1 = std::chrono::high_resolution_clock::now();
    const double ecMs = std::chrono::duration<double, std::milli>(tEC1 - tEC0).count();
    entMs = ecMs * 0.5; conMs = ecMs * 0.5;
#endif

    // Host-Transfers (gepinnt, async)
    if (h_entropy.capacity()  < size_t(numTiles)) h_entropy.reserve(size_t(numTiles));
    if (h_contrast.capacity() < size_t(numTiles)) h_contrast.reserve(size_t(numTiles));
    ensureHostPinned(h_entropy,  s_hostRegEntropyPtr,  s_hostRegEntropyBytes);
    ensureHostPinned(h_contrast, s_hostRegContrastPtr, s_hostRegContrastBytes);
    h_entropy.resize(size_t(numTiles)); h_contrast.resize(size_t(numTiles));

    ensureCopyStreamOnce();
    CUDA_CHECK(hipMemcpyAsync(h_entropy.data(),  d_entropy.get(),  enBytes, hipMemcpyDeviceToHost, s_copyStrm));
    CUDA_CHECK(hipMemcpyAsync(h_contrast.data(), d_contrast.get(), ctBytes, hipMemcpyDeviceToHost, s_copyStrm));
    CUDA_CHECK(hipStreamSynchronize(s_copyStrm));

    // Zoom-Entscheidung (hier neutral)
    shouldZoom = false; newOffset = offset;

#if !defined(__CUDA_ARCH__)
    const auto t1 = std::chrono::high_resolution_clock::now();
    const double totalMs = std::chrono::duration<double, std::milli>(t1 - t0).count();
    state.lastTimings.valid            = true;
    state.lastTimings.pboMap           = mapMs;
    state.lastTimings.mandelbrotTotal  = mbMs;
    state.lastTimings.mandelbrotLaunch = 0.0;
    state.lastTimings.mandelbrotSync   = 0.0;
    state.lastTimings.entropy          = entMs;
    state.lastTimings.contrast         = conMs;
    state.lastTimings.deviceLogFlush   = 0.0;

    if constexpr (Settings::performanceLogging)
        LUCHS_LOG_HOST("[PERF] path=gpu mp=%.2f mb=%.2f en=%.2f ct=%.2f tt=%.2f", mapMs, mbMs, entMs, conMs, totalMs);
#endif
}

// ---- Sonstiges API ----------------------------------------------------------
void setPauseZoom(bool pause){ s_pauseZoom = pause; }
bool getPauseZoom(){ return s_pauseZoom; }

bool precheckCudaRuntime() {
    int deviceCount = 0;
    hipError_t e1 = hipFree(0);
    hipError_t e2 = hipGetDeviceCount(&deviceCount);
    if constexpr (Settings::debugLogging)
        LUCHS_LOG_HOST("[CUDA] precheck err1=%d err2=%d count=%d", (int)e1, (int)e2, deviceCount);
    return e1==hipSuccess && e2==hipSuccess && deviceCount>0;
}

bool verifyCudaGetErrorStringSafe() {
    const char* msg = hipGetErrorString(hipErrorInvalidValue);
    if (msg) { if constexpr (Settings::debugLogging) LUCHS_LOG_HOST("[CHECK] hipGetErrorString(dummy)=\"%s\"", msg); return true; }
    LUCHS_LOG_HOST("[FATAL] hipGetErrorString returned null"); return false;
}

static inline int getAttrSafe(hipDeviceAttribute_t a, int dev){ int v=0; (void)hipDeviceGetAttribute(&v,a,dev); return v; }
void logCudaDeviceContext(const char* tag) {
    if constexpr (!(Settings::debugLogging || Settings::performanceLogging)) { (void)tag; return; }
    int dev=-1; hipError_t e0=hipGetDevice(&dev);
    int rt=0, drv=0; hipRuntimeGetVersion(&rt); hipDriverGetVersion(&drv);
    char name[256]={0};
    if (dev>=0){ hipDeviceProp_t p{}; if (hipGetDeviceProperties(&p,dev)==hipSuccess) std::strncpy(name,p.name,sizeof(name)-1); }
    if (e0==hipSuccess && dev>=0) {
        const int ccM=getAttrSafe(hipDeviceAttributeComputeCapabilityMajor,dev);
        const int ccN=getAttrSafe(hipDeviceAttributeComputeCapabilityMinor,dev);
        const int sms=getAttrSafe(hipDeviceAttributeMultiprocessorCount,dev);
        const int warp=getAttrSafe(hipDeviceAttributeWarpSize,dev);
        size_t mf=0, mt=0; hipMemGetInfo(&mf,&mt);
        LUCHS_LOG_HOST("[CUDA] ctx tag=%s rt=%d drv=%d dev=%d name=\"%s\" cc=%d.%d sms=%d warp=%d memMB free=%zu total=%zu",
            (tag?tag:"(null)"), rt, drv, dev, name, ccM, ccN, sms, warp, (mf>>20), (mt>>20));
    } else {
        LUCHS_LOG_HOST("[CUDA] ctx tag=%s deviceQuery failed e0=%d dev=%d", (tag?tag:"(null)"), (int)e0, dev);
    }
}

} // namespace CudaInterop
