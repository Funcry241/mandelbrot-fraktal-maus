#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu

#pragma once

#ifdef _WIN32
    #define NOMINMAX
    #include <windows.h>    // Erst Windows-Header
#endif

#include <GL/gl.h>           // Dann OpenGL (GL.h)
#include <hip/hip_runtime.h>    // Dann CUDA
#include <cuda_gl_interop.h> // CUDA-OpenGL Interop

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"


namespace CudaInterop {

#define CHECK_CUDA_STEP(call, msg) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        std::fprintf(stderr, "[CUDA ERROR] %s: %s\n", msg, hipGetErrorString(err)); \
        std::exit(EXIT_FAILURE); \
    } \
}

#define DEBUG_PRINT(fmt, ...) \
    do { if (Settings::debugLogging) { std::fprintf(stdout, "[DEBUG] " fmt "\n", ##__VA_ARGS__); } } while(0)

void renderCudaFrame(
    hipGraphicsResource_t cudaPboRes,
    int                   width,
    int                   height,
    float&                zoom,
    float2&               offset,
    int                   maxIter,
    float*                d_complexity,
    std::vector<float>&   h_complexity
) {
    DEBUG_PRINT("Starting frame render");

    uchar4* d_img = nullptr;
    size_t  imgSize = 0;

    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "hipGraphicsMapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "hipGraphicsResourceGetMappedPointer");

    if (Settings::debugGradient) {
        DEBUG_PRINT("Launching debug gradient kernel");
        launch_debugGradient(d_img, width, height);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "DebugGradient synchronize");
    } else {
        DEBUG_PRINT("Launching Mandelbrot kernel");
        launch_mandelbrotHybrid(d_img, width, height, zoom, offset, maxIter);
        CHECK_CUDA_STEP(hipGetLastError(), "launch_mandelbrotHybrid");

        int totalTiles = static_cast<int>(h_complexity.size());

        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "hipMemset d_complexity");

        dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
        dim3 gridDim((width + Settings::TILE_W - 1) / Settings::TILE_W,
                     (height + Settings::TILE_H - 1) / Settings::TILE_H);

        DEBUG_PRINT("Launching complexity kernel with Grid (%d, %d) Block (%d, %d)", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

        computeComplexity<<<gridDim, blockDim>>>(d_img, width, height, d_complexity);
        CHECK_CUDA_STEP(hipGetLastError(), "computeComplexity kernel launch");
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "computeComplexity synchronize");

        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy d_complexity -> h_complexity");

        DEBUG_PRINT("Searching for tile with highest variance...");

        int tilesX = (width + Settings::TILE_W - 1) / Settings::TILE_W;
        float bestVariance = -1.0f;
        int   bestIdx = 0;

        for (int i = 0; i < totalTiles; ++i) {
            if (h_complexity[i] > bestVariance) {
                bestVariance = h_complexity[i];
                bestIdx = i;
            }
        }

        DEBUG_PRINT("Best variance found: %.6f", bestVariance);

        bool offsetChanged = false;
        bool zoomChanged = false;

        if (bestVariance > 0.0f) {
            int bx = bestIdx % tilesX;
            int by = bestIdx / tilesX;
            float newOffX = offset.x + ((bx + 0.5f) * Settings::TILE_W - width * 0.5f) / zoom;
            float newOffY = offset.y + ((by + 0.5f) * Settings::TILE_H - height * 0.5f) / zoom;

            offsetChanged = (std::fabs(newOffX - offset.x) > 1e-6f) || (std::fabs(newOffY - offset.y) > 1e-6f);
            if (std::isfinite(newOffX) && std::isfinite(newOffY)) {
                offset.x = newOffX;
                offset.y = newOffY;
            }

            DEBUG_PRINT("New offset: (%.6f, %.6f)", offset.x, offset.y);
        }

        float newZoom = zoom * Settings::zoomFactor;
        constexpr float maxZoomAllowed = 1e15f;

        zoomChanged = (std::fabs(newZoom - zoom) > 1e-6f);
        if (std::isfinite(newZoom) && newZoom < maxZoomAllowed) {
            zoom = newZoom;
            DEBUG_PRINT("New zoom: %.6f", zoom);
        }

        if (offsetChanged || zoomChanged) {
            DEBUG_PRINT("Offset or zoom changed — resetting iterations");
            resetIterations();
        }
    }

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "hipGraphicsUnmapResources");

    DEBUG_PRINT("Frame render complete");
}

void checkDynamicParallelismSupport() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    if (prop.major < 3 || (prop.major == 3 && prop.minor < 5)) {
        std::fprintf(stderr, "Dynamic Parallelism not supported. Compute Capability 3.5+ required.\n");
        std::exit(EXIT_FAILURE);
    }
}

} // namespace CudaInterop
