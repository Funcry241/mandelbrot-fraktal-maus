#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// Zeilen: 208
// 🐅 Maus-Kommentar: CUDA/OpenGL-Interop – jetzt mit doppelter Genauigkeit bei Zoom & Offset für stabile Navigation. Float bleibt im Kernel. Schneefuchs: „Nur wer präzise zielt, braucht nicht zu rudern.“

#include "pch.hpp"  // 💡 Muss als erstes stehen!
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"  // 🧠 Zugriff auf smoothedTargetOffset

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource = nullptr;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    if (cudaPboResource != nullptr) {
        std::cerr << "[ERROR] registerPBO called but resource is already registered!\n";
        return;
    }
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard));
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void renderCudaFrame(
    int* d_iterations,
    float* d_entropy,
    int width,
    int height,
    double zoom,
    double2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    float2& newOffset,
    bool& shouldZoom,
    int tileSize
) {
    if (!cudaPboResource) {
        std::cerr << "[FATAL] CUDA PBO not registered before renderCudaFrame.\n";
        std::exit(EXIT_FAILURE);
    }

    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));
    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));

    // 🔄 Übergabe intern als float (GPU bleibt in Single Precision)
    float zoom_f = static_cast<float>(zoom);
    float2 offset_f = make_float2(static_cast<float>(offset.x), static_cast<float>(offset.y));

    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom_f, offset_f, maxIterations);
    computeTileEntropy(d_iterations, d_entropy, width, height, tileSize, maxIterations);

    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    h_entropy.resize(numTiles);

    if (devPtr == nullptr) {
        std::cerr << "[FATAL] devPtr is null after hipGraphicsResourceGetMappedPointer!\n";
        std::exit(EXIT_FAILURE);
    }
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    shouldZoom = false;

    if (!pauseZoom) {
        const float dynamicThreshold = std::max(Settings::VARIANCE_THRESHOLD / std::log2(zoom_f + 2.0f), Settings::MIN_VARIANCE_THRESHOLD);

        float2 bestOffset = offset_f;
        float bestEntropy = 0.0f;
        float bestScore = -1.0f;
        int bestIndex = -1;

        for (int i = 0; i < numTiles; ++i) {
            int bx = i % tilesX;
            int by = i / tilesX;

            float centerX = (bx + 0.5f) * tileSize;
            float centerY = (by + 0.5f) * tileSize;

            float2 tileCenter = {
                (centerX - width  / 2.0f) / zoom_f + offset_f.x,
                (centerY - height / 2.0f) / zoom_f + offset_f.y
            };

            float2 delta = { tileCenter.x - offset_f.x, tileCenter.y - offset_f.y };
            float dist = std::sqrt(delta.x * delta.x + delta.y * delta.y);
            float score = h_entropy[i] / (1.0f + Settings::ENTROPY_NEARBY_BIAS * dist);

            if (h_entropy[i] > dynamicThreshold && score > bestScore) {
                bestScore = score;
                bestOffset = tileCenter;
                bestEntropy = h_entropy[i];
                bestIndex = i;
            }
        }

        static int lastIndex = -1;
        if (Settings::debugLogging && bestIndex != lastIndex) {
            std::printf("[DEBUG] Zoom = %.6f | Threshold = %.8f\n", zoom_f, dynamicThreshold);
            if (bestIndex >= 0) {
                std::printf("[DEBUG] Best Tile = %d | Score = %.6f | Entropy = %.6f\n", bestIndex, bestScore, bestEntropy);
            } else {
                std::puts("[DEBUG] No tile passed threshold. Zoom paused.");
            }
            lastIndex = bestIndex;
        }

        if (bestIndex >= 0) {
            extern RendererState* globalRendererState;
            auto& state = *globalRendererState;

            constexpr float SMOOTHING_ALPHA   = 0.15f;
            constexpr float SCORE_THRESHOLD   = 0.95f;
            constexpr float NEWTARGET_DIST    = 0.001f;

            float2 delta = {
                bestOffset.x - state.smoothedTargetOffset.x,
                bestOffset.y - state.smoothedTargetOffset.y
            };
            float dist = std::sqrt(delta.x * delta.x + delta.y * delta.y);
            bool isNewTarget = bestScore > state.smoothedTargetScore * SCORE_THRESHOLD || dist > NEWTARGET_DIST;

            if (isNewTarget) {
                state.smoothedTargetOffset = bestOffset;
                state.smoothedTargetScore = bestScore;                
            }

            newOffset = state.smoothedTargetOffset;
            shouldZoom = true;
        }
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
}

void setPauseZoom(bool pause) {
    pauseZoom = pause;
}

bool getPauseZoom() {
    return pauseZoom;
}

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS && (key == GLFW_KEY_P || key == GLFW_KEY_SPACE)) {
        pauseZoom = !pauseZoom;
        std::cout << "[Zoom] Auto-Zoom " << (pauseZoom ? "paused" : "resumed") << "\n";
    }
}

} // namespace CudaInterop
