#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// Zeilen: 278
/* 🐭 Maus-Kommentar: CUDA-Interop mit kompaktem ASCII-Logging für Zoomanalyse.
   Jetzt mit dO (OffsetDist), dPx (Bildschirmpixel), Score, Entropie, Kontrast und Zielstatus – alles CSV-freundlich.
   Schneefuchs sieht klar: Kein Wildsprung bleibt unbemerkt.
   → Signatur fixiert: renderCudaFrame(...) mit double / double2 für präzise Tiefe. Linker-Link ist happy.
*/

#include "pch.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
#include "zoom_logic.hpp"
#include "heatmap_overlay.hpp"
#include <vector>
#include <cstdio>

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource = nullptr;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    if (cudaPboResource != nullptr) {
        std::cerr << "[ERROR] registerPBO called but resource is already registered!\n";
        return;
    }
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard));
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void renderCudaFrame(
    int* d_iterations,
    float* d_entropy,
    int width,
    int height,
    double zoom,
    double2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    double2& newOffset,
    bool& shouldZoom,
    int tileSize,
    int supersampling,
    RendererState& state
) {
    if (!cudaPboResource) {
        throw std::runtime_error("[FATAL] CUDA PBO not registered before renderCudaFrame.");
    }

    if (Settings::debugLogging) {
        std::printf("[Zoom] Auto-Zoom is %s\n", pauseZoom ? "PAUSED" : "ACTIVE");
    }

    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));
    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));
    if (Settings::debugLogging) {
        std::printf("[DEBUG] PBO mapped: %p (size = %zu)\n", (void*)devPtr, size);
    }

    float zoom_f = static_cast<float>(zoom);
    float2 offset_f = make_float2(static_cast<float>(offset.x), static_cast<float>(offset.y));

    if (Settings::debugLogging) {
        std::printf("[DEBUG] Launch MandelbrotKernel zoom %.2f maxIter %d supersampling %d\n", zoom, maxIterations, supersampling);
    }

    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom_f, offset_f, maxIterations, supersampling);

    hipDeviceSynchronize();  // sicherstellen, dass Fehler jetzt sichtbar sind
    hipError_t kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        std::fprintf(stderr, "[CUDA ERROR] MandelbrotKernel launch failed: %s\n", hipGetErrorString(kernelErr));
    }

    computeTileEntropy(d_iterations, d_entropy, width, height, tileSize, maxIterations);

    if (Settings::debugLogging) {
        std::puts("[DEBUG] PBO unmapped");
    }

    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    h_entropy.resize(numTiles);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    shouldZoom = false;

    if (!pauseZoom) {
        ZoomLogic::ZoomResult result = ZoomLogic::evaluateZoomTarget(
            h_entropy,
            offset,
            zoom_f,
            width,
            height,
            tileSize,
            make_float2(static_cast<float>(state.offset.x), static_cast<float>(state.offset.y)),
            state.zoomResult.bestIndex,
            state.zoomResult.bestEntropy,
            state.zoomResult.bestContrast
        );

        if (result.bestIndex >= 0) {
            newOffset = result.newOffset;
            shouldZoom = result.shouldZoom;

            if (result.isNewTarget) {
                state.zoomResult.bestEntropy  = result.bestEntropy;
                state.zoomResult.bestContrast = result.bestContrast;
                state.zoomResult.bestIndex    = result.bestIndex;
            }
        }

        if (Settings::debugLogging) {
            if (result.bestIndex >= 0) {
                float minJump = Settings::MIN_JUMP_DISTANCE / zoom_f;
                std::printf(
                    "Zoom Z %.1e I %d E %.3f C %.3f S %.3f dO %.2e dPx %.1f minJ %.2e dE %.3f dC %.3f RelE %.2f RelC %.2f New %d\n",
                    zoom_f,
                    result.bestIndex,
                    result.bestEntropy,
                    result.bestContrast,
                    result.bestScore,
                    result.distance,
                    result.distance * zoom_f * width,
                    minJump,
                    result.relEntropyGain,
                    result.relContrastGain,
                    result.relEntropyGain,
                    result.relContrastGain,
                    result.isNewTarget ? 1 : 0
                );
            } else {
                float avgEntropy = 0.0f;
                int countAbove = 0;
                for (float h : h_entropy) {
                    avgEntropy += h;
                    if (h > Settings::VARIANCE_THRESHOLD) countAbove++;
                }
                avgEntropy /= h_entropy.size();
                std::printf("Zoom NoZoom TilesAbove %d AvgEntropy %.5f\n", countAbove, avgEntropy);
            }
        }

        if (!result.isNewTarget) {
            state.zoomResult = result;
        }
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
}

void setPauseZoom(bool pause) {
    pauseZoom = pause;
}

bool getPauseZoom() {
    return pauseZoom;
}

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS) {
        if (key == GLFW_KEY_P || key == GLFW_KEY_SPACE) {
            pauseZoom = !pauseZoom;
            std::cout << "[Zoom] Auto-Zoom " << (pauseZoom ? "paused" : "resumed") << "\n";
        }

        if (key == GLFW_KEY_H) {
            HeatmapOverlay::toggle();
            if (Settings::debugLogging) {
                std::puts("[DEBUG] Heatmap overlay toggled (H)");
            }
        }
    }
}

// 🐭 Maus-Kommentar: CSV-Log-Ausgabe aller Tile-Iterationen eines Frames – dient Analyse der Tiefenverteilung.
// Schneefuchs: Nur ASCII, keine GPU-Belastung, immer verständlich.
void logZoomEvaluation(const int* d_iterations, int width, int height, int tileSize, double zoom) {
    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;

    std::vector<int> h_iterations(width * height);
    hipMemcpy(h_iterations.data(), d_iterations, sizeof(int) * width * height, hipMemcpyDeviceToHost);

    for (int ty = 0; ty < tilesY; ++ty) {
        for (int tx = 0; tx < tilesX; ++tx) {
            int sum = 0;
            int count = 0;

            for (int dy = 0; dy < tileSize; ++dy) {
                for (int dx = 0; dx < tileSize; ++dx) {
                    int x = tx * tileSize + dx;
                    int y = ty * tileSize + dy;
                    if (x >= width || y >= height) continue;
                    sum += h_iterations[y * width + x];
                    ++count;
                }
            }

            float avg = (count > 0) ? (float)sum / count : 0.0f;
            std::printf("[ZoomEvalCSV] %d,%d,%.4f,%.2f\n", tx, ty, zoom, avg);
        }
    }
}

} // namespace CudaInterop
