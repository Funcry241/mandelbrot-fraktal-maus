#include "hip/hip_runtime.h"
// 🐭 Maus-Kommentar: CUDA-OpenGL Interop mit vollständigem Debug-Logging

#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include <GL/gl.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <stdexcept>  // 🐭 Exception für Fehler-Handling

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"

namespace CudaInterop {

// 🐭 Fehlerbehandlung: wirf Exception statt std::exit!
#define CHECK_CUDA_STEP(call, msg) do { \
    if (hipError_t err = (call); err != hipSuccess) { \
        throw std::runtime_error(std::string("[CUDA ERROR] ") + msg + ": " + hipGetErrorString(err)); \
    } \
} while (0)

#define DEBUG_PRINT(fmt, ...) do { \
    if (Settings::debugLogging) \
        std::fprintf(stdout, "[DEBUG] " fmt "\n", ##__VA_ARGS__); \
} while (0)

void renderCudaFrame(hipGraphicsResource_t cudaPboRes, int w, int h, float& zoom, float2& offset,
                     int maxIter, float* d_complexity, std::vector<float>& h_complexity, int* d_iterations) {
    DEBUG_PRINT("Starting frame render");

    uchar4* d_img = nullptr;
    size_t imgSize = 0;
    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "MapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "GetMappedPointer");

    if (Settings::debugGradient) {
        DEBUG_PRINT("Launching debug kernel");
        launch_debugGradient(d_img, w, h);
    } else {
        DEBUG_PRINT("Launching Mandelbrot kernel");
        launch_mandelbrotHybrid(d_img, d_iterations, w, h, zoom, offset, maxIter);

        int totalTiles = static_cast<int>(h_complexity.size());
        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "Memset complexity");

        dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
        dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
        DEBUG_PRINT("Launching complexity kernel Grid(%d, %d) Block(%d, %d)", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

        computeComplexity<<<gridDim, blockDim>>>(d_iterations, w, h, d_complexity);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "complexity sync");
        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "Memcpy complexity");

        // 🐭 Debugging: Analyse der Komplexitäten
        int nonzeroTiles = 0;
        float maxComplexity = -1.0f;
        float minComplexity = 1e30f;
        float sumComplexity = 0.0f;

        for (int i = 0; i < totalTiles; ++i) {
            float val = h_complexity[i];
            if (val > 0.0f) {
                nonzeroTiles++;
                if (val > maxComplexity) maxComplexity = val;
                if (val < minComplexity) minComplexity = val;
                sumComplexity += val;
            }
        }

        float avgComplexity = (nonzeroTiles > 0) ? (sumComplexity / nonzeroTiles) : 0.0f;

        std::printf(
            "[DEBUG] Complexity Stats: Nonzero Tiles: %d / %d | Max: %.6e | Min: %.6e | Avg: %.6e\n",
            nonzeroTiles, totalTiles, maxComplexity, minComplexity, avgComplexity
        );

        DEBUG_PRINT("Searching best tile...");
        int tilesX = (w + Settings::TILE_W - 1) / Settings::TILE_W;
        float bestVariance = -1.0f;
        int bestIdx = -1;

        // 🐭 Dynamischer Threshold basierend auf Zoom
        float dynamicThreshold = Settings::dynamicVarianceThreshold(zoom);

        for (int i = 0; i < totalTiles; ++i) {
            if (h_complexity[i] > dynamicThreshold && h_complexity[i] > bestVariance) {
                bestVariance = h_complexity[i];
                bestIdx = i;
            }
        }

        // 🐭 Logging ob ein Tile gefunden wurde
        if (bestIdx == -1) {
            std::printf("[DEBUG] No suitable tile found in current frame.\n");
        } else {
            std::printf("[DEBUG] Best Tile Index: %d | Variance Score: %.6e\n", bestIdx, bestVariance);
        }

        if (bestIdx != -1) {
            int bx = bestIdx % tilesX;
            int by = bestIdx / tilesX;
            float tx = (bx + 0.5f) * Settings::TILE_W - w * 0.5f;
            float ty = (by + 0.5f) * Settings::TILE_H - h * 0.5f;
            float targetOffX = offset.x + tx / zoom;
            float targetOffY = offset.y + ty / zoom;

            if (std::isfinite(targetOffX) && std::isfinite(targetOffY)) {
                auto step = [](float delta, float factor, float zoom) {
                    float s = factor / zoom;
                    float dynamicMinStep = fmaxf(Settings::MIN_OFFSET_STEP, 1e-5f / zoom);
                    s = fmaxf(s, dynamicMinStep);

                    if (std::fabs(delta) > s)
                        delta = (delta > 0 ? s : -s);
                    if (std::fabs(delta) < dynamicMinStep)
                        delta = (delta > 0 ? dynamicMinStep : -dynamicMinStep);
                    return delta;
                };
                offset.x += step(targetOffX - offset.x, Settings::OFFSET_STEP_FACTOR, zoom);
                offset.y += step(targetOffY - offset.y, Settings::OFFSET_STEP_FACTOR, zoom);
                DEBUG_PRINT("New offset: (%.12f, %.12f)", offset.x, offset.y);
            }

            float targetZoom = zoom * Settings::zoomFactor;
            if (std::isfinite(targetZoom) && targetZoom < 1e15f) {
                float zoomDelta = targetZoom - zoom;
                float maxStep = Settings::ZOOM_STEP_FACTOR * zoom;
                if (std::fabs(zoomDelta) > maxStep)
                    zoomDelta = (zoomDelta > 0 ? maxStep : -maxStep);
                if (std::fabs(zoomDelta) < Settings::MIN_ZOOM_STEP)
                    zoomDelta = (zoomDelta > 0 ? Settings::MIN_ZOOM_STEP : -Settings::MIN_ZOOM_STEP);
                zoom += zoomDelta;
                DEBUG_PRINT("New zoom: %.12f", zoom);
            }
        }
    }
    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "UnmapResources");
    DEBUG_PRINT("Frame render complete");
}

} // namespace CudaInterop
