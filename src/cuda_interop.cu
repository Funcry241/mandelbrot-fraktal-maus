#include "hip/hip_runtime.h"
///// Otter: Nacktmull-ABI fix - Prototyp und Aufrufreihenfolge korrigiert; GPU-Iteration erzwungen.
///// Schneefuchs: Fruehes Unmap bei Fehler; kompakte [PERF]-Logs; Groessen/Tile-Sanity bleibt aktiv.
///// Maus: Deterministischer Orchestrator; ASCII-only; keine Host-Iteration mehr.
///// Datei: src/cuda_interop.cu

#include "pch.hpp"
#include "luchs_log_host.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
#include "hermelin_buffer.hpp"
#include "bear_CudaPBOResource.hpp"
#include "nacktmull_shade.cuh"

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <vector>
#include <stdexcept>
#include <cstdint>
#if !defined(__CUDA_ARCH__)
  #include <chrono>
#endif

#include "nacktmull_anchor.hpp"
#include "nacktmull_host.hpp"

// Nacktmull-Export: extern "C" + Signatur (out, d_it, w, h, zoom, offset, maxIter, tile)
extern "C" void launch_mandelbrotHybrid(
    uchar4* out, int* d_it,
    int w, int h, float zoom, float2 offset,
    int maxIter, int tile
);

// Emergency fill (nur fuer isolierte Tests)
static __global__ void fill_rgba_kernel(uchar4* dst, int w, int h, uchar4 c) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;
    dst[y * w + x] = c;
}

namespace CudaInterop {

static bear_CudaPBOResource* pboResource      = nullptr;
static bool  pauseZoom                        = false;
static bool  s_deviceInitDone                 = false;

static void*  s_hostRegEntropyPtr   = nullptr;
static size_t s_hostRegEntropyBytes = 0;
static void*  s_hostRegContrastPtr  = nullptr;
static size_t s_hostRegContrastBytes= 0;

static inline void ensureDeviceOnce() {
    if (!s_deviceInitDone) { CUDA_CHECK(hipSetDevice(0)); s_deviceInitDone = true; }
}

static inline void ensureHostPinned(std::vector<float>& vec, void*& regPtr, size_t& regBytes) {
    const size_t cap = vec.capacity();
    if (cap == 0) {
        if (regPtr) { CUDA_CHECK(hipHostUnregister(regPtr)); regPtr=nullptr; regBytes=0; }
        return;
    }
    void* ptr = static_cast<void*>(vec.data());
    const size_t bytes = cap * sizeof(float);
    if (ptr != regPtr || bytes != regBytes) {
        if (regPtr) CUDA_CHECK(hipHostUnregister(regPtr));
        CUDA_CHECK(hipHostRegister(ptr, bytes, hipHostRegisterPortable));
        regPtr  = ptr; regBytes = bytes;
        if constexpr (Settings::debugLogging) LUCHS_LOG_HOST("[PIN] host-register ptr=%p bytes=%zu", ptr, bytes);
    }
}

void registerPBO(const Hermelin::GLBuffer& pbo) {
    if (pboResource) { if constexpr (Settings::debugLogging) LUCHS_LOG_HOST("[ERROR] registerPBO: already registered!"); return; }
    ensureDeviceOnce();

    GLint prev=0; glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &prev);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo.id());
    GLint now=0; glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &now);
    if (now != (GLint)pbo.id()) {
        LUCHS_LOG_HOST("[FATAL] GL bind failed - buffer %u was not bound (GL says %d)", pbo.id(), now);
        throw std::runtime_error("glBindBuffer(GL_PIXEL_UNPACK_BUFFER) failed");
    }

    pboResource = new bear_CudaPBOResource(pbo.id());

    size_t warm=0;
    if (auto* ptr = pboResource->mapAndLog(warm)) {
        (void)ptr; pboResource->unmap();
        if constexpr (Settings::debugLogging) LUCHS_LOG_HOST("[PBO] warm-up map/unmap done (%zu bytes)", warm);
    }
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, (GLuint)prev);
}

void renderCudaFrame(
    Hermelin::CudaDeviceBuffer& d_iterations,
    Hermelin::CudaDeviceBuffer& d_entropy,
    Hermelin::CudaDeviceBuffer& d_contrast,
    int width, int height,
    float zoom, float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    std::vector<float>& h_contrast,
    float2& newOffset, bool& shouldZoom,
    int tileSize, RendererState& state
) {
#if !defined(__CUDA_ARCH__)
    const auto t0 = std::chrono::high_resolution_clock::now();
    double mapMs=0.0, mbMs=0.0, entMs=0.0, conMs=0.0;
#endif
    if (!pboResource) throw std::runtime_error("[FATAL] CUDA PBO not registered!");

    // Sanity
    if (width <= 0 || height <= 0) {
        LUCHS_LOG_HOST("[FATAL] invalid dims w=%d h=%d", width, height);
        throw std::runtime_error("invalid framebuffer dims");
    }
    if (tileSize <= 0) {
        int oldTs = tileSize;
        tileSize = Settings::BASE_TILE_SIZE > 0 ? Settings::BASE_TILE_SIZE : 16;
        LUCHS_LOG_HOST("[WARN] tileSize<=0 (%d) -> using %d", oldTs, tileSize);
    }

    const size_t totalPixels = size_t(width) * size_t(height);
    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    const size_t it_bytes       = totalPixels * sizeof(int);
    const size_t entropy_bytes  = size_t(numTiles) * sizeof(float);
    const size_t contrast_bytes = size_t(numTiles) * sizeof(float);

    if (d_iterations.size() < it_bytes || d_entropy.size() < entropy_bytes || d_contrast.size() < contrast_bytes) {
        LUCHS_LOG_HOST("[FATAL] device buffers too small it=%zu/%zu en=%zu/%zu ct=%zu/%zu",
                       d_iterations.size(), it_bytes, d_entropy.size(), entropy_bytes, d_contrast.size(), contrast_bytes);
        throw std::runtime_error("CudaInterop::renderCudaFrame: device buffers undersized");
    }

#if !defined(__CUDA_ARCH__)
    const auto tMap0 = std::chrono::high_resolution_clock::now();
#endif
    size_t surfBytes=0;
    uchar4* devSurface = static_cast<uchar4*>(pboResource->mapAndLog(surfBytes));
#if !defined(__CUDA_ARCH__)
    const auto tMap1 = std::chrono::high_resolution_clock::now();
    mapMs = std::chrono::duration<double, std::milli>(tMap1 - tMap0).count();
#endif
    if (!devSurface) throw std::runtime_error("pboResource->map() returned null");

    const size_t expectedBytes = size_t(width) * size_t(height) * sizeof(uchar4);
    if (surfBytes < expectedBytes) {
        LUCHS_LOG_HOST("[FATAL] PBO bytes mismatch have=%zu need>=%zu", surfBytes, expectedBytes);
        pboResource->unmap();
        throw std::runtime_error("PBO byte size mismatch");
    }

    // GPU launch (Nacktmull)
    (void)hipGetLastError(); // clear sticky

    hipEvent_t ev0=nullptr, ev1=nullptr;
    if constexpr (Settings::debugLogging || Settings::performanceLogging) {
        CUDA_CHECK(hipEventCreate(&ev0));
        CUDA_CHECK(hipEventCreate(&ev1));
        CUDA_CHECK(hipEventRecord(ev0, 0));
    }

    // *** KORREKTE REIHENFOLGE: (out, d_it, w, h, zoom, offset, maxIter, tile) ***
    launch_mandelbrotHybrid(
        devSurface,
        static_cast<int*>(d_iterations.get()),
        width, height,
        zoom, offset,
        maxIterations,
        tileSize
    );

    hipError_t mbErr = hipGetLastError();
    if constexpr (Settings::debugLogging || Settings::performanceLogging) {
        hipError_t syncErr = hipDeviceSynchronize();
        if (mbErr == hipSuccess && syncErr != hipSuccess) mbErr = syncErr;
    }

    const bool ok = (mbErr == hipSuccess);
    if constexpr (Settings::debugLogging) {
        LUCHS_LOG_HOST("[MB] ok=%d err=%d w=%d h=%d tile=%d itMax=%d zoom=%.6f off=(%.6f,%.6f) surf=%p bytes=%zu it_bytes=%zu",
                       ok?1:0, (int)mbErr, width, height, tileSize, maxIterations,
                       (double)zoom, (double)offset.x, (double)offset.y,
                       (void*)devSurface, surfBytes, it_bytes);
    }

    if (!ok) {
        pboResource->unmap();
#if !defined(__CUDA_ARCH__)
        const auto t1 = std::chrono::high_resolution_clock::now();
        const double totalMs = std::chrono::duration<double, std::milli>(t1 - t0).count();
        state.lastTimings.valid            = true;
        state.lastTimings.pboMap           = mapMs;
        state.lastTimings.mandelbrotTotal  = 0.0;
        state.lastTimings.mandelbrotLaunch = 0.0;
        state.lastTimings.mandelbrotSync   = 0.0;
        state.lastTimings.entropy          = 0.0;
        state.lastTimings.contrast         = 0.0;
        state.lastTimings.deviceLogFlush   = 0.0;
        if constexpr (Settings::performanceLogging) {
            LUCHS_LOG_HOST("[PERF] path=gpu FAIL mp=%.2f tt=%.2f", mapMs, totalMs);
        }
#endif
        (void)hipGetLastError();
        shouldZoom = false; newOffset = offset;
        throw std::runtime_error("CUDA failure: mandelbrot kernel");
    }

    if constexpr (Settings::debugLogging || Settings::performanceLogging) {
        CUDA_CHECK(hipEventRecord(ev1, 0));
        CUDA_CHECK(hipEventSynchronize(ev1));
        float ms=0.0f; CUDA_CHECK(hipEventElapsedTime(&ms, ev0, ev1));
#if !defined(__CUDA_ARCH__)
        mbMs = (double)ms;
#endif
        CUDA_CHECK(hipEventDestroy(ev0));
        CUDA_CHECK(hipEventDestroy(ev1));
    }

#if !defined(__CUDA_ARCH__)
    const auto tEC0 = std::chrono::high_resolution_clock::now();
#endif
    ::computeCudaEntropyContrast(
        static_cast<const int*>(d_iterations.get()),
        static_cast<float*>(d_entropy.get()),
        static_cast<float*>(d_contrast.get()),
        width, height, tileSize, maxIterations
    );
#if !defined(__CUDA_ARCH__)
    const auto tEC1 = std::chrono::high_resolution_clock::now();
    const double ecMs = std::chrono::duration<double, std::milli>(tEC1 - tEC0).count();
    entMs = ecMs * 0.5; conMs = ecMs * 0.5;
#endif

    // Host copies
    if (h_entropy.capacity()  < size_t(numTiles)) h_entropy.reserve(size_t(numTiles));
    if (h_contrast.capacity() < size_t(numTiles)) h_contrast.reserve(size_t(numTiles));
    ensureHostPinned(h_entropy,  s_hostRegEntropyPtr,  s_hostRegEntropyBytes);
    ensureHostPinned(h_contrast, s_hostRegContrastPtr, s_hostRegContrastBytes);
    h_entropy.resize(size_t(numTiles));
    h_contrast.resize(size_t(numTiles));

    CUDA_CHECK(hipMemcpy(h_entropy.data(),  d_entropy.get(),  entropy_bytes,  hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_contrast.data(), d_contrast.get(), contrast_bytes, hipMemcpyDeviceToHost));

    shouldZoom = false; newOffset = offset;

    pboResource->unmap();

#if !defined(__CUDA_ARCH__)
    const auto t1 = std::chrono::high_resolution_clock::now();
    const double totalMs = std::chrono::duration<double, std::milli>(t1 - t0).count();
    state.lastTimings.valid            = true;
    state.lastTimings.pboMap           = mapMs;
    state.lastTimings.mandelbrotTotal  = mbMs;
    state.lastTimings.mandelbrotLaunch = 0.0;
    state.lastTimings.mandelbrotSync   = 0.0;
    state.lastTimings.entropy          = entMs;
    state.lastTimings.contrast         = conMs;
    state.lastTimings.deviceLogFlush   = 0.0;

    if constexpr (Settings::performanceLogging) {
        LUCHS_LOG_HOST("[PERF] path=gpu mp=%.2f mb=%.2f en=%.2f ct=%.2f tt=%.2f",
                       mapMs, mbMs, entMs, conMs, totalMs);
    } else if constexpr (Settings::debugLogging) {
        LUCHS_LOG_HOST("[TIME] total=%.2f", totalMs);
    }
#endif
}

void setPauseZoom(bool pause) { pauseZoom = pause; }
bool getPauseZoom()           { return pauseZoom; }

bool precheckCudaRuntime() {
    int deviceCount = 0;
    hipError_t e1 = hipFree(0);
    hipError_t e2 = hipGetDeviceCount(&deviceCount);
    if constexpr (Settings::debugLogging)
        LUCHS_LOG_HOST("[CUDA] precheck err1=%d err2=%d count=%d", (int)e1, (int)e2, deviceCount);
    return e1 == hipSuccess && e2 == hipSuccess && deviceCount > 0;
}

bool verifyCudaGetErrorStringSafe() {
    hipError_t dummy = hipErrorInvalidValue;
    const char* msg = hipGetErrorString(dummy);
    if (msg) { if constexpr (Settings::debugLogging) LUCHS_LOG_HOST("[CHECK] hipGetLastError(dummy) = \"%s\"", msg); return true; }
    LUCHS_LOG_HOST("[FATAL] hipGetErrorString returned null"); return false;
}

void unregisterPBO() {
    if (s_hostRegEntropyPtr)  { hipHostUnregister(s_hostRegEntropyPtr);  s_hostRegEntropyPtr=nullptr;  s_hostRegEntropyBytes=0; }
    if (s_hostRegContrastPtr) { hipHostUnregister(s_hostRegContrastPtr); s_hostRegContrastPtr=nullptr; s_hostRegContrastBytes=0; }
    delete pboResource; pboResource = nullptr;
}

void logCudaDeviceContext(const char* tag) {
    int dev=-1; hipError_t e0=hipGetDevice(&dev);
    hipDeviceProp_t prop{}; hipError_t e1=(e0==hipSuccess && dev>=0) ? hipGetDeviceProperties(&prop, dev) : hipErrorInvalidDevice;
    if constexpr (Settings::debugLogging || Settings::performanceLogging) {
        if (e0==hipSuccess && e1==hipSuccess) {
            LUCHS_LOG_HOST("[CUDA] ctx tag=%s device=%d name=\"%s\" cc=%d.%d sms=%d vram=%lluMB",
                (tag?tag:"(null)"), dev, prop.name, prop.major, prop.minor, prop.multiProcessorCount,
                (unsigned long long)(prop.totalGlobalMem / (1024ull*1024ull)));
        } else {
            LUCHS_LOG_HOST("[CUDA] ctx tag=%s deviceQuery failed e0=%d e1=%d dev=%d",
                (tag?tag:"(null)"), (int)e0, (int)e1, dev);
        }
    }
}

} // namespace CudaInterop
