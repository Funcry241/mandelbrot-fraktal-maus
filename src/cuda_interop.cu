#include "hip/hip_runtime.h"
///// Otter: OpenGL PBO interop; map/unmap + pointer retrieval logged deterministically.
///// Schneefuchs: Precheck cuda runtime; numeric rc codes only; no getErrorString.
///// Maus: Immediate device-log flush on CUDA errors; one line per event.
///// Datei: src/cuda_interop.cu

#include "pch.hpp"
#include "luchs_log_host.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "renderer_state.hpp"
#include "hermelin_buffer.hpp"
#include "bear_CudaPBOResource.hpp"

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include <unordered_map>
#include <vector>
#include <stdexcept>
#include <cstdint>
#include <cstring>
#if !defined(__CUDA_ARCH__)
  #include <chrono>
#endif

// ---- Kernel (extern C) ------------------------------------------------------
// NEU: Stream-Parameter in der Deklaration
extern "C" void launch_mandelbrotHybrid(
    uchar4* out, uint16_t* d_it,
    int w, int h, float zoom, float2 offset,
    int maxIter, int tile,
    hipStream_t stream
);

namespace CudaInterop {

// ---- TU-lokaler Zustand -----------------------------------------------------
static bear_CudaPBOResource*                     s_pboActive = nullptr;
static std::unordered_map<GLuint, bear_CudaPBOResource*> s_pboMap;

static bool           s_pauseZoom = false;
static bool           s_deviceOk  = false;

static void*          s_hostRegEntropyPtr   = nullptr;  static size_t s_hostRegEntropyBytes   = 0;
static void*          s_hostRegContrastPtr  = nullptr;  static size_t s_hostRegContrastBytes  = 0;

static hipEvent_t    s_evStart = nullptr, s_evStop = nullptr; static bool s_evInit = false;
// NOTE [4f]: TU-lokaler Copy-Stream entfernt — kommt jetzt als Funktionsparameter.

// ---- Helpers ----------------------------------------------------------------
static inline void ensureDeviceOnce() {
    if (!s_deviceOk) { CUDA_CHECK(hipSetDevice(0)); s_deviceOk = true; }
}

static inline void ensureEventsOnce() {
    if (s_evInit) return;
    CUDA_CHECK(hipEventCreate(&s_evStart));
    CUDA_CHECK(hipEventCreate(&s_evStop));
    s_evInit = (s_evStart && s_evStop);
    if constexpr (Settings::debugLogging)
        LUCHS_LOG_HOST("[CUDA][ZK] events %s", s_evInit ? "created" : "FAILED");
}

static inline void destroyEventsIfAny() {
    if (!s_evInit) return;
    hipEventDestroy(s_evStart); s_evStart=nullptr;
    hipEventDestroy(s_evStop);  s_evStop =nullptr;
    s_evInit=false;
}

static inline void ensureHostPinned(std::vector<float>& vec, void*& regPtr, size_t& regBytes) {
    const size_t cap = vec.capacity();
    void* ptr = cap ? (void*)vec.data() : nullptr;
    const size_t bytes = cap * sizeof(float);
    if (ptr == regPtr && bytes == regBytes) return;
    if (regPtr) CUDA_CHECK(hipHostUnregister(regPtr));
    if (ptr)    CUDA_CHECK(hipHostRegister(ptr, bytes, hipHostRegisterPortable));
    regPtr = ptr; regBytes = bytes;
    if constexpr (Settings::debugLogging)
        LUCHS_LOG_HOST("[PIN] host-register ptr=%p bytes=%zu", regPtr, regBytes);
}

static inline void enforceWriteDiscard(bear_CudaPBOResource* res) {
    if (!res) return;
    if (auto* gr = res->get()) {
        (void)cudaGraphicsResourceSetMapFlags(gr, cudaGraphicsMapFlagsWriteDiscard);
    }
}

struct MapGuard {
    bear_CudaPBOResource* r=nullptr;
    void* ptr=nullptr; size_t bytes=0;
    explicit MapGuard(bear_CudaPBOResource* rr):r(rr){ if(r){ ptr=r->mapAndLog(bytes);} }
    ~MapGuard(){ if(r) r->unmap(); }
    MapGuard(const MapGuard&) = delete; MapGuard& operator=(const MapGuard&) = delete;
};

// ---- PBO-Verwaltung ---------------------------------------------------------
void registerAllPBOs(const GLuint* ids, int count) {
    ensureDeviceOnce();

    if (s_hostRegEntropyPtr)  { hipHostUnregister(s_hostRegEntropyPtr);  s_hostRegEntropyPtr=nullptr;  s_hostRegEntropyBytes=0; }
    if (s_hostRegContrastPtr) { hipHostUnregister(s_hostRegContrastPtr); s_hostRegContrastPtr=nullptr; s_hostRegContrastBytes=0; }
    destroyEventsIfAny();

    // [4f] kein TU-lokaler Copy-Stream mehr -> nichts zu zerstören hier

    for (auto &kv : s_pboMap) delete kv.second; s_pboMap.clear(); s_pboActive=nullptr;
    if (!ids || count<=0) return;

    for (int i=0;i<count;++i) {
        if (!ids[i]) continue;
        auto* res = new bear_CudaPBOResource(ids[i]);
        if (res && res->get()) {
            enforceWriteDiscard(res);
            s_pboMap[ids[i]] = res;
        } else {
            delete res;
        }
    }
    for (int i=0;i<count && !s_pboActive;++i){ auto it=s_pboMap.find(ids[i]); if(it!=s_pboMap.end()) s_pboActive=it->second; }
}

void unregisterAllPBOs() {
    if (s_hostRegEntropyPtr)  { hipHostUnregister(s_hostRegEntropyPtr);  s_hostRegEntropyPtr=nullptr;  s_hostRegEntropyBytes=0; }
    if (s_hostRegContrastPtr) { hipHostUnregister(s_hostRegContrastPtr); s_hostRegContrastPtr=nullptr; s_hostRegContrastBytes=0; }
    destroyEventsIfAny();

    // [4f] kein TU-lokaler Copy-Stream mehr -> nichts zu zerstören hier

    for (auto &kv : s_pboMap) delete kv.second; s_pboMap.clear(); s_pboActive=nullptr;
}

void registerPBO(const Hermelin::GLBuffer& pbo) {
    ensureDeviceOnce();
    const GLuint id = pbo.id();
    auto it = s_pboMap.find(id);
    if (it == s_pboMap.end()) {
        auto* res = new bear_CudaPBOResource(id);
        if (res && res->get()) {
            enforceWriteDiscard(res);
            s_pboMap[id]=res;
            if constexpr (Settings::debugLogging)
                LUCHS_LOG_HOST("[CUDA-Interop] auto-registered PBO id=%u", id);
        } else {
            delete res;
            LUCHS_LOG_HOST("[FATAL] failed to create CudaPBOResource id=%u", id);
            return;
        }
        it = s_pboMap.find(id);
    }
    s_pboActive = it->second;
}

void unregisterPBO() {
    if (s_hostRegEntropyPtr)  { hipHostUnregister(s_hostRegEntropyPtr);  s_hostRegEntropyPtr=nullptr;  s_hostRegEntropyBytes=0; }
    if (s_hostRegContrastPtr) { hipHostUnregister(s_hostRegContrastPtr); s_hostRegContrastPtr=nullptr; s_hostRegContrastBytes=0; }
    destroyEventsIfAny();

    // [4f] kein TU-lokaler Copy-Stream mehr -> nichts zu zerstören hier

    if (s_pboActive) {
        for (auto it = s_pboMap.begin(); it != s_pboMap.end(); ++it) {
            if (it->second == s_pboActive) { delete it->second; s_pboMap.erase(it); break; }
        }
        s_pboActive = nullptr;
    }
}

// ---- Hauptpfad --------------------------------------------------------------
void renderCudaFrame(
    Hermelin::CudaDeviceBuffer& d_iterations,
    Hermelin::CudaDeviceBuffer& d_entropy,
    Hermelin::CudaDeviceBuffer& d_contrast,
    int width, int height,
    float zoom, float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    std::vector<float>& h_contrast,
    float2& newOffset, bool& shouldZoom,
    int tileSize, RendererState& state,
    hipStream_t renderStream,
    hipStream_t copyStream
){
#if !defined(__CUDA_ARCH__)
    const auto t0 = std::chrono::high_resolution_clock::now();
    double mapMs=0.0, mbMs=0.0, entMs=0.0, conMs=0.0;
#endif
    if (!s_pboActive) throw std::runtime_error("[FATAL] CUDA PBO not registered!");
    if (width<=0 || height<=0)  throw std::runtime_error("invalid framebuffer dims");
    if (tileSize<=0) { int was=tileSize; tileSize = Settings::BASE_TILE_SIZE>0 ? Settings::BASE_TILE_SIZE : 16; LUCHS_LOG_HOST("[WARN] tileSize<=0 (%d) -> using %d", was, tileSize); }

    const size_t totalPx = size_t(width)*size_t(height);
    const int tilesX = (width  + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    const size_t itBytes = totalPx * sizeof(uint16_t);
    const size_t enBytes = size_t(numTiles) * sizeof(float);
    const size_t ctBytes = size_t(numTiles) * sizeof(float);

    if (d_iterations.size()<itBytes || d_entropy.size()<enBytes || d_contrast.size()<ctBytes)
        throw std::runtime_error("CudaInterop::renderCudaFrame: device buffers undersized");

#if !defined(__CUDA_ARCH__)
    const auto tMap0 = std::chrono::high_resolution_clock::now();
#endif
    MapGuard map(s_pboActive);
    if (!map.ptr) throw std::runtime_error("pboResource->map() returned null");

#if !defined(__CUDA_ARCH__)
    const auto tMap1 = std::chrono::high_resolution_clock::now();
    mapMs = std::chrono::duration<double, std::milli>(tMap1 - tMap0).count();
#endif
    const size_t needBytes = size_t(width)*size_t(height)*sizeof(uchar4);
    if (map.bytes < needBytes) throw std::runtime_error("PBO byte size mismatch");

    ensureEventsOnce();
    (void)hipGetLastError();

    // Timing-Event auf DEM Render-Stream (nicht Stream 0)
    CUDA_CHECK(hipEventRecord(s_evStart, renderStream));

    // Kernel-Launch auf dem übergebenen Stream
    launch_mandelbrotHybrid(static_cast<uchar4*>(map.ptr),
                            static_cast<uint16_t*>(d_iterations.get()),
                            width, height, zoom, offset, maxIterations, tileSize,
                            renderStream);
    hipError_t mbErrLaunch = hipGetLastError();

    // Stop-Event & Sync ebenfalls auf renderStream
    CUDA_CHECK(hipEventRecord(s_evStop, renderStream));
    hipError_t mbErrSync = hipEventSynchronize(s_evStop);

#if !defined(__CUDA_ARCH__)
    if (mbErrSync==hipSuccess) {
        float ms=0.0f; hipEventElapsedTime(&ms, s_evStart, s_evStop); mbMs = ms;
    }
#endif
    if (mbErrLaunch != hipSuccess || mbErrSync != hipSuccess)
        throw std::runtime_error("CUDA failure: mandelbrot kernel");

#if !defined(__CUDA_ARCH__)
    const auto tEC0 = std::chrono::high_resolution_clock::now();
#endif
    ::computeCudaEntropyContrast(
        static_cast<const uint16_t*>(d_iterations.get()),
        static_cast<float*>(d_entropy.get()),
        static_cast<float*>(d_contrast.get()),
        width, height, tileSize, maxIterations
    );
#if !defined(__CUDA_ARCH__)
    const auto tEC1 = std::chrono::high_resolution_clock::now();
    const double ecMs = std::chrono::duration<double, std::milli>(tEC1 - tEC0).count();
    entMs = ecMs * 0.5; conMs = ecMs * 0.5;
#endif

    // Host-Transfers (Copy-Stream wartet auf Render-Stream-Event)
    if (h_entropy.capacity()  < size_t(numTiles)) h_entropy.reserve(size_t(numTiles));
    if (h_contrast.capacity() < size_t(numTiles)) h_contrast.reserve(size_t(numTiles));
    ensureHostPinned(h_entropy,  s_hostRegEntropyPtr,  s_hostRegEntropyBytes);
    ensureHostPinned(h_contrast, s_hostRegContrastPtr, s_hostRegContrastBytes);
    h_entropy.resize(size_t(numTiles)); h_contrast.resize(size_t(numTiles));

    // [4f] expliziter copyStream aus dem RendererState
    CUDA_CHECK(hipStreamWaitEvent(copyStream, s_evStop, 0)); // warte auf Ende des Render-Streams

    CUDA_CHECK(hipMemcpyAsync(h_entropy.data(),  d_entropy.get(),  enBytes, hipMemcpyDeviceToHost, copyStream));
    CUDA_CHECK(hipMemcpyAsync(h_contrast.data(), d_contrast.get(), ctBytes, hipMemcpyDeviceToHost, copyStream));
    CUDA_CHECK(hipStreamSynchronize(copyStream));

    shouldZoom = false; newOffset = offset;

#if !defined(__CUDA_ARCH__)
    const auto t1 = std::chrono::high_resolution_clock::now();
    const double totalMs = std::chrono::duration<double, std::milli>(t1 - t0).count();
    state.lastTimings.valid            = true;
    state.lastTimings.pboMap           = mapMs;
    state.lastTimings.mandelbrotTotal  = mbMs;
    state.lastTimings.mandelbrotLaunch = 0.0;
    state.lastTimings.mandelbrotSync   = 0.0;
    state.lastTimings.entropy          = entMs;
    state.lastTimings.contrast         = conMs;
    state.lastTimings.deviceLogFlush   = 0.0;

    if constexpr (Settings::performanceLogging)
        LUCHS_LOG_HOST("[PERF][ZK] mp=%.2f mb=%.2f en=%.2f ct=%.2f tt=%.2f", mapMs, mbMs, entMs, conMs, totalMs);
#endif
}

// ---- Sonstiges API ----------------------------------------------------------
void setPauseZoom(bool pause){ s_pauseZoom = pause; }
bool getPauseZoom(){ return s_pauseZoom; }

bool precheckCudaRuntime() {
    int deviceCount = 0;
    hipError_t e1 = hipFree(0);
    hipError_t e2 = hipGetDeviceCount(&deviceCount);
    if constexpr (Settings::debugLogging)
        LUCHS_LOG_HOST("[CUDA] precheck err1=%d err2=%d count=%d", (int)e1, (int)e2, deviceCount);
    return e1==hipSuccess && e2==hipSuccess && deviceCount>0;
}

bool verifyCudaGetErrorStringSafe() {
    const char* msg = hipGetErrorString(hipErrorInvalidValue);
    if (msg) { if constexpr (Settings::debugLogging) LUCHS_LOG_HOST("[CHECK] hipGetErrorString(dummy)=\"%s\"", msg); return true; }
    LUCHS_LOG_HOST("[FATAL] hipGetErrorString returned null"); return false;
}

static inline int getAttrSafe(hipDeviceAttribute_t a, int dev){ int v=0; (void)hipDeviceGetAttribute(&v,a,dev); return v; }
void logCudaDeviceContext(const char* tag) {
    if constexpr (!(Settings::debugLogging || Settings::performanceLogging)) { (void)tag; return; }
    int dev=-1; hipError_t e0=hipGetDevice(&dev);
    int rt=0, drv=0; hipRuntimeGetVersion(&rt); hipDriverGetVersion(&drv);
    char name[256]={0};
    if (dev>=0){ hipDeviceProp_t p{}; if (hipGetDeviceProperties(&p,dev)==hipSuccess) std::strncpy(name,p.name,sizeof(name)-1); }
    if (e0==hipSuccess && dev>=0) {
        const int ccM=getAttrSafe(hipDeviceAttributeComputeCapabilityMajor,dev);
        const int ccN=getAttrSafe(hipDeviceAttributeComputeCapabilityMinor,dev);
        const int sms=getAttrSafe(hipDeviceAttributeMultiprocessorCount,dev);
        const int warp=getAttrSafe(hipDeviceAttributeWarpSize,dev);
        size_t mf=0, mt=0; hipMemGetInfo(&mf,&mt);
        LUCHS_LOG_HOST("[CUDA] ctx tag=%s rt=%d drv=%d dev=%d name=\"%s\" cc=%d.%d sms=%d warp=%d memMB free=%zu total=%zu",
            (tag?tag:"(null)"), rt, drv, dev, name, ccM, ccN, sms, warp, (mf>>20), (mt>>20));
    } else {
        LUCHS_LOG_HOST("[CUDA] ctx tag=%s deviceQuery failed e0=%d dev=%d", (tag?tag:"(null)"), (int)e0, dev);
    }
}

} // namespace CudaInterop
