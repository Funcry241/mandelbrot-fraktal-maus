#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐭 Maus-Kommentar: Verbesserte Auto-Zoom-Logik – Vermeidet Fernziele & Precision-Stalls bei hohem Zoom

#define GL_DO_NOT_INCLUDE_GL_H   // 🧠 Verhindert Konflikt mit gl.h aus GLEW
#include <GL/glew.h>             // ✅ GLEW zuerst
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <GLFW/glfw3.h>          // Fensterhandling (kein Konflikt)

#include "cuda_interop.hpp"
#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"

#include <cmath>
#include <vector>
#include <stdexcept>
#include <algorithm>
#include <cstdio>

namespace CudaInterop {

#define CHECK_CUDA_STEP(call, msg) do { \
    if (hipError_t err = (call); err != hipSuccess) { \
        throw std::runtime_error(std::string("[CUDA ERROR] ") + msg + ": " + hipGetErrorString(err)); \
    } \
} while (0)

#define DEBUG_PRINT(fmt, ...) do { \
    if (Settings::debugLogging) \
        std::fprintf(stdout, "[DEBUG] " fmt "\n", ##__VA_ARGS__); \
} while (0)

static bool pauseZoom = false;
static bool autoZoomEnabled = true;

void setPauseZoom(bool state) { pauseZoom = state; }
bool getPauseZoom() { return pauseZoom; }
bool getAutoZoomEnabled() { return autoZoomEnabled; }

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS) {
        switch (key) {
            case GLFW_KEY_SPACE:
                autoZoomEnabled = !autoZoomEnabled;
                std::printf("[INFO] Auto-Zoom %s\n", autoZoomEnabled ? "ENABLED" : "DISABLED");
                break;
            case GLFW_KEY_P:
                pauseZoom = !pauseZoom;
                std::printf("[INFO] Zoom %s\n", pauseZoom ? "PAUSED" : "RESUMED");
                break;
            default: break;
        }
    }
}

void renderCudaFrame(
    hipGraphicsResource_t cudaPboRes,
    int w,
    int h,
    float& zoom,
    float2& offset,
    int maxIter,
    float* d_complexity,
    std::vector<float>& h_complexity,
    int* d_iterations,
    bool autoZoomEnabledParam
) {
    DEBUG_PRINT("Starting frame render");

    static float2 targetOffset = offset;
    static float lastBestGradient = -1.0f;
    static int noChangeFrames = 0;

    int tileSize = Settings::dynamicTileSize(zoom);
    int tilesX = (w + tileSize - 1) / tileSize;
    int tilesY = (h + tileSize - 1) / tileSize;

    DEBUG_PRINT("TileSize dynamically adjusted to %d", tileSize);

    uchar4* d_img = nullptr;
    size_t imgSize = 0;
    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "MapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "GetMappedPointer");

    if (Settings::debugGradient) {
        launch_debugGradient(d_img, w, h, zoom);
    } else {
        launch_mandelbrotHybrid(d_img, d_iterations, w, h, zoom, offset, maxIter);

        int totalTiles = tilesX * tilesY;
        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "Memset complexity");

        dim3 blockDim(tileSize, tileSize);
        dim3 gridDim(tilesX, tilesY);
        size_t sharedMemSize = (2 * tileSize * tileSize * sizeof(float)) + (tileSize * tileSize * sizeof(int));

        DEBUG_PRINT("Launching complexity kernel Grid(%d, %d) Block(%d, %d) TileSize %d", gridDim.x, gridDim.y, blockDim.x, blockDim.y, tileSize);
        computeComplexity<<<gridDim, blockDim, sharedMemSize>>>(d_iterations, w, h, d_complexity, tileSize);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "complexity sync");
        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "Memcpy complexity");

        int currTileX = static_cast<int>((offset.x * zoom + w * 0.5f) / tileSize);
        int currTileY = static_cast<int>((offset.y * zoom + h * 0.5f) / tileSize);

        int dynamicRadius = std::clamp(static_cast<int>(std::sqrt(zoom) * Settings::DYNAMIC_RADIUS_SCALE), Settings::DYNAMIC_RADIUS_MIN, Settings::DYNAMIC_RADIUS_MAX);
        DEBUG_PRINT("Search Radius: %d", dynamicRadius);

        float bestGradient = -1.0f;
        float bestScore = -1.0f;
        int bestIdx = -1;

        for (int dy = -dynamicRadius; dy <= dynamicRadius; ++dy) {
            for (int dx = -dynamicRadius; dx <= dynamicRadius; ++dx) {
                if (dx * dx + dy * dy > dynamicRadius * dynamicRadius) continue;
                int tx = currTileX + dx;
                int ty = currTileY + dy;
                if (tx >= 0 && ty >= 0 && tx < tilesX && ty < tilesY) {
                    int idx = ty * tilesX + tx;
                    float variance = h_complexity[idx];

                    float neighborVariance = 0.0f;
                    int neighborCount = 0;
                    for (int ny = -1; ny <= 1; ++ny) {
                        for (int nx = -1; nx <= 1; ++nx) {
                            if (nx == 0 && ny == 0) continue;
                            int ntx = tx + nx, nty = ty + ny;
                            if (ntx >= 0 && nty >= 0 && ntx < tilesX && nty < tilesY) {
                                neighborVariance += h_complexity[nty * tilesX + ntx];
                                neighborCount++;
                            }
                        }
                    }
                    if (neighborCount > 0) neighborVariance /= neighborCount;

                    float gradient = fabsf(variance - neighborVariance);
                    float dist2 = dx * dx + dy * dy + 1e-5f;
                    float distanceWeight = std::pow(std::sqrt(dist2), 1.5f);
                    float score = gradient / (distanceWeight + 1.0f);

                    if (score > bestScore || (score < 1e-10f && gradient > bestGradient)) {
                        bestGradient = gradient;
                        bestScore = score;
                        bestIdx = idx;
                    }
                }
            }
        }

        if (zoom > 1e4f && bestGradient < 1e-9f) {
            DEBUG_PRINT("Resetting bestGradient due to precision stall at high zoom");
            lastBestGradient = 0.0f;
        }

        float threshold = std::max(Settings::VARIANCE_THRESHOLD, lastBestGradient * 0.98f);
        if (bestIdx != -1 && bestGradient > threshold) {
            noChangeFrames = 0;
            lastBestGradient = bestGradient;
            int bx = bestIdx % tilesX;
            int by = bestIdx / tilesX;
            float tx = (bx + 0.5f) * tileSize - w * 0.5f;
            float ty = (by + 0.5f) * tileSize - h * 0.5f;
            float newTargetX = offset.x + tx / zoom;
            float newTargetY = offset.y + ty / zoom;
            if (std::isfinite(newTargetX) && std::isfinite(newTargetY)) {
                targetOffset = { newTargetX, newTargetY };
                DEBUG_PRINT("New Target Offset: (%.12f, %.12f)", targetOffset.x, targetOffset.y);
            }
        } else {
            noChangeFrames++;
            if (noChangeFrames > 100) {
                lastBestGradient = 0.0f;
                DEBUG_PRINT("Resetting lastBestGradient after %d frames", noChangeFrames);
            }
            DEBUG_PRINT("No better tile found — continuing.");
        }

        offset.x += (targetOffset.x - offset.x) * Settings::LERP_FACTOR;
        offset.y += (targetOffset.y - offset.y) * Settings::LERP_FACTOR;
        DEBUG_PRINT("Smoothed Offset: (%.12f, %.12f)", offset.x, offset.y);
    }

    if (autoZoomEnabledParam && !pauseZoom) {
        if (std::isfinite(zoom) && zoom < 1e18f) {
            zoom += Settings::ZOOM_STEP_FACTOR * zoom;
            DEBUG_PRINT("Zoom Updated: %.12f", zoom);
        }
    }

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "UnmapResources");
    DEBUG_PRINT("Frame render complete");
}

} // namespace CudaInterop
