#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// Zeilen: 178
// 🐭 Maus-Kommentar: CUDA-Interop delegiert Zielanalyse jetzt an ZoomLogic. Kompakter, modularer, klarer. Schneefuchs: „Nur wer delegiert, bleibt flexibel.“

#include "pch.hpp"  // 💡 Muss als erstes stehen!
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
#include "zoom_logic.hpp"
#include "heatmap_overlay.hpp"  // 🔥 Overlay-Toggle per Taste

#define ENABLE_ZOOM_LOGGING 1  // Set to 0 to disable local zoom analysis logs

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource = nullptr;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    if (cudaPboResource != nullptr) {
        std::cerr << "[ERROR] registerPBO called but resource is already registered!\n";
        return;
    }
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard));
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void logZoomEvaluation(const int* d_iterations, int width, int height, int maxIterations, double zoom) {
#if ENABLE_ZOOM_LOGGING
    std::vector<int> h_iters(width * height);
    CUDA_CHECK(hipMemcpy(h_iters.data(), d_iterations, h_iters.size() * sizeof(int), hipMemcpyDeviceToHost));

    double sum = 0.0, sumSq = 0.0;
    int minIt = maxIterations;
    int maxIt = 0;
    int escapeCount = 0;

    for (int it : h_iters) {
        sum += it;
        sumSq += it * it;
        if (it < minIt) minIt = it;
        if (it > maxIt) maxIt = it;
        if (it < 5) escapeCount++;
    }

    const int total = static_cast<int>(h_iters.size());
    const double mean = sum / total;
    const double variance = (sumSq / total) - (mean * mean);
    const double escapeRatio = static_cast<double>(escapeCount) / total;

    bool valid = (escapeRatio < 0.98) && (variance > 0.05) && (mean > 5.0);

    std::printf("ZoomEval Z %.1e MeanIt %.2f VarIt %.2f Escape %.3f Min %d Max %d Valid %d\n",
        zoom, mean, variance, escapeRatio, minIt, maxIt, valid ? 1 : 0);
#endif
}

void renderCudaFrame(
    int* d_iterations,
    float* d_entropy,
    int width,
    int height,
    double zoom,
    double2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    double2& newOffset,
    bool& shouldZoom,
    int tileSize,
    RendererState& state
) {
    if (!cudaPboResource) {
        throw std::runtime_error("[FATAL] CUDA PBO not registered before renderCudaFrame.");
    }

    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));
    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));

    float zoom_f = static_cast<float>(zoom);
    float2 offset_f = make_float2(static_cast<float>(offset.x), static_cast<float>(offset.y));

    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom_f, offset_f, maxIterations);
    computeTileEntropy(d_iterations, d_entropy, width, height, tileSize, maxIterations);

    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    h_entropy.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    shouldZoom = false;

    if (!pauseZoom) {
        ZoomLogic::ZoomResult result = ZoomLogic::evaluateZoomTarget(
            h_entropy,
            offset,
            zoom_f,
            width,
            height,
            tileSize,
            make_float2(static_cast<float>(state.offset.x), static_cast<float>(state.offset.y)),
            state.zoomResult.bestIndex,
            state.zoomResult.bestEntropy,
            state.zoomResult.bestContrast
        );

        if (result.bestIndex >= 0) {
            shouldZoom = result.shouldZoom;
            newOffset = result.newOffset;
        }

#if ENABLE_ZOOM_LOGGING
        if (shouldZoom) {
            std::printf(
                "ZoomLog Z %.5e Idx %d Ent %.5f S %.5f Dist %.6f Min %.6f dE %.4f dC %.4f RelE %.3f RelC %.3f dI %d New %d\n",
                zoom_f, result.bestIndex, result.bestEntropy, result.bestScore, result.distance, result.minDistance,
                result.bestEntropy - state.zoomResult.bestEntropy,
                result.bestContrast - state.zoomResult.bestContrast,
                result.relEntropyGain, result.relContrastGain,
                (result.bestIndex != state.zoomResult.bestIndex) ? 1 : 0,
                result.isNewTarget ? 1 : 0
            );
            logZoomEvaluation(d_iterations, width, height, maxIterations, zoom);
        } else {
            float avgEntropy = 0.0f;
            int countAbove = 0;
            for (float h : h_entropy) {
                avgEntropy += h;
                if (h > Settings::VARIANCE_THRESHOLD) countAbove++;
            }
            avgEntropy /= h_entropy.size();
            std::printf("ZoomLog NoZoom TilesAbove %d AvgEntropy %.5f\n", countAbove, avgEntropy);
        }
#endif

        state.zoomResult = result;  // 🔁 ZoomResult speichern (auch Kontrast-Heatmap!)
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
}

void setPauseZoom(bool pause) {
    pauseZoom = pause;
}

bool getPauseZoom() {
    return pauseZoom;
}

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS) {
        if (key == GLFW_KEY_P || key == GLFW_KEY_SPACE) {
            pauseZoom = !pauseZoom;
            std::cout << "[Zoom] Auto-Zoom " << (pauseZoom ? "paused" : "resumed") << "\n";
        }

        if (key == GLFW_KEY_H) {
            HeatmapOverlay::toggle();
            if (Settings::debugLogging) {
                std::puts("[DEBUG] Heatmap overlay toggled (H)");
            }
        }
    }
}

} // namespace CudaInterop
