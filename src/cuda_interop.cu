#include "hip/hip_runtime.h"
// 🐭 Maus-Kommentar: CUDA-OpenGL Interop mit sanftem Zoom- und Offset-Gliding inkl. Pause-Funktion mit Leertaste + Ziel-Glättung gegen Zittern

#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <stdexcept>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"

namespace CudaInterop {

#define CHECK_CUDA_STEP(call, msg) do { \
    if (hipError_t err = (call); err != hipSuccess) { \
        throw std::runtime_error(std::string("[CUDA ERROR] ") + msg + ": " + hipGetErrorString(err)); \
    } \
} while (0)

#define DEBUG_PRINT(fmt, ...) do { \
    if (Settings::debugLogging) \
        std::fprintf(stdout, "[DEBUG] " fmt "\n", ##__VA_ARGS__); \
} while (0)

static bool pauseZoom = false;

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (key == GLFW_KEY_SPACE && action == GLFW_PRESS) {
        pauseZoom = !pauseZoom;
        if (Settings::debugLogging)
            std::fprintf(stdout, "[DEBUG] Zoom %s\n", pauseZoom ? "paused" : "resumed");
    }
}

void renderCudaFrame(
    hipGraphicsResource_t cudaPboRes,
    int w,
    int h,
    float& zoom,
    float2& offset,
    int maxIter,
    float* d_complexity,
    std::vector<float>& h_complexity,
    int* d_iterations,
    bool autoZoomEnabled
) {
    DEBUG_PRINT("Starting frame render");

    static float2 targetOffset = offset; // 🐭 Sanftes Ziel
    static float lastBestVariance = -1.0f; // 🐭 Letzte beste Varianz

    uchar4* d_img = nullptr;
    size_t imgSize = 0;
    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "MapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "GetMappedPointer");

    if (Settings::debugGradient) {
        DEBUG_PRINT("Launching debug kernel");
        launch_debugGradient(d_img, w, h);
    } else {
        DEBUG_PRINT("Launching Mandelbrot kernel");
        launch_mandelbrotHybrid(d_img, d_iterations, w, h, zoom, offset, maxIter);

        int totalTiles = static_cast<int>(h_complexity.size());
        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "Memset complexity");

        dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
        dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
        DEBUG_PRINT("Launching complexity kernel Grid(%d, %d) Block(%d, %d)", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

        computeComplexity<<<gridDim, blockDim>>>(d_iterations, w, h, d_complexity);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "complexity sync");
        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "Memcpy complexity");

        int nonzeroTiles = 0;
        float maxComplexity = -1.0f;
        float minComplexity = 1e30f;
        float sumComplexity = 0.0f;

        for (int i = 0; i < totalTiles; ++i) {
            float val = h_complexity[i];
            if (val > 0.0f) {
                nonzeroTiles++;
                if (val > maxComplexity) maxComplexity = val;
                if (val < minComplexity) minComplexity = val;
                sumComplexity += val;
            }
        }

        float avgComplexity = (nonzeroTiles > 0) ? (sumComplexity / nonzeroTiles) : 0.0f;

        DEBUG_PRINT("Complexity Stats: Nonzero Tiles: %d / %d | Max: %.6e | Min: %.6e | Avg: %.6e", nonzeroTiles, totalTiles, maxComplexity, minComplexity, avgComplexity);

        DEBUG_PRINT("Searching best tile...");
        int tilesX = (w + Settings::TILE_W - 1) / Settings::TILE_W;
        float bestVariance = -1.0f;
        int bestIdx = -1;

        float dynamicThreshold = Settings::dynamicVarianceThreshold(zoom);

        for (int i = 0; i < totalTiles; ++i) {
            if (h_complexity[i] > dynamicThreshold && h_complexity[i] > bestVariance) {
                bestVariance = h_complexity[i];
                bestIdx = i;
            }
        }

        if (bestIdx == -1) {
            DEBUG_PRINT("No suitable tile found in current frame.");
        } else {
            DEBUG_PRINT("Best Tile Index: %d | Variance Score: %.6e", bestIdx, bestVariance);

            if (bestVariance > lastBestVariance * 1.02f || lastBestVariance < 0.0f) {
                lastBestVariance = bestVariance;

                int bx = bestIdx % tilesX;
                int by = bestIdx / tilesX;
                float tx = (bx + 0.5f) * Settings::TILE_W - w * 0.5f;
                float ty = (by + 0.5f) * Settings::TILE_H - h * 0.5f;

                float newTargetX = offset.x + tx / zoom;
                float newTargetY = offset.y + ty / zoom;

                if (std::isfinite(newTargetX) && std::isfinite(newTargetY)) {
                    targetOffset.x = newTargetX;
                    targetOffset.y = newTargetY;
                    DEBUG_PRINT("New target offset set: (%.12f, %.12f)", targetOffset.x, targetOffset.y);
                }
            }
        }

        float lerpFactor = 0.05f;
        offset.x += (targetOffset.x - offset.x) * lerpFactor;
        offset.y += (targetOffset.y - offset.y) * lerpFactor;
        DEBUG_PRINT("Smoothed offset: (%.12f, %.12f)", offset.x, offset.y);
    }

    if (autoZoomEnabled && !pauseZoom) {
        if (std::isfinite(zoom) && zoom < 1e15f) {
            zoom += Settings::ZOOM_STEP_FACTOR * zoom;
            DEBUG_PRINT("Zoom updated: %.12f", zoom);
        }
    }

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "UnmapResources");
    DEBUG_PRINT("Frame render complete");
}

} // namespace CudaInterop
