#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// Zeilen: 172
// 🐅 Maus-Kommentar: CUDA/OpenGL-Interop für PBO-Mapping & Fraktalberechnung. Logging jetzt differenzierter: keine Flut, aber exakte Scores & Schwellen bei Bedarf. Schneefuchs: „Das klügste Logging ist das, das nur redet, wenn es etwas zu sagen hat.“

#include "pch.hpp"  // 💡 Muss als erstes stehen!

#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource = nullptr;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    if (cudaPboResource != nullptr) {
        std::cerr << "[ERROR] registerPBO called but resource is already registered!\n";
        return;
    }
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard));
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void renderCudaFrame(
    int* d_iterations,
    float* d_entropy,
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    float2& newOffset,
    bool& shouldZoom,
    int tileSize
) {
    if (!cudaPboResource) {
        std::cerr << "[FATAL] CUDA PBO not registered before renderCudaFrame.\n";
        std::exit(EXIT_FAILURE);
    }

    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));
    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));

    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations);
    computeTileEntropy(d_iterations, d_entropy, width, height, tileSize, maxIterations);

    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    int numTiles = tilesX * tilesY;
    h_entropy.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    shouldZoom = false;

    if (!pauseZoom) {
        const float dynamicThreshold = std::max(Settings::VARIANCE_THRESHOLD / std::log2(zoom + 2.0f), Settings::MIN_VARIANCE_THRESHOLD);

        int bestIndex = -1;
        float bestScore = -1.0f;
        float bestEntropy = 0.0f;

        for (int i = 0; i < numTiles; ++i) {
            int bx = i % tilesX;
            int by = i / tilesX;

            float centerX = (bx + 0.5f) * tileSize;
            float centerY = (by + 0.5f) * tileSize;

            float2 tileCenter = {
                (centerX - width  / 2.0f) / zoom + offset.x,
                (centerY - height / 2.0f) / zoom + offset.y
            };

            float2 delta = { tileCenter.x - offset.x, tileCenter.y - offset.y };
            float dist = std::sqrt(delta.x * delta.x + delta.y * delta.y);
            float score = h_entropy[i] / (1.0f + Settings::ENTROPY_NEARBY_BIAS * dist);

            if (h_entropy[i] > dynamicThreshold && score > bestScore) {
                bestScore = score;
                bestIndex = i;
                bestEntropy = h_entropy[i];
            }
        }

        static int lastIndex = -1;
        if (Settings::debugLogging) {
            if (bestIndex != lastIndex) {
                std::printf("[DEBUG] Zoom = %.6f | Dynamic Entropy Threshold = %.8f\n", zoom, dynamicThreshold);
                if (bestIndex >= 0) {
                    std::printf("[DEBUG] Best tile index = %d | Score = %.8f | Entropy = %.8f\n", bestIndex, bestScore, bestEntropy);
                } else {
                    std::puts("[DEBUG] No tile passed the entropy threshold. Zoom paused.");
                }
                lastIndex = bestIndex;
            }
        }

        if (bestIndex >= 0) {
            int bx = bestIndex % tilesX;
            int by = bestIndex / tilesX;

            float centerX = (bx + 0.5f) * tileSize;
            float centerY = (by + 0.5f) * tileSize;

            newOffset = {
                (centerX - width  / 2.0f) / zoom + offset.x,
                (centerY - height / 2.0f) / zoom + offset.y
            };

            shouldZoom = true;
        }
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
}

void setPauseZoom(bool pause) {
    pauseZoom = pause;
}

bool getPauseZoom() {
    return pauseZoom;
}

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS && (key == GLFW_KEY_P || key == GLFW_KEY_SPACE)) {
        pauseZoom = !pauseZoom;
        std::cout << "[Zoom] Auto-Zoom " << (pauseZoom ? "paused" : "resumed") << "\n";
    }
}

} // namespace CudaInterop
