#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐽 Maus-Kommentar: Supersampling entfernt - launch_mandelbrotHybrid jetzt minimal und direkt. Logging auf LUCHS_LOG_HOST. Otter: Klarer Fokus. Schneefuchs: deterministisch, transparent.

#include "pch.hpp"
#include "luchs_log_host.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
#include "zoom_logic.hpp"
#include <cuda_gl_interop.h>
#include <vector>

#ifndef __CUDA_ARCH__
  #include <chrono>
#endif

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource = nullptr;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    if (cudaPboResource) {
        LUCHS_LOG_HOST("[ERROR] registerPBO: already registered!");
        return;
    }

    // --- Expliziter GL-Bind-Check vor dem Binding ---
    GLint boundBefore = 0;
    glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &boundBefore);
    LUCHS_LOG_HOST("[CHECK] GL bind state BEFORE bind: %d", boundBefore);

    // --- Dummy-Unbind + echtes Bind-Kommando ---
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0); // Reset bind state
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo); // Versuche Bind durchzuführen

    // --- Expliziter GL-Bind-Check nach dem Binding ---
    GLint boundAfter = 0;
    glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &boundAfter);
    LUCHS_LOG_HOST("[CHECK] GL bind state AFTER  bind: %d (expected: %u)", boundAfter, pbo);

    // Optional: Abbruch wenn Binding fehlschlug
    if (boundAfter != static_cast<GLint>(pbo)) {
        LUCHS_LOG_HOST("[FATAL] GL bind failed - buffer %u was not bound (GL reports: %d)", pbo, boundAfter);
        throw std::runtime_error("glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo) failed - buffer not active");
    }

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[CU-PBO] Preparing to register PBO ID %u", pbo);

    hipError_t err = hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard);
    if (err != hipSuccess) {
        LUCHS_LOG_HOST("[CU-PBO] hipGraphicsGLRegisterBuffer FAILED: %s", hipGetErrorString(err));
        throw std::runtime_error("hipGraphicsGLRegisterBuffer failed");
    }

    if (Settings::debugLogging) {
        LUCHS_LOG_HOST("[CU-PBO] Registered GL buffer ID %u -> cudaPboResource: %p", pbo, (void*)cudaPboResource);
    }
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void renderCudaFrame(
    int* d_iterations, float* d_entropy, float* d_contrast,
    int width, int height, float zoom, float2 offset, int maxIterations,
    std::vector<float>& h_entropy, std::vector<float>& h_contrast,
    float2& newOffset, bool& shouldZoom, int tileSize,
    RendererState& state
) {
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[ENTER] renderCudaFrame()");

    if (!cudaPboResource)
        throw std::runtime_error("[FATAL] CUDA PBO not registered!");

#ifndef __CUDA_ARCH__
    const auto t0 = std::chrono::high_resolution_clock::now();
#endif
    
    const int totalPixels = width * height;
    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    if (Settings::debugLogging) {
        hipError_t err;

        err = hipMemset(d_iterations, 0, totalPixels * sizeof(int));
        LUCHS_LOG_HOST("hipMemset d_iterations: %d", static_cast<int>(err));
        if (err != hipSuccess) throw std::runtime_error("hipMemset d_iterations failed");

        err = hipMemset(d_entropy, 0, numTiles * sizeof(float));
        LUCHS_LOG_HOST("hipMemset d_entropy: %d", static_cast<int>(err));
        if (err != hipSuccess) throw std::runtime_error("hipMemset d_entropy failed");

        err = hipMemset(d_contrast, 0, numTiles * sizeof(float));
        LUCHS_LOG_HOST("hipMemset d_contrast: %d", static_cast<int>(err));
        if (err != hipSuccess) throw std::runtime_error("hipMemset d_contrast failed");
    }

    // --- Mapping & Prüfung ---
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[MAP] hipGraphicsMapResources → %p", (void*)cudaPboResource);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));

    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[MAP] Mapped pointer: %p (%zu bytes)", (void*)devPtr, size);

    // --- Kernel-Logik ---
    if (!devPtr) {
        LUCHS_LOG_HOST("[FATAL] Kernel skipped: surface pointer is null");
    } else if (Settings::debugLogging) {
        int dbg_before[3]{};
        CUDA_CHECK(hipMemcpy(dbg_before, d_iterations, sizeof(dbg_before), hipMemcpyDeviceToHost));

        LUCHS_LOG_HOST("[KERNEL] launch_mandelbrotHybrid(surface=%p, w=%d, h=%d, zoom=%.5f, offset=(%.5f %.5f), iter=%d)",
                       (void*)devPtr, width, height, zoom, offset.x, offset.y, maxIterations);

        launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations, tileSize);

        if (Settings::debugLogging)
            LUCHS_LOG_HOST("[KERNEL] mandelbrotKernel(...) launched");

        int dbg_after[3]{};
        CUDA_CHECK(hipMemcpy(dbg_after, d_iterations, sizeof(dbg_after), hipMemcpyDeviceToHost));
        LUCHS_LOG_HOST("[KERNEL] iters changed: %d→%d | %d→%d | %d→%d",
                       dbg_before[0], dbg_after[0],
                       dbg_before[1], dbg_after[1],
                       dbg_before[2], dbg_after[2]);
    } else {
        launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations, tileSize);
    }

    ::computeCudaEntropyContrast(d_iterations, d_entropy, d_contrast, width, height, tileSize, maxIterations);

    h_entropy.resize(numTiles);
    h_contrast.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_contrast.data(), d_contrast, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    // --- Zoomlogik ---
    shouldZoom = false;
    if (!pauseZoom) {
        const auto result = ZoomLogic::evaluateZoomTarget(
            h_entropy, h_contrast, offset, zoom, width, height, tileSize,
            state.offset, state.zoomResult.bestIndex, state.zoomResult.bestEntropy, state.zoomResult.bestContrast
        );

        if (result.bestIndex >= 0) {
            newOffset = result.newOffset;
            shouldZoom = result.shouldZoom;
            state.zoomResult = result;

            if (Settings::debugLogging) {
                LUCHS_LOG_HOST("[ZOOM] idx=%d entropy=%.3f contrast=%.3f → (%.5f %.5f) new=%d zoom=%d",
                               result.bestIndex,
                               result.bestEntropy,
                               result.bestContrast,
                               result.newOffset.x, result.newOffset.y,
                               result.isNewTarget ? 1 : 0,
                               result.shouldZoom ? 1 : 0);
            }
        } else if (Settings::debugLogging) {
            LUCHS_LOG_HOST("[ZOOM] No suitable target");
        }
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));

    if (Settings::debugLogging)
    LUCHS_LOG_HOST("[KERNEL] renderCudaFrame finished");

#ifndef __CUDA_ARCH__
    const auto t1 = std::chrono::high_resolution_clock::now();
    const float totalMs = std::chrono::duration<float, std::milli>(t1 - t0).count();
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[PERF] renderCudaFrame() = %.2f ms", totalMs);
#endif
}

void setPauseZoom(bool pause) { pauseZoom = pause; }
[[nodiscard]] bool getPauseZoom() { return pauseZoom; }

bool precheckCudaRuntime() {
    int deviceCount = 0;
    hipError_t err1 = hipFree(0); // zwingt Init
    hipError_t err2 = hipGetDeviceCount(&deviceCount);

    LUCHS_LOG_HOST("[CUDA] precheck err1=%d err2=%d count=%d", (int)err1, (int)err2, deviceCount);
    return (err1 == hipSuccess && err2 == hipSuccess && deviceCount > 0);
}

bool verifyCudaGetErrorStringSafe() {
    // 🐽 Maus-Kommentar: Wir rufen hipGetErrorString in völliger Isolation auf.
    // Schneefuchs: Wenn es hier kracht, kracht alles. Otter: Und wir wissen wenigstens warum.

    hipError_t dummy = hipErrorInvalidValue;
    const char* msg = hipGetErrorString(dummy); // potenziell kritisch

    if (msg) {
        LUCHS_LOG_HOST("[CHECK] hipGetErrorString(dummy) = \"%s\"", msg);
        LUCHS_LOG_HOST("[PASS] Host-seitige Fehleraufloesung funktioniert gefahrlos");
        return true;
    } else {
        LUCHS_LOG_HOST("[FATAL] hipGetErrorString returned null - das riecht nach Treibergift");
        return false;
    }
}

} // namespace CudaInterop
