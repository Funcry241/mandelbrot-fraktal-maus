#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu

// 🐭 Maus-Kommentar: Umgebaut auf RUNTIME-Wechsel per Settings::debugGradient, plus Reset bei Zoom-/Offset-Change

#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"

namespace CudaInterop {

// Debug-Utilities
#define CHECK_CUDA_STEP(call, msg) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        std::fprintf(stderr, "[CUDA ERROR] %s: %s\n", msg, hipGetErrorString(err)); \
    } \
}

void renderCudaFrame(
    hipGraphicsResource_t cudaPboRes,
    int                   width,
    int                   height,
    float&                zoom,
    float2&               offset,
    int                   maxIter,
    float*                d_complexity,
    std::vector<float>&   h_complexity
) {
    std::fprintf(stdout, "[INFO] Starte Frame-Render\n");

    // 1) PBO mappen → d_img holen
    uchar4* d_img = nullptr;
    size_t  imgSize = 0;

    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "hipGraphicsMapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "hipGraphicsResourceGetMappedPointer");

    if (Settings::debugGradient) {
        std::fprintf(stdout, "[INFO] Starte Debug-Gradient\n");
        launch_debugGradient(d_img, width, height);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "DebugGradient Synchronize");
    } else {
        std::fprintf(stdout, "[INFO] Starte Mandelbrot-Kernel\n");
        launch_mandelbrotHybrid(d_img, width, height, zoom, offset, maxIter);
        CHECK_CUDA_STEP(hipGetLastError(), "launch_mandelbrotHybrid");

        int totalTiles = static_cast<int>(h_complexity.size());

        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "hipMemset d_complexity");

        dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
        dim3 gridDim((width + Settings::TILE_W - 1) / Settings::TILE_W,
                     (height + Settings::TILE_H - 1) / Settings::TILE_H);
        std::fprintf(stdout, "[INFO] Starte Complexity-Kernel mit Grid (%d,%d) Block (%d,%d)\n",
            gridDim.x, gridDim.y, blockDim.x, blockDim.y);

        computeComplexity<<<gridDim, blockDim>>>(d_img, width, height, d_complexity);
        CHECK_CUDA_STEP(hipGetLastError(), "computeComplexity Kernel-Start");
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "computeComplexity Synchronize");

        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy d_complexity->h_complexity");

        std::fprintf(stdout, "[INFO] Suche Bereich mit höchster Komplexität...\n");

        int tilesX = (width + Settings::TILE_W - 1) / Settings::TILE_W;
        float bestScore = -1.0f;
        int   bestIdx   = 0;
        for (int i = 0; i < totalTiles; ++i) {
            if (h_complexity[i] > bestScore) {
                bestScore = h_complexity[i];
                bestIdx   = i;
            }
        }

        bool offsetChanged = false;
        bool zoomChanged = false;

        if (bestScore > 0.0f) {
            int bx = bestIdx % tilesX;
            int by = bestIdx / tilesX;
            float newOffX = offset.x + ((bx + 0.5f) * Settings::TILE_W - width * 0.5f) / zoom;
            float newOffY = offset.y + ((by + 0.5f) * Settings::TILE_H - height * 0.5f) / zoom;
            offsetChanged = (std::fabs(newOffX - offset.x) > 1e-6f) || (std::fabs(newOffY - offset.y) > 1e-6f);
            if (std::isfinite(newOffX) && std::isfinite(newOffY)) {
                offset.x = newOffX;
                offset.y = newOffY;
            }
            std::fprintf(stdout, "[INFO] Neue Offset-Position: (%.6f, %.6f)\n", offset.x, offset.y);
        }

        float newZoom = zoom * Settings::zoomFactor;
        constexpr float maxZoomAllowed = 1e6f;
        zoomChanged = (std::fabs(newZoom - zoom) > 1e-6f);
        if (std::isfinite(newZoom) && newZoom < maxZoomAllowed) {
            zoom = newZoom;
            std::fprintf(stdout, "[INFO] Neuer Zoom: %.6f\n", zoom);
        }

        // Reset Iterationen, wenn Zoom oder Offset sich geändert haben
        if (offsetChanged || zoomChanged) {
            std::fprintf(stdout, "[INFO] Zoom oder Offset geändert — Iterationen werden zurückgesetzt.\n");
            resetIterations();
        }
    }

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "hipGraphicsUnmapResources");

    std::fprintf(stdout, "[INFO] Frame-Render abgeschlossen\n");
}

} // namespace CudaInterop
