#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu

#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include "pch.hpp"  // 🧠 Vorab-Header: Windows, OpenGL, CUDA, Standard-C++
#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"
#include "common.hpp"

namespace CudaInterop {

static hipGraphicsResource_t cudaResource = nullptr;  // 🔗 CUDA-Handle zum OpenGL-PBO
static bool pauseZoom = false;                         // ⏸️ Zoom-Steuerung durch Nutzer

// ✂️ Deregistriert PBO von CUDA – notwendig bei Resize oder Shutdown
void unregisterPBO() {
    if (cudaResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaResource));
        cudaResource = nullptr;
    }
}

// 🔗 Registriert neues OpenGL-PBO bei CUDA
void registerPBO(GLuint pbo) {
    if (cudaResource) unregisterPBO();
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaResource, pbo, cudaGraphicsMapFlagsWriteDiscard));
}

// 🚀 Hauptfunktion für CUDA-Frame-Rendering inkl. Entropieanalyse pro Tile
void renderCudaFrame(uchar4*, int* d_iterations, float* d_entropy, float* d_stddev,
                     int width, int height, float zoom, float2 offset, int maxIter,
                     std::vector<float>& h_entropy, float2& newOffset, bool& shouldZoom, int tileSize) {

    if (!cudaResource) {
        std::fprintf(stderr, "[ERROR] CUDA resource not registered!\n");
        return;
    }

    // 🔄 CUDA<->OpenGL Mapping
    CUDA_CHECK(hipGraphicsMapResources(1, &cudaResource, 0));
    uchar4* devPtr;
    size_t size;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaResource));

    // 🌀 CUDA-Kernel starten (Fraktal + Entropie)
    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIter);
    computeTileEntropy(d_iterations, d_entropy, width, height, tileSize, maxIter);

    // 📊 Host-seitige Entropie-Puffer vorbereiten
    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    int totalTiles = tilesX * tilesY;

    h_entropy.resize(totalTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, totalTiles * sizeof(float), hipMemcpyDeviceToHost));

    // 📉 Entropie-Diagnose (optional bei Debug)
#if defined(DEBUG) || Settings::debugLogging
    float minE = 1e10f, maxE = -1.0f, sumE = 0.0f;
    for (int i = 0; i < totalTiles; ++i) {
        float e = h_entropy[i];
        minE = std::min(minE, e);
        maxE = std::max(maxE, e);
        sumE += e;
    }
    float meanE = sumE / totalTiles;
    float threshold = Settings::dynamicVarianceThreshold(zoom);
    std::printf("[DEBUG] Entropy stats: min=%.12f | max=%.12f | mean=%.12f | threshold=%.12f\n",
                minE, maxE, meanE, threshold);
#else
    float threshold = Settings::dynamicVarianceThreshold(zoom);
#endif

    // 🔍 Beste Zoom-Region bestimmen
    float bestScore = -1.0f;
    float2 bestOffset = {};
    shouldZoom = false;

    for (int y = 0; y < tilesY; ++y) {
        for (int x = 0; x < tilesX; ++x) {
            int idx = y * tilesX + x;
            float entropy = h_entropy[idx];
            if (entropy < threshold) continue;

            float2 cand = {
                offset.x + ((x + 0.5f) * tileSize - width * 0.5f) / zoom,
                offset.y + ((y + 0.5f) * tileSize - height * 0.5f) / zoom
            };

            float dist = std::hypot(cand.x - offset.x, cand.y - offset.y);
            float cent = std::hypot(cand.x + 0.75f, cand.y);  // Bias: Zentrumsnähe
            float score = entropy / (dist + 1.0f) / (cent + 0.1f);  // Heuristik

            if (score > bestScore) {
                bestScore = score;
                bestOffset = cand;
                shouldZoom = true;
            }
        }
    }

    // 🧭 Neue Zielkoordinaten setzen (falls sinnvoll)
    if (shouldZoom) {
#if defined(DEBUG)
        std::printf("[ZOOM] Best score = %.10f (threshold = %.10f)\n", bestScore, threshold);
#endif
        newOffset = bestOffset;
    }

    // 🔄 CUDA<->OpenGL Unmapping
    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaResource, 0));
}

// 🛑 Zoom-Pause-Toggle via HUD/Keybinding
bool getPauseZoom() { return pauseZoom; }
void setPauseZoom(bool p) { pauseZoom = p; }

// ⌨️ SPACE oder P zum Pausieren der Auto-Zoom-Logik
void keyCallback(GLFWwindow*, int key, int, int action, int) {
    if (action != GLFW_PRESS) return;
    if (key == GLFW_KEY_SPACE || key == GLFW_KEY_P) {
        pauseZoom = !pauseZoom;
#if defined(DEBUG)
        std::printf("[INFO] Auto-Zoom %s\n", pauseZoom ? "PAUSIERT" : "AKTIV");
#endif
    }
}

} // namespace CudaInterop
