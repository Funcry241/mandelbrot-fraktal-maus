#include "hip/hip_runtime.h"
// Datei: cuda_interop.cpp
// 🐭 Maus-Kommentar: CUDA-OpenGL Interop mit sanftem Auto-Zoom und adaptivem Target

#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include <GL/gl.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <stdexcept>  // 🐭 Exception für Fehler-Handling

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"

namespace CudaInterop {

// 🐭 Fehlerbehandlung: wirf Exception statt std::exit!
#define CHECK_CUDA_STEP(call, msg) do { \
    if (hipError_t err = (call); err != hipSuccess) { \
        throw std::runtime_error(std::string("[CUDA ERROR] ") + msg + ": " + hipGetErrorString(err)); \
    } \
} while (0)

#define DEBUG_PRINT(fmt, ...) do { \
    if (Settings::debugLogging) \
        std::fprintf(stdout, "[DEBUG] " fmt "\n", ##__VA_ARGS__); \
} while (0)

// 🐭 Globale Ziele für sanftes Gleiten
static float2 targetOffset = { Settings::initialOffsetX, Settings::initialOffsetY };
static float targetZoom = Settings::initialZoom;
static float currentBestVariance = -1.0f;

void renderCudaFrame(hipGraphicsResource_t cudaPboRes, int w, int h, float& zoom, float2& offset,
                     int maxIter, float* d_complexity, std::vector<float>& h_complexity, int* d_iterations) {
    DEBUG_PRINT("Starting frame render");

    uchar4* d_img = nullptr;
    size_t imgSize = 0;
    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "MapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "GetMappedPointer");

    if (Settings::debugGradient) {
        DEBUG_PRINT("Launching debug kernel");
        launch_debugGradient(d_img, w, h);
    } else {
        DEBUG_PRINT("Launching Mandelbrot kernel");
        launch_mandelbrotHybrid(d_img, d_iterations, w, h, zoom, offset, maxIter);

        int totalTiles = static_cast<int>(h_complexity.size());
        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "Memset complexity");

        dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
        dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
        DEBUG_PRINT("Launching complexity kernel Grid(%d, %d) Block(%d, %d)", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

        computeComplexity<<<gridDim, blockDim>>>(d_iterations, w, h, d_complexity);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "complexity sync");
        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "Memcpy complexity");

        int nonzeroTiles = 0;
        float maxComplexity = -1.0f;
        float minComplexity = 1e30f;
        float sumComplexity = 0.0f;

        for (int i = 0; i < totalTiles; ++i) {
            float val = h_complexity[i];
            if (val > 0.0f) {
                nonzeroTiles++;
                if (val > maxComplexity) maxComplexity = val;
                if (val < minComplexity) minComplexity = val;
                sumComplexity += val;
            }
        }

        float avgComplexity = (nonzeroTiles > 0) ? (sumComplexity / nonzeroTiles) : 0.0f;

        DEBUG_PRINT("Complexity Stats: Nonzero Tiles: %d / %d | Max: %.6e | Min: %.6e | Avg: %.6e", nonzeroTiles, totalTiles, maxComplexity, minComplexity, avgComplexity);

        DEBUG_PRINT("Searching best tile...");
        int tilesX = (w + Settings::TILE_W - 1) / Settings::TILE_W;
        float bestVariance = -1.0f;
        int bestIdx = -1;

        float dynamicThreshold = Settings::dynamicVarianceThreshold(zoom);

        for (int i = 0; i < totalTiles; ++i) {
            if (h_complexity[i] > dynamicThreshold && h_complexity[i] > bestVariance) {
                bestVariance = h_complexity[i];
                bestIdx = i;
            }
        }

        if (bestIdx == -1) {
            DEBUG_PRINT("No suitable tile found in current frame.");
        } else {
            DEBUG_PRINT("Best Tile Index: %d | Variance Score: %.6e", bestIdx, bestVariance);

            int bx = bestIdx % tilesX;
            int by = bestIdx / tilesX;
            float tx = (bx + 0.5f) * Settings::TILE_W - w * 0.5f;
            float ty = (by + 0.5f) * Settings::TILE_H - h * 0.5f;
            float candidateOffX = offset.x + tx / zoom;
            float candidateOffY = offset.y + ty / zoom;

            if (std::isfinite(candidateOffX) && std::isfinite(candidateOffY)) {
                if (bestVariance > 1.1f * currentBestVariance) {  // Nur wenn wirklich besser
                    currentBestVariance = bestVariance;
                    targetOffset.x = candidateOffX;
                    targetOffset.y = candidateOffY;
                    targetZoom = zoom * Settings::zoomFactor;
                    DEBUG_PRINT("New target set: (%.12f, %.12f) zoom %.12f", targetOffset.x, targetOffset.y, targetZoom);
                }
            }
        }

        // 🐭 Sanftes Gleiten
        auto smoothStep = [](float current, float target, float factor) {
            return current + (target - current) * factor;
        };

        offset.x = smoothStep(offset.x, targetOffset.x, 0.05f);
        offset.y = smoothStep(offset.y, targetOffset.y, 0.05f);
        zoom     = smoothStep(zoom, targetZoom, 0.05f);

        DEBUG_PRINT("Smoothed offset: (%.12f, %.12f) zoom %.12f", offset.x, offset.y, zoom);
    }

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "UnmapResources");
    DEBUG_PRINT("Frame render complete");
}

} // namespace CudaInterop
