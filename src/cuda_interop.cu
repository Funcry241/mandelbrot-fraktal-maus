#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐭 Maus-Kommentar: CUDA/OpenGL-Interop, Auto-Zoom via Entropieanalyse, PBO-Mapping, Key-Handling

#include "pch.hpp" // 💡 Muss als erstes stehen!

#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard));
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void renderCudaFrame(    
    int* d_iterations,
    float* d_entropy,    
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    float2& newOffset,
    bool& shouldZoom,
    int tileSize
) {
    // PBO an CUDA binden
    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));
    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));

    // CUDA-Kernel starten
    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations);

    // Entropie berechnen
    computeTileEntropy(d_iterations, d_entropy, width, height, tileSize, maxIterations);

    // Analyse zurück an Host
    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    int numTiles = tilesX * tilesY;
    h_entropy.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    // Auto-Zoom nur wenn nicht pausiert
    if (!pauseZoom) {
        int bestIndex = -1;
        float bestScore = -1.0f;

        for (int i = 0; i < numTiles; ++i) {
            if (h_entropy[i] > bestScore) {
                bestScore = h_entropy[i];
                bestIndex = i;
            }
        }

        if (bestIndex >= 0) {
            int bx = bestIndex % tilesX;
            int by = bestIndex / tilesX;

            float scaleX = 1.0f / (zoom * width);
            float scaleY = 1.0f / (zoom * height);
            float2 tileCenter = {
                offset.x + (bx + 0.5f) * tileSize * scaleX,
                offset.y + (by + 0.5f) * tileSize * scaleY
            };

#if defined(DEBUG) || defined(_DEBUG)
            float2 delta = { tileCenter.x - offset.x, tileCenter.y - offset.y };
            float dist = std::sqrt(delta.x * delta.x + delta.y * delta.y);
            std::cout << "[Zoom DEBUG] Entropy max at tile (" << bx << "," << by << ")\n";
            std::cout << "[Zoom DEBUG] Current offset: (" << offset.x << ", " << offset.y << ")\n";
            std::cout << "[Zoom DEBUG] New offset:     (" << tileCenter.x << ", " << tileCenter.y << ")\n";
            std::cout << "[Zoom DEBUG] Shift distance: " << dist << "\n";
#endif

            newOffset = tileCenter;
            shouldZoom = true;
        } else {
            shouldZoom = false;
        }
    } else {
        shouldZoom = false;
    }

    // PBO unmap
    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
}

void setPauseZoom(bool pause) {
    pauseZoom = pause;
}

bool getPauseZoom() {
    return pauseZoom;
}

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS && (key == GLFW_KEY_P || key == GLFW_KEY_SPACE)) {
        pauseZoom = !pauseZoom;
        std::cout << "[Zoom] Auto-Zoom " << (pauseZoom ? "paused" : "resumed") << "\n";
    }
}

} // namespace CudaInterop
