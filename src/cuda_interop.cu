#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐜 Schwarze Ameise: Klare Parametrisierung, deterministisches Logging, robustes Ressourcenhandling.
// 🦦 Otter: Explizite und einheitliche Übergabe aller Parameter. Fehler- und Kontextlogging überall.
// 🦊 Schneefuchs: Keine impliziten Zugriffe, transparente Speicher- und Fehlerprüfung.

#include "pch.hpp"
#include "luchs_log_host.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
#include "zoom_logic.hpp"
#include "luchs_cuda_log_buffer.hpp"
#include "hermelin_buffer.hpp"
#include <cuda_gl_interop.h>
#include <vector>

#ifndef CUDA_ARCH
#include <chrono>
#endif

namespace CudaInterop {

// ─── Test-Kernel: einfacher Farbverlauf mit Device-Log ─────────────────────
__global__ void testKernel(uchar4* out, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x == 0 && y == 0) {
        LUCHS_LOG_DEVICE("testKernel invoked on device");
    }
    if (x >= width || y >= height) return;
    int idx = y * width + x;
    unsigned char r = static_cast<unsigned char>(255 * x / width);
    unsigned char g = static_cast<unsigned char>(255 * y / height);
    unsigned char b = 0;
    out[idx] = make_uchar4(r, g, b, 255);
}

static hipGraphicsResource_t cudaPboResource = nullptr;
static bool pauseZoom = false;
static bool luchsBabyInitDone = false;

void logCudaDeviceContext(const char* context) {
    int device = -1;
    hipError_t err = hipGetDevice(&device);
    LUCHS_LOG_HOST("[CTX] %s: hipGetDevice() = %d (%s)", context, device, hipGetErrorString(err));
}

void registerPBO(const Hermelin::GLBuffer& pbo) {
    if (cudaPboResource) {
        LUCHS_LOG_HOST("[ERROR] registerPBO: already registered!");
        return;
    }

    GLint boundBefore = 0;
    glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &boundBefore);
    LUCHS_LOG_HOST("[CHECK] GL bind state BEFORE bind: %d", boundBefore);

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo.id());
    GLint boundAfter = 0;
    glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &boundAfter);
    LUCHS_LOG_HOST("[CHECK] GL bind state AFTER bind: %d (expected: %u)", boundAfter, pbo.id());

    if (boundAfter != static_cast<GLint>(pbo.id())) {
        LUCHS_LOG_HOST("[FATAL] GL bind failed - buffer %u was not bound (GL reports: %d)", pbo.id(), boundAfter);
        throw std::runtime_error("glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo) failed");
    }

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[CU-PBO] Preparing to register PBO ID %u", pbo.id());

    hipError_t err = hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo.id(), hipGraphicsRegisterFlagsWriteDiscard);
    if (err != hipSuccess) {
        LUCHS_LOG_HOST("[CU-PBO] hipGraphicsGLRegisterBuffer FAILED: %s", hipGetErrorString(err));
        throw std::runtime_error("hipGraphicsGLRegisterBuffer failed");
    }

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[CU-PBO] Registered GL buffer ID %u -> cudaPboResource: %p", pbo.id(), (void*)cudaPboResource);

    logCudaDeviceContext("after registerPBO");
}

void renderCudaFrame(
    Hermelin::CudaDeviceBuffer& d_iterations,
    Hermelin::CudaDeviceBuffer& d_entropy,
    Hermelin::CudaDeviceBuffer& d_contrast,
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    std::vector<float>& h_contrast,
    float2& newOffset,
    bool& shouldZoom,
    int tileSize,
    RendererState& state
) {
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[ENTER] renderCudaFrame()");

    logCudaDeviceContext("renderCudaFrame ENTER");

    if (!cudaPboResource)
        throw std::runtime_error("[FATAL] CUDA PBO not registered!");

    #ifndef CUDA_ARCH
    const auto t0 = std::chrono::high_resolution_clock::now();
    #endif

    const int totalPixels = width * height;
    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipMemset(d_iterations.get(), 0, d_iterations.size()));
    LUCHS_LOG_HOST("[MEM] d_iterations memset: %d pixels -> %zu bytes", totalPixels, d_iterations.size());
    CUDA_CHECK(hipMemset(d_entropy.get(),   0, d_entropy.size()));
    CUDA_CHECK(hipMemset(d_contrast.get(),  0, d_contrast.size()));

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[MAP] Mapping CUDA-GL resource %p", (void*)cudaPboResource);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));

    uchar4* devPtr = nullptr;
    size_t sizeBytes = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &sizeBytes, cudaPboResource));

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[MAP] Mapped pointer: %p (%zu bytes)", (void*)devPtr, sizeBytes);

    if (!devPtr) {
        LUCHS_LOG_HOST("[FATAL] Kernel skipped: surface pointer is null");
        return;
    }

    if (!luchsBabyInitDone) {
        LuchsLogger::initCudaLogBuffer(0);
        luchsBabyInitDone = true;
    }

    // ─── Debug-Gradient-Test ───────────────────────────────────────────────────
    LUCHS_LOG_HOST("[CHECK] debugGradient flag = %d", Settings::debugGradient);
    if (Settings::debugGradient) {
        dim3 block(16,16);
        dim3 grid((width+15)/16, (height+15)/16);
        LUCHS_LOG_HOST("[CHECK] Launching testKernel with grid=(%d,%d), block=(%d,%d)", grid.x, grid.y, block.x, block.y);
        testKernel<<<grid,block>>>(devPtr, width, height);
        CUDA_CHECK(hipDeviceSynchronize());
        LuchsLogger::flushDeviceLogToHost();
        LUCHS_LOG_HOST("[UNMAP DEBUG] PBO unmapped after testKernel");
        CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
        return;
    }
    // ────────────────────────────────────────────────────────────────────────────

    // immer Mandelbrot-Kernel starten
    if (Settings::debugLogging) {
        LUCHS_LOG_HOST("[KERNEL] launch_mandelbrotHybrid(surface=%p, w=%d, h=%d, zoom=%.5f, offset=(%.5f,%.5f), iter=%d, tile=%d)",
                       (void*)devPtr, width, height, zoom, offset.x, offset.y, maxIterations, tileSize);
    }
    launch_mandelbrotHybrid(devPtr,
                            static_cast<int*>(d_iterations.get()),
                            width, height, zoom, offset, maxIterations, tileSize);
    LuchsLogger::flushDeviceLogToHost();

    if (Settings::debugLogging) {
        int dbg_after[3] = {};
        CUDA_CHECK(hipMemcpy(dbg_after, d_iterations.get(), sizeof(dbg_after), hipMemcpyDeviceToHost));
        LUCHS_LOG_HOST("[KERNEL] iters sample: %d %d %d", dbg_after[0], dbg_after[1], dbg_after[2]);
    }

    ::computeCudaEntropyContrast(
        static_cast<const int*>(d_iterations.get()),
        static_cast<float*>(d_entropy.get()),
        static_cast<float*>(d_contrast.get()),
        width, height, tileSize, maxIterations
    );

    h_entropy.resize(numTiles);
    h_contrast.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(),  d_entropy.get(),   numTiles * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_contrast.data(), d_contrast.get(),  numTiles * sizeof(float), hipMemcpyDeviceToHost));

    shouldZoom = false;
    if (!pauseZoom) {
        auto result = ZoomLogic::evaluateZoomTarget(
            h_entropy, h_contrast, offset, zoom, width, height, tileSize,
            state.offset, state.zoomResult.bestIndex,
            state.zoomResult.bestEntropy, state.zoomResult.bestContrast
        );
        if (result.bestIndex >= 0) {
            newOffset = result.newOffset;
            shouldZoom = result.shouldZoom;
            state.zoomResult = result;
            if (Settings::debugLogging) {
                LUCHS_LOG_HOST("[ZOOM] idx=%d entropy=%.3f contrast=%.3f -> (%.5f,%.5f) new=%d zoom=%d",
                               result.bestIndex, result.bestEntropy, result.bestContrast,
                               result.newOffset.x, result.newOffset.y,
                               result.isNewTarget ? 1 : 0, result.shouldZoom ? 1 : 0);
            }
        } else if (Settings::debugLogging) {
            LUCHS_LOG_HOST("[ZOOM] No suitable target");
        }
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
    if (Settings::debugLogging) {
        LUCHS_LOG_HOST("[UNMAP] PBO unmapped successfully");
        LUCHS_LOG_HOST("[KERNEL] renderCudaFrame finished");
    }

    #ifndef CUDA_ARCH
    const auto t1 = std::chrono::high_resolution_clock::now();
    float totalMs = std::chrono::duration<float,std::milli>(t1 - t0).count();
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[PERF] renderCudaFrame() = %.2f ms", totalMs);
    #endif
}

void setPauseZoom(bool pause) { pauseZoom = pause; }
bool getPauseZoom()           { return pauseZoom; }

bool precheckCudaRuntime() {
    int deviceCount = 0;
    hipError_t e1 = hipFree(0);
    hipError_t e2 = hipGetDeviceCount(&deviceCount);
    LUCHS_LOG_HOST("[CUDA] precheck err1=%d err2=%d count=%d", (int)e1, (int)e2, deviceCount);
    return e1 == hipSuccess && e2 == hipSuccess && deviceCount > 0;
}

bool verifyCudaGetErrorStringSafe() {
    hipError_t dummy = hipErrorInvalidValue;
    const char* msg = hipGetErrorString(dummy);
    if (msg) {
        LUCHS_LOG_HOST("[CHECK] hipGetErrorString(dummy) = \"%s\"", msg);
        LUCHS_LOG_HOST("[PASS] Host-seitige Fehleraufloesung funktioniert gefahrlos");
        return true;
    } else {
        LUCHS_LOG_HOST("[FATAL] hipGetErrorString returned null");
        return false;
    }
}

void unregisterPBO() {
    if (cudaPboResource) {
        hipGraphicsUnregisterResource(cudaPboResource);
        cudaPboResource = nullptr;
        if (Settings::debugLogging)
            LUCHS_LOG_HOST("[CU-PBO] Unregistered PBO resource");
    }
}

} // namespace CudaInterop
