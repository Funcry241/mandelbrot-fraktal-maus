#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu

#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"

namespace CudaInterop {

// Debug-Utilities
#define CHECK_CUDA_STEP(call, msg) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        std::fprintf(stderr, "[CUDA ERROR] %s: %s\n", msg, hipGetErrorString(err)); \
    } \
}

void renderCudaFrame(
    hipGraphicsResource_t cudaPboRes,
    int                   width,
    int                   height,
    float&                zoom,
    float2&               offset,
    int                   maxIter,
    float*                d_complexity,
    std::vector<float>&   h_complexity
) {
    std::fprintf(stdout, "[INFO] Starte Frame-Render\n");

    // 1) PBO mappen → d_img holen
    uchar4* d_img = nullptr;
    size_t  imgSize = 0;

    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "hipGraphicsMapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "hipGraphicsResourceGetMappedPointer");

    std::fprintf(stdout, "[INFO] Starte Mandelbrot-Kernel\n");

#if defined(DEBUG_GRADIENT)
    launch_debugGradient(d_img, width, height);
    CHECK_CUDA_STEP(hipDeviceSynchronize(), "DebugGradient Synchronize");
#else
    launch_mandelbrotHybrid(d_img, width, height, zoom, offset, maxIter);
    CHECK_CUDA_STEP(hipGetLastError(), "launch_mandelbrotHybrid");

    int totalTiles = static_cast<int>(h_complexity.size());

    CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "hipMemset d_complexity");

    dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
    dim3 gridDim((width + Settings::TILE_W - 1) / Settings::TILE_W,
                 (height + Settings::TILE_H - 1) / Settings::TILE_H);
    std::fprintf(stdout, "[INFO] Starte Complexity-Kernel mit Grid (%d,%d) Block (%d,%d)\n",
        gridDim.x, gridDim.y, blockDim.x, blockDim.y);

    computeComplexity<<<gridDim, blockDim>>>(d_img, width, height, d_complexity);
    CHECK_CUDA_STEP(hipGetLastError(), "computeComplexity Kernel-Start");
    CHECK_CUDA_STEP(hipDeviceSynchronize(), "computeComplexity Synchronize");

    CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy d_complexity->h_complexity");

    std::fprintf(stdout, "[INFO] Suche Bereich mit höchster Komplexität...\n");

    int tilesX = (width + Settings::TILE_W - 1) / Settings::TILE_W;
    float bestScore = -1.0f;
    int   bestIdx   = 0;
    for (int i = 0; i < totalTiles; ++i) {
        if (h_complexity[i] > bestScore) {
            bestScore = h_complexity[i];
            bestIdx   = i;
        }
    }

    if (bestScore > 0.0f) {
        int bx = bestIdx % tilesX;
        int by = bestIdx / tilesX;
        float newOffX = offset.x + ((bx + 0.5f) * Settings::TILE_W - width * 0.5f) / zoom;
        float newOffY = offset.y + ((by + 0.5f) * Settings::TILE_H - height * 0.5f) / zoom;
        if (std::isfinite(newOffX) && std::isfinite(newOffY)) {
            offset.x = newOffX;
            offset.y = newOffY;
        }
        std::fprintf(stdout, "[INFO] Neue Offset-Position: (%.6f, %.6f)\n", offset.x, offset.y);
    }

    float newZoom = zoom * Settings::zoomFactor;
    constexpr float maxZoomAllowed = 1e6f;
    if (std::isfinite(newZoom) && newZoom < maxZoomAllowed) {
        zoom = newZoom;
        std::fprintf(stdout, "[INFO] Neuer Zoom: %.6f\n", zoom);
    }
#endif

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "hipGraphicsUnmapResources");

    std::fprintf(stdout, "[INFO] Frame-Render abgeschlossen\n");
}

} // namespace CudaInterop
