#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐜 Schwarze Ameise: Klare Parametrisierung, deterministisches Logging, robustes Ressourcenhandling.
// 🦦 Otter: Explizite und einheitliche Übergabe aller Parameter. Fehler- und Kontextlogging überall.
// 🦊 Schneefuchs: Keine impliziten Zugriffe, transparente Speicher- und Fehlerprüfung.

#include "pch.hpp"
#include "luchs_log_host.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
#include "zoom_logic.hpp"
#include "luchs_cuda_log_buffer.hpp"
#include "hermelin_buffer.hpp"
#include "bear_CudaPBOResource.hpp"
#include <cuda_gl_interop.h>
#include <vector>

#ifndef CUDA_ARCH
#include <chrono>
#endif

namespace CudaInterop {

static bear_CudaPBOResource* pboResource = nullptr;
static bool pauseZoom = false;
static bool luchsBabyInitDone = false;

void logCudaDeviceContext(const char* context) {
    int device = -1;
    hipError_t err = hipGetDevice(&device);
    LUCHS_LOG_HOST("[CTX] %s: hipGetDevice() = %d (%s)", context, device, hipGetErrorString(err));
}

void registerPBO(const Hermelin::GLBuffer& pbo) {
    if (pboResource) {
        LUCHS_LOG_HOST("[ERROR] registerPBO: already registered!");
        return;
    }

    GLint boundBefore = 0;
    glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &boundBefore);
    LUCHS_LOG_HOST("[CHECK] GL bind state BEFORE bind: %d", boundBefore);

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo.id());
    GLint boundAfter = 0;
    glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &boundAfter);
    LUCHS_LOG_HOST("[CHECK] GL bind state AFTER bind: %d (expected: %u)", boundAfter, pbo.id());

    if (boundAfter != static_cast<GLint>(pbo.id())) {
        LUCHS_LOG_HOST("[FATAL] GL bind failed - buffer %u was not bound (GL reports: %d)", pbo.id(), boundAfter);
        throw std::runtime_error("glBindBuffer(GL_PIXEL_UNPACK_BUFFER) failed");
    }

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[CU-PBO] Preparing to register PBO ID %u", pbo.id());

    pboResource = new bear_CudaPBOResource(pbo.id());

    logCudaDeviceContext("after registerPBO");
}

void renderCudaFrame(
    Hermelin::CudaDeviceBuffer& d_iterations,
    Hermelin::CudaDeviceBuffer& d_entropy,
    Hermelin::CudaDeviceBuffer& d_contrast,
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    std::vector<float>& h_contrast,
    float2& newOffset,
    bool& shouldZoom,
    int tileSize,
    RendererState& state
) {
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[ENTER] renderCudaFrame()");

    logCudaDeviceContext("renderCudaFrame ENTER");

    if (!pboResource)
        throw std::runtime_error("[FATAL] CUDA PBO not registered!");

#ifndef CUDA_ARCH
    const auto t0 = std::chrono::high_resolution_clock::now();
#endif

    const int totalPixels = width * height;
    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipMemset(d_iterations.get(), 0, d_iterations.size()));
    LUCHS_LOG_HOST("[MEM] d_iterations memset: %d pixels -> %zu bytes", totalPixels, d_iterations.size());
    CUDA_CHECK(hipMemset(d_entropy.get(),   0, d_entropy.size()));
    CUDA_CHECK(hipMemset(d_contrast.get(),  0, d_contrast.size()));

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[MAP] Using Bär to map CUDA-GL resource");

    CUDA_CHECK(hipDeviceSynchronize());

    size_t sizeBytes = 0;
    uchar4* devPtr = static_cast<uchar4*>(pboResource->mapAndLog(sizeBytes));

    if (!devPtr) {
        LUCHS_LOG_HOST("[FATAL] Kernel skipped: surface pointer is null");
        return;
    }

    if (!luchsBabyInitDone) {
        LuchsLogger::initCudaLogBuffer(0);
        luchsBabyInitDone = true;
    }

    if (Settings::debugLogging) {
        LUCHS_LOG_HOST(
            "[KERNEL] launch_mandelbrotHybrid(surface=%p, w=%d, h=%d, zoom=%.5f, offset=(%.5f,%.5f), iter=%d, tile=%d)",
            (void*)devPtr, width, height, zoom, offset.x, offset.y, maxIterations, tileSize
        );
    }
    launch_mandelbrotHybrid(
        devPtr,
        static_cast<int*>(d_iterations.get()),
        width, height, zoom, offset, maxIterations, tileSize
    );

    LuchsLogger::flushDeviceLogToHost();

    if (Settings::debugLogging) {
        int dbg_after[3] = {};
        CUDA_CHECK(hipMemcpy(dbg_after, d_iterations.get(), sizeof(dbg_after), hipMemcpyDeviceToHost));
        LUCHS_LOG_HOST("[KERNEL] iters sample: %d %d %d", dbg_after[0], dbg_after[1], dbg_after[2]);
    }

    ::computeCudaEntropyContrast(
        static_cast<const int*>(d_iterations.get()),
        static_cast<float*>(d_entropy.get()),
        static_cast<float*>(d_contrast.get()),
        width, height, tileSize, maxIterations
    );

    h_entropy.resize(numTiles);
    h_contrast.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(),  d_entropy.get(),   numTiles * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_contrast.data(), d_contrast.get(),  numTiles * sizeof(float), hipMemcpyDeviceToHost));

    shouldZoom = false;
    if (!pauseZoom) {
        auto result = ZoomLogic::evaluateZoomTarget(
            h_entropy, h_contrast, offset, zoom, width, height, tileSize,
            state.offset, state.zoomResult.bestIndex,
            state.zoomResult.bestEntropy, state.zoomResult.bestContrast
        );
        if (result.bestIndex >= 0) {
            newOffset = result.newOffset;
            shouldZoom = result.shouldZoom;
            state.zoomResult = result;
            if (Settings::debugLogging) {
                LUCHS_LOG_HOST(
                    "[ZOOM] idx=%d entropy=%.3f contrast=%.3f -> (%.5f,%.5f) new=%d zoom=%d",
                    result.bestIndex, result.bestEntropy, result.bestContrast,
                    result.newOffset.x, result.newOffset.y,
                    result.isNewTarget ? 1 : 0, result.shouldZoom ? 1 : 0
                );
            }
        } else if (Settings::debugLogging) {
            LUCHS_LOG_HOST("[ZOOM] No suitable target");
        }

        // 🦦 Otter-Test: Zoom erzwingen für Debug-Zweck
        if (!shouldZoom && Settings::debugLogging) {
            shouldZoom = true;
            newOffset = offset; // 🐜 keine Bewegung, nur Zoom aktivieren
            LUCHS_LOG_HOST("[ZOOM] Otter-Test aktiviert: Zoom erzwungen für Debug-Zweck");
        }
    }

    pboResource->unmap();

    if (Settings::debugLogging) {
        LUCHS_LOG_HOST("[UNMAP] PBO unmapped successfully");
        LUCHS_LOG_HOST("[KERNEL] renderCudaFrame finished");
    }

#ifndef CUDA_ARCH
    const auto t1 = std::chrono::high_resolution_clock::now();
    float totalMs = std::chrono::duration<float,std::milli>(t1 - t0).count();
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[PERF] renderCudaFrame() = %.2f ms", totalMs);
#endif
}

void setPauseZoom(bool pause) { pauseZoom = pause; }
bool getPauseZoom()           { return pauseZoom; }

bool precheckCudaRuntime() {
    int deviceCount = 0;
    hipError_t e1 = hipFree(0);
    hipError_t e2 = hipGetDeviceCount(&deviceCount);
    LUCHS_LOG_HOST("[CUDA] precheck err1=%d err2=%d count=%d", (int)e1, (int)e2, deviceCount);
    return e1 == hipSuccess && e2 == hipSuccess && deviceCount > 0;
}

bool verifyCudaGetErrorStringSafe() {
    hipError_t dummy = hipErrorInvalidValue;
    const char* msg = hipGetErrorString(dummy);
    if (msg) {
        LUCHS_LOG_HOST("[CHECK] hipGetErrorString(dummy) = \"%s\"", msg);
        LUCHS_LOG_HOST("[PASS] Host-seitige Fehlerauflösung funktioniert gefahrlos");
        return true;
    } else {
        LUCHS_LOG_HOST("[FATAL] hipGetErrorString returned null");
        return false;
    }
}

void unregisterPBO() {
    delete pboResource;
    pboResource = nullptr;
}

} // namespace CudaInterop
