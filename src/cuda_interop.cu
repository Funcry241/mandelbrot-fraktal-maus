#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐭 Maus-Kommentar: CUDA-OpenGL Interop mit kreisförmiger, zentrumsnaher Auto-Zoom-Optimierung

#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <stdexcept>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"

namespace CudaInterop {

#define CHECK_CUDA_STEP(call, msg) do { \
    if (hipError_t err = (call); err != hipSuccess) { \
        throw std::runtime_error(std::string("[CUDA ERROR] ") + msg + ": " + hipGetErrorString(err)); \
    } \
} while (0)

#define DEBUG_PRINT(fmt, ...) do { \
    if (Settings::debugLogging) \
        std::fprintf(stdout, "[DEBUG] " fmt "\n", ##__VA_ARGS__); \
} while (0)

static bool pauseZoom = false;
static bool autoZoomEnabled = true;

void setPauseZoom(bool state) { pauseZoom = state; }
bool getPauseZoom() { return pauseZoom; }
bool getAutoZoomEnabled() { return autoZoomEnabled; }

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS) {
        switch (key) {
            case GLFW_KEY_SPACE:
                autoZoomEnabled = !autoZoomEnabled;
                std::printf("[INFO] Auto-Zoom %s\n", autoZoomEnabled ? "ENABLED" : "DISABLED");
                break;
            case GLFW_KEY_P:
                pauseZoom = !pauseZoom;
                std::printf("[INFO] Zoom %s\n", pauseZoom ? "PAUSED" : "RESUMED");
                break;
            default:
                break;
        }
    }
}

void renderCudaFrame(
    hipGraphicsResource_t cudaPboRes,
    int w,
    int h,
    float& zoom,
    float2& offset,
    int maxIter,
    float* d_complexity,
    std::vector<float>& h_complexity,
    int* d_iterations,
    bool autoZoomEnabledParam
) {
    DEBUG_PRINT("Starting frame render");

    static float2 targetOffset = offset;
    static float lastBestVariance = -1.0f;

    uchar4* d_img = nullptr;
    size_t imgSize = 0;
    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "MapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "GetMappedPointer");

    if (Settings::debugGradient) {
        DEBUG_PRINT("Launching debug kernel");
        launch_debugGradient(d_img, w, h);
    } else {
        DEBUG_PRINT("Launching Mandelbrot kernel");
        launch_mandelbrotHybrid(d_img, d_iterations, w, h, zoom, offset, maxIter);

        int totalTiles = static_cast<int>(h_complexity.size());
        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "Memset complexity");

        dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
        dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
        DEBUG_PRINT("Launching complexity kernel Grid(%d, %d) Block(%d, %d)", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

        computeComplexity<<<gridDim, blockDim>>>(d_iterations, w, h, d_complexity);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "complexity sync");
        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "Memcpy complexity");

        int nonzeroTiles = 0;
        float maxComplexity = -1.0f, minComplexity = 1e30f, sumComplexity = 0.0f;

        for (float val : h_complexity) {
            if (val > 0.0f) {
                nonzeroTiles++;
                maxComplexity = std::max(maxComplexity, val);
                minComplexity = std::min(minComplexity, val);
                sumComplexity += val;
            }
        }

        float avgComplexity = (nonzeroTiles > 0) ? (sumComplexity / nonzeroTiles) : 0.0f;

        DEBUG_PRINT("Complexity Stats: Nonzero: %d / %d | Max: %.6e | Min: %.6e | Avg: %.6e", nonzeroTiles, totalTiles, maxComplexity, minComplexity, avgComplexity);

        DEBUG_PRINT("Searching best tile locally with center focus...");
        int tilesX = (w + Settings::TILE_W - 1) / Settings::TILE_W;
        int tilesY = (h + Settings::TILE_H - 1) / Settings::TILE_H;
        int currTileX = static_cast<int>((offset.x * zoom + w * 0.5f) / Settings::TILE_W);
        int currTileY = static_cast<int>((offset.y * zoom + h * 0.5f) / Settings::TILE_H);

        int searchRadius = 5;
        float bestScore = -1.0f;
        int bestIdx = -1;

        for (int dy = -searchRadius; dy <= searchRadius; ++dy) {
            for (int dx = -searchRadius; dx <= searchRadius; ++dx) {
                if (dx * dx + dy * dy > searchRadius * searchRadius) continue;
                int tx = currTileX + dx;
                int ty = currTileY + dy;
                if (tx >= 0 && ty >= 0 && tx < tilesX && ty < tilesY) {
                    int idx = ty * tilesX + tx;
                    float variance = h_complexity[idx];
                    float dist2 = dx * dx + dy * dy + 1e-5f;
                    float score = variance / dist2;
                    if (score > bestScore) {
                        bestScore = score;
                        bestIdx = idx;
                    }
                }
            }
        }

        if (bestIdx != -1) {
            DEBUG_PRINT("Best Local Tile: %d | Score: %.6e", bestIdx, bestScore);
            if (bestScore > lastBestVariance * 1.02f || lastBestVariance < 0.0f) {
                lastBestVariance = bestScore;
                int bx = bestIdx % tilesX;
                int by = bestIdx / tilesX;
                float tx = (bx + 0.5f) * Settings::TILE_W - w * 0.5f;
                float ty = (by + 0.5f) * Settings::TILE_H - h * 0.5f;
                float newTargetX = offset.x + tx / zoom;
                float newTargetY = offset.y + ty / zoom;
                if (std::isfinite(newTargetX) && std::isfinite(newTargetY)) {
                    targetOffset = { newTargetX, newTargetY };
                    DEBUG_PRINT("New Target Offset: (%.12f, %.12f)", targetOffset.x, targetOffset.y);
                }
            }
        } else {
            DEBUG_PRINT("No suitable tile found locally.");
        }

        offset.x += (targetOffset.x - offset.x) * Settings::LERP_FACTOR;
        offset.y += (targetOffset.y - offset.y) * Settings::LERP_FACTOR;
        DEBUG_PRINT("Smoothed Offset: (%.12f, %.12f)", offset.x, offset.y);
    }

    if (autoZoomEnabledParam && !pauseZoom) {
        if (std::isfinite(zoom) && zoom < 1e15f) {
            zoom += Settings::ZOOM_STEP_FACTOR * zoom;
            DEBUG_PRINT("Zoom Updated: %.12f", zoom);
        }
    }

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "UnmapResources");
    DEBUG_PRINT("Frame render complete");
}

} // namespace CudaInterop
