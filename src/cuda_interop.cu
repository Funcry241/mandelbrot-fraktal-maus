#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// Zeilen: 316
/* 🐭 Maus-Kommentar: CUDA-Interop mit Entropie und Kontrast für Heatmap und Auto-Zoom.
   Flugente: Alle Koordinaten sind jetzt float2! Kein double2 mehr im Spiel.
   Panda: Entropie+Kontrast-Analyse bleibt vollständig erhalten.
   Schneefuchs: „Wer float kann, braucht kein double – solange keine Galaxie explodiert.“
   Log bleibt ASCII-only.
*/

#include "pch.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
#include "zoom_logic.hpp"
#include "heatmap_overlay.hpp"
#include <vector>
#include <cstdio>

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource = nullptr;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    if (cudaPboResource != nullptr) {
        std::cerr << "[ERROR] registerPBO called but resource is already registered!\n";
        return;
    }
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard));
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void renderCudaFrame(
    int* d_iterations,
    float* d_entropy,
    float* d_contrast,
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    std::vector<float>& h_contrast,
    float2& newOffset,
    bool& shouldZoom,
    int tileSize,
    int supersampling,
    RendererState& state
) {
    if (!cudaPboResource) {
        throw std::runtime_error("[FATAL] CUDA PBO not registered before renderCudaFrame.");
    }

    if (Settings::debugLogging) {
        std::printf("[Zoom] Auto-Zoom is %s\n", pauseZoom ? "PAUSED" : "ACTIVE");
    }

    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));
    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));

    if (Settings::debugLogging) {
        std::printf("[DEBUG] PBO mapped: %p (size = %zu)\n", (void*)devPtr, size);
    }

    if (Settings::debugLogging) {
        std::printf("[DEBUG] Launch MandelbrotKernel zoom %.2f maxIter %d supersampling %d\n", zoom, maxIterations, supersampling);
    }

    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations, supersampling);

    hipDeviceSynchronize();
    hipError_t kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        std::fprintf(stderr, "[CUDA ERROR] MandelbrotKernel launch failed: %s\n", hipGetErrorString(kernelErr));
    }

    computeEntropyContrast(d_iterations, d_entropy, d_contrast, width, height, tileSize, maxIterations);

    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    h_entropy.resize(numTiles);
    h_contrast.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_contrast.data(), d_contrast, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    shouldZoom = false;

    if (!pauseZoom) {
        // Volle Flugente: float2 überall!
        ZoomLogic::ZoomResult result = ZoomLogic::evaluateZoomTarget(
            h_entropy,
            h_contrast,
            offset,
            zoom,
            width,
            height,
            tileSize,
            state.offset,
            state.zoomResult.bestIndex,
            state.zoomResult.bestEntropy,
            state.zoomResult.bestContrast
        );

        if (result.bestIndex >= 0) {
            newOffset = result.newOffset;   // float2 zu float2 – keine Umwandlung
            shouldZoom = result.shouldZoom;

            if (result.isNewTarget) {
                state.zoomResult.bestEntropy  = result.bestEntropy;
                state.zoomResult.bestContrast = result.bestContrast;
                state.zoomResult.bestIndex    = result.bestIndex;
            }
        }

        if (Settings::debugLogging) {
            if (result.bestIndex >= 0) {
                float minJump = Settings::MIN_JUMP_DISTANCE / zoom;
                std::printf(
                    "Zoom Z %.1e I %d E %.3f C %.3f S %.3f dO %.2e dPx %.1f minJ %.2e dE %.3f dC %.3f RelE %.2f RelC %.2f New %d\n",
                    zoom,
                    result.bestIndex,
                    result.bestEntropy,
                    result.bestContrast,
                    result.bestScore,
                    result.distance,
                    result.distance * zoom * width,
                    minJump,
                    result.relEntropyGain,
                    result.relContrastGain,
                    result.relEntropyGain,
                    result.relContrastGain,
                    result.isNewTarget ? 1 : 0
                );
            } else {
                float avgEntropy = 0.0f;
                int countAbove = 0;
                for (float h : h_entropy) {
                    avgEntropy += h;
                    if (h > Settings::VARIANCE_THRESHOLD) countAbove++;
                }
                avgEntropy /= h_entropy.size();
                std::printf("Zoom NoZoom TilesAbove %d AvgEntropy %.5f\n", countAbove, avgEntropy);
            }
        }

        if (!result.isNewTarget) {
            state.zoomResult = result;
        }
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
}

void setPauseZoom(bool pause) {
    pauseZoom = pause;
}

bool getPauseZoom() {
    return pauseZoom;
}

void logZoomEvaluation(const int* d_iterations, int width, int height, int tileSize, float zoom) {
    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;

    std::vector<int> h_iterations(width * height);
    hipMemcpy(h_iterations.data(), d_iterations, sizeof(int) * width * height, hipMemcpyDeviceToHost);

    for (int ty = 0; ty < tilesY; ++ty) {
        for (int tx = 0; tx < tilesX; ++tx) {
            int sum = 0;
            int count = 0;

            for (int dy = 0; dy < tileSize; ++dy) {
                for (int dx = 0; dx < tileSize; ++dx) {
                    int x = tx * tileSize + dx;
                    int y = ty * tileSize + dy;
                    if (x >= width || y >= height) continue;
                    sum += h_iterations[y * width + x];
                    ++count;
                }
            }

            float avg = (count > 0) ? (float)sum / count : 0.0f;
            std::printf("[ZoomEvalCSV] %d,%d,%.4f,%.2f\n", tx, ty, zoom, avg);
        }
    }
}

} // namespace CudaInterop
