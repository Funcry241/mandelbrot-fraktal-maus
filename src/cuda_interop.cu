#include "hip/hip_runtime.h"
// ASCII-Only CUDA-Interop für Mandelbrot-Renderer – PBO-Mapping, Fraktal-Rendering & Auto-Zoom mit Entropieanalyse

#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <algorithm>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"
#include "common.hpp"

namespace CudaInterop {

static hipGraphicsResource_t cudaResource = nullptr;
static bool pauseZoom = false;

// ✂️ Deregistriert PBO von CUDA – notwendig bei Resize oder Shutdown
void unregisterPBO() {
    if (cudaResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaResource));
        cudaResource = nullptr;
    }
}

// 🔗 Registriert neues OpenGL-PBO bei CUDA
void registerPBO(GLuint pbo) {
    if (cudaResource) unregisterPBO();
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaResource, pbo, cudaGraphicsMapFlagsWriteDiscard));
}

// 🚀 Hauptfunktion für CUDA-Frame-Rendering inkl. Entropieanalyse pro Tile
void renderCudaFrame(uchar4*, int* d_iterations, float* d_entropy, float* d_stddev,
                     int width, int height, float zoom, float2 offset, int maxIter,
                     std::vector<float>& h_entropy, float2& newOffset, bool& shouldZoom, int tileSize) {

    if (!cudaResource) {
        std::fprintf(stderr, "[ERROR] CUDA resource not registered!\n");
        return;
    }

    // 🔄 CUDA<->OpenGL Mapping
    CUDA_CHECK(hipGraphicsMapResources(1, &cudaResource, 0));
    uchar4* devPtr;
    size_t size;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaResource));

    // 🌀 CUDA-Kernel starten (Fraktal + Entropie)
    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIter);
    computeTileEntropy(d_iterations, d_entropy, width, height, tileSize, maxIter);

    // 📊 Host-seitige Entropie-Puffer vorbereiten
    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    int totalTiles = tilesX * tilesY;

    h_entropy.resize(totalTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, totalTiles * sizeof(float), hipMemcpyDeviceToHost));

    // 📉 Entropie-Diagnose
#if defined(DEBUG) || Settings::debugLogging
    float minE = 1e10f, maxE = -1.0f, sumE = 0.0f;
    for (int i = 0; i < totalTiles; ++i) {
        float e = h_entropy[i];
        minE = std::min(minE, e);
        maxE = std::max(maxE, e);
        sumE += e;
    }
    float meanE = sumE / totalTiles;
    float threshold = Settings::dynamicVarianceThreshold(zoom);
    std::printf("[DEBUG] Entropy stats: min=%.12f | max=%.12f | mean=%.12f | threshold=%.12f\n",
                minE, maxE, meanE, threshold);
#else
    float threshold = Settings::dynamicVarianceThreshold(zoom);
#endif

    // 🔍 Beste Zoom-Region bestimmen
    float bestScore = -1.0f;
    float2 bestOffset = {};
    shouldZoom = false;

    for (int y = 0; y < tilesY; ++y) {
        for (int x = 0; x < tilesX; ++x) {
            int idx = y * tilesX + x;
            float entropy = h_entropy[idx];
            if (entropy < threshold) continue;

            float2 cand = {
                offset.x + ((x + 0.5f) * tileSize - width * 0.5f) / zoom,
                offset.y + ((y + 0.5f) * tileSize - height * 0.5f) / zoom
            };

            float dist = std::hypot(cand.x - offset.x, cand.y - offset.y);
            float cent = std::hypot(cand.x + 0.75f, cand.y);
            float score = entropy / (dist + 1.0f) / (cent + 0.1f);

            if (score > bestScore) {
                bestScore = score;
                bestOffset = cand;
                shouldZoom = true;
            }
        }
    }

    // 🧭 Neue Zielkoordinaten setzen (falls sinnvoll)
    if (shouldZoom) {
#if defined(DEBUG)
        std::printf("[ZOOM] Best score = %.10f (threshold = %.10f)\n", bestScore, threshold);
#endif
        newOffset = bestOffset;
    }

    // 🔄 CUDA<->OpenGL Unmapping
    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaResource, 0));
}

// 🛑 Zoom-Pause-Toggle via HUD/Keybinding
bool getPauseZoom() { return pauseZoom; }
void setPauseZoom(bool p) { pauseZoom = p; }

// ⌨️ SPACE oder P zum Pausieren der Auto-Zoom-Logik
void keyCallback(GLFWwindow*, int key, int, int action, int) {
    if (action != GLFW_PRESS) return;
    if (key == GLFW_KEY_SPACE || key == GLFW_KEY_P) {
        pauseZoom = !pauseZoom;
#if defined(DEBUG)
        std::printf("[INFO] Auto-Zoom %s\n", pauseZoom ? "PAUSIERT" : "AKTIV");
#endif
    }
}

} // namespace CudaInterop
