#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐭 Maus-Kommentar: Logging auf LUCHS_LOG_HOST umgestellt. Kein iostream mehr. Schneefuchs: deterministisch.

#include "pch.hpp"
#include "luchs_log_host.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
#include "zoom_logic.hpp"
#include <cuda_gl_interop.h>
#include <vector>
#include <cstdio>
#include <iomanip>
#include <sstream>

#ifndef __CUDA_ARCH__
  #include <chrono>
#endif

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource = nullptr;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    if (cudaPboResource) {
        LUCHS_LOG_HOST("[ERROR] registerPBO: already registered!");
        return;
    }

    LUCHS_LOG_HOST("[CU-INFO] registerPBO called with pbo=%u", pbo);

    hipError_t err = hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard);
    if (err != hipSuccess) {
        LUCHS_LOG_HOST("[CU-ERROR] hipGraphicsGLRegisterBuffer failed: %s", hipGetErrorString(err));
    } else {
        LUCHS_LOG_HOST("[CU-INFO] hipGraphicsGLRegisterBuffer succeeded");
    }
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void renderCudaFrame(
    int* d_iterations, float* d_entropy, float* d_contrast,
    int width, int height, float zoom, float2 offset, int maxIterations,
    std::vector<float>& h_entropy, std::vector<float>& h_contrast,
    float2& newOffset, bool& shouldZoom, int tileSize,
    RendererState& state
) {
    if (!cudaPboResource)
        throw std::runtime_error("[FATAL] CUDA PBO not registered!");

#ifndef __CUDA_ARCH__
    const auto t0 = std::chrono::high_resolution_clock::now();
#endif

    const int totalPixels = width * height;
    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    if (Settings::debugLogging) {
        CUDA_CHECK(hipMemset(d_iterations, 0, totalPixels * sizeof(int)));
        CUDA_CHECK(hipMemset(d_entropy, 0, numTiles * sizeof(float)));
        CUDA_CHECK(hipMemset(d_contrast, 0, numTiles * sizeof(float)));
    }

    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));
    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));

    if (Settings::debugLogging) {
        LUCHS_LOG_HOST("[CU-FRAME] zoom=%.5f offset=(%.5f %.5f) iter=%d tile=%d",
                       zoom, offset.x, offset.y, maxIterations, tileSize);
    }

    if (Settings::debugLogging) {
        int dbg_before[3]{};
        CUDA_CHECK(hipMemcpy(dbg_before, d_iterations, sizeof(dbg_before), hipMemcpyDeviceToHost));
        launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations, tileSize);
        int dbg_after[3]{};
        CUDA_CHECK(hipMemcpy(dbg_after, d_iterations, sizeof(dbg_after), hipMemcpyDeviceToHost));
        LUCHS_LOG_HOST("[CU-KERNEL] iters: %d->%d | %d->%d | %d->%d",
                       dbg_before[0], dbg_after[0],
                       dbg_before[1], dbg_after[1],
                       dbg_before[2], dbg_after[2]);
    } else {
        launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations, tileSize);
    }

    computeCudaEntropyContrast(d_iterations, d_entropy, d_contrast, width, height, tileSize, maxIterations);

    h_entropy.resize(numTiles);
    h_contrast.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_contrast.data(), d_contrast, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    shouldZoom = false;
    if (!pauseZoom) {
        const auto result = ZoomLogic::evaluateZoomTarget(
            h_entropy, h_contrast, offset, zoom, width, height, tileSize,
            state.offset, state.zoomResult.bestIndex, state.zoomResult.bestEntropy, state.zoomResult.bestContrast
        );
        if (result.bestIndex >= 0) {
            newOffset = result.newOffset;
            shouldZoom = result.shouldZoom;
            state.zoomResult = result;

            if (Settings::debugLogging) {
                LUCHS_LOG_HOST("[CU-ZOOM] idx=%d entropy=%.3f contrast=%.3f -> (%.5f %.5f) new=%d zoom=%d",
                               result.bestIndex,
                               result.bestEntropy,
                               result.bestContrast,
                               result.newOffset.x, result.newOffset.y,
                               result.isNewTarget ? 1 : 0,
                               result.shouldZoom ? 1 : 0);
            }
        } else if (Settings::debugLogging) {
            LUCHS_LOG_HOST("[CU-ZOOM] No suitable target");
        }
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));

#ifndef __CUDA_ARCH__
    const auto t1 = std::chrono::high_resolution_clock::now();
    const float totalMs = std::chrono::duration<float, std::milli>(t1 - t0).count();
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[CU-PERF] total=%.2f ms", totalMs);
#endif
}

void setPauseZoom(bool pause) { pauseZoom = pause; }
[[nodiscard]] bool getPauseZoom() { return pauseZoom; }

} // namespace CudaInterop
