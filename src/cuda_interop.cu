#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu

#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"

namespace CudaInterop {

#define CHECK_CUDA_STEP(call, msg) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        std::fprintf(stderr, "[CUDA ERROR] %s: %s\n", msg, hipGetErrorString(err)); \
    } \
}

#define DEBUG_PRINT(fmt, ...) \
    do { if (Settings::debugLogging) { std::fprintf(stdout, "[DEBUG] " fmt "\n", ##__VA_ARGS__); } } while(0)

void renderCudaFrame(
    hipGraphicsResource_t cudaPboRes,
    int                   width,
    int                   height,
    float&                zoom,
    float2&               offset,
    int                   maxIter,
    float*                d_complexity,
    std::vector<float>&   h_complexity
) {
    DEBUG_PRINT("Starte Frame-Render");

    uchar4* d_img = nullptr;
    size_t  imgSize = 0;

    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "hipGraphicsMapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "hipGraphicsResourceGetMappedPointer");

    if (Settings::debugGradient) {
        DEBUG_PRINT("Starte Debug-Gradient");
        launch_debugGradient(d_img, width, height);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "DebugGradient Synchronize");
    } else {
        DEBUG_PRINT("Starte Mandelbrot-Kernel");
        launch_mandelbrotHybrid(d_img, width, height, zoom, offset, maxIter);
        CHECK_CUDA_STEP(hipGetLastError(), "launch_mandelbrotHybrid");

        int totalTiles = static_cast<int>(h_complexity.size());

        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "hipMemset d_complexity");

        dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
        dim3 gridDim((width + Settings::TILE_W - 1) / Settings::TILE_W,
                     (height + Settings::TILE_H - 1) / Settings::TILE_H);

        DEBUG_PRINT("Starte Complexity-Kernel mit Grid (%d,%d) Block (%d,%d)", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

        computeComplexity<<<gridDim, blockDim>>>(d_img, width, height, d_complexity);
        CHECK_CUDA_STEP(hipGetLastError(), "computeComplexity Kernel-Start");
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "computeComplexity Synchronize");

        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy d_complexity->h_complexity");

        DEBUG_PRINT("Suche Bereich mit größter Varianz...");

        int tilesX = (width + Settings::TILE_W - 1) / Settings::TILE_W;
        float bestVariance = -1.0f;
        int   bestIdx = 0;

        for (int i = 0; i < totalTiles; ++i) {
            if (h_complexity[i] > bestVariance) {
                bestVariance = h_complexity[i];
                bestIdx = i;
            }
        }

        DEBUG_PRINT("Beste gefundene Varianz: %.6f", bestVariance);

        bool offsetChanged = false;
        bool zoomChanged = false;

        if (bestVariance > 0.0f) {
            int bx = bestIdx % tilesX;
            int by = bestIdx / tilesX;
            float newOffX = offset.x + ((bx + 0.5f) * Settings::TILE_W - width * 0.5f) / zoom;
            float newOffY = offset.y + ((by + 0.5f) * Settings::TILE_H - height * 0.5f) / zoom;

            offsetChanged = (std::fabs(newOffX - offset.x) > 1e-6f) || (std::fabs(newOffY - offset.y) > 1e-6f);
            if (std::isfinite(newOffX) && std::isfinite(newOffY)) {
                offset.x = newOffX;
                offset.y = newOffY;
            }

            DEBUG_PRINT("Neue Offset-Position: (%.6f, %.6f)", offset.x, offset.y);
        }

        float newZoom = zoom * Settings::zoomFactor;
        constexpr float maxZoomAllowed = 1e15f; // 🐭 deutlich höher für Deep-Zooms

        zoomChanged = (std::fabs(newZoom - zoom) > 1e-6f);
        if (std::isfinite(newZoom) && newZoom < maxZoomAllowed) {
            zoom = newZoom;
            DEBUG_PRINT("Neuer Zoom: %.6f", zoom);
        }

        if (offsetChanged || zoomChanged) {
            DEBUG_PRINT("Zoom oder Offset geändert — Iterationen werden zurückgesetzt.");
            resetIterations();
        }
    }

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "hipGraphicsUnmapResources");

    DEBUG_PRINT("Frame-Render abgeschlossen");
}

} // namespace CudaInterop
