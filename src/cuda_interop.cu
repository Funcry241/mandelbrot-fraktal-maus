#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🍝 Maus-Kommentar: Auto-Zoom mit Gradient-Erkennung für wirklich interessante Fraktalbereiche

#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <stdexcept>
#include <algorithm>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"

namespace CudaInterop {

#define CHECK_CUDA_STEP(call, msg) do { \
    if (hipError_t err = (call); err != hipSuccess) { \
        throw std::runtime_error(std::string("[CUDA ERROR] ") + msg + ": " + hipGetErrorString(err)); \
    } \
} while (0)

#define DEBUG_PRINT(fmt, ...) do { \
    if (Settings::debugLogging) \
        std::fprintf(stdout, "[DEBUG] " fmt "\n", ##__VA_ARGS__); \
} while (0)

static bool pauseZoom = false;
static bool autoZoomEnabled = true;

void setPauseZoom(bool state) { pauseZoom = state; }
bool getPauseZoom() { return pauseZoom; }
bool getAutoZoomEnabled() { return autoZoomEnabled; }

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS) {
        switch (key) {
            case GLFW_KEY_SPACE:
                autoZoomEnabled = !autoZoomEnabled;
                std::printf("[INFO] Auto-Zoom %s\n", autoZoomEnabled ? "ENABLED" : "DISABLED");
                break;
            case GLFW_KEY_P:
                pauseZoom = !pauseZoom;
                std::printf("[INFO] Zoom %s\n", pauseZoom ? "PAUSED" : "RESUMED");
                break;
            default:
                break;
        }
    }
}

void renderCudaFrame(
    hipGraphicsResource_t cudaPboRes,
    int w,
    int h,
    float& zoom,
    float2& offset,
    int maxIter,
    float* d_complexity,
    std::vector<float>& h_complexity,
    int* d_iterations,
    bool autoZoomEnabledParam
) {
    DEBUG_PRINT("Starting frame render");

    static float2 targetOffset = offset;
    static float lastBestGradient = -1.0f;

    uchar4* d_img = nullptr;
    size_t imgSize = 0;
    CHECK_CUDA_STEP(hipGraphicsMapResources(1, &cudaPboRes), "MapResources");
    CHECK_CUDA_STEP(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_img), &imgSize, cudaPboRes), "GetMappedPointer");

    if (Settings::debugGradient) {
        DEBUG_PRINT("Launching debug kernel");
        launch_debugGradient(d_img, w, h);
    } else {
        DEBUG_PRINT("Launching Mandelbrot kernel");
        launch_mandelbrotHybrid(d_img, d_iterations, w, h, zoom, offset, maxIter);

        int totalTiles = static_cast<int>(h_complexity.size());
        CHECK_CUDA_STEP(hipMemset(d_complexity, 0, totalTiles * sizeof(float)), "Memset complexity");

        dim3 blockDim(Settings::TILE_W, Settings::TILE_H);
        dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
        DEBUG_PRINT("Launching complexity kernel Grid(%d, %d) Block(%d, %d)", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

        computeComplexity<<<gridDim, blockDim>>>(d_iterations, w, h, d_complexity);
        CHECK_CUDA_STEP(hipDeviceSynchronize(), "complexity sync");
        CHECK_CUDA_STEP(hipMemcpy(h_complexity.data(), d_complexity, totalTiles * sizeof(float), hipMemcpyDeviceToHost), "Memcpy complexity");

        int tilesX = (w + Settings::TILE_W - 1) / Settings::TILE_W;
        int tilesY = (h + Settings::TILE_H - 1) / Settings::TILE_H;
        int currTileX = static_cast<int>((offset.x * zoom + w * 0.5f) / Settings::TILE_W);
        int currTileY = static_cast<int>((offset.y * zoom + h * 0.5f) / Settings::TILE_H);

        int dynamicRadius = static_cast<int>(std::sqrt(zoom) * Settings::DYNAMIC_RADIUS_SCALE);
        dynamicRadius = std::clamp(dynamicRadius, Settings::DYNAMIC_RADIUS_MIN, Settings::DYNAMIC_RADIUS_MAX);

        DEBUG_PRINT("Search Radius: %d", dynamicRadius);

        float bestGradient = -1.0f;
        int bestIdx = -1;

        for (int dy = -dynamicRadius; dy <= dynamicRadius; ++dy) {
            for (int dx = -dynamicRadius; dx <= dynamicRadius; ++dx) {
                if (dx * dx + dy * dy > dynamicRadius * dynamicRadius) continue;
                int tx = currTileX + dx;
                int ty = currTileY + dy;
                if (tx >= 0 && ty >= 0 && tx < tilesX && ty < tilesY) {
                    int idx = ty * tilesX + tx;
                    float variance = h_complexity[idx];

                    float neighborVariance = 0.0f;
                    int neighborCount = 0;
                    for (int ny = -1; ny <= 1; ++ny) {
                        for (int nx = -1; nx <= 1; ++nx) {
                            if (nx == 0 && ny == 0) continue;
                            int ntx = tx + nx;
                            int nty = ty + ny;
                            if (ntx >= 0 && nty >= 0 && ntx < tilesX && nty < tilesY) {
                                int nidx = nty * tilesX + ntx;
                                neighborVariance += h_complexity[nidx];
                                neighborCount++;
                            }
                        }
                    }
                    if (neighborCount > 0) neighborVariance /= neighborCount;

                    float gradient = fabsf(variance - neighborVariance);
                    float dist2 = dx * dx + dy * dy + 1e-5f;
                    float score = gradient / dist2;

                    if (score > bestGradient) {
                        bestGradient = score;
                        bestIdx = idx;
                    }
                }
            }
        }

        if (bestIdx != -1 && bestGradient > lastBestGradient * 1.02f) {
            lastBestGradient = bestGradient;
            int bx = bestIdx % tilesX;
            int by = bestIdx / tilesX;
            float tx = (bx + 0.5f) * Settings::TILE_W - w * 0.5f;
            float ty = (by + 0.5f) * Settings::TILE_H - h * 0.5f;
            float newTargetX = offset.x + tx / zoom;
            float newTargetY = offset.y + ty / zoom;
            if (std::isfinite(newTargetX) && std::isfinite(newTargetY)) {
                targetOffset = { newTargetX, newTargetY };
                DEBUG_PRINT("New Target Offset: (%.12f, %.12f)", targetOffset.x, targetOffset.y);
            }
        } else {
            DEBUG_PRINT("No better tile found — continuing.");
        }

        offset.x += (targetOffset.x - offset.x) * Settings::LERP_FACTOR;
        offset.y += (targetOffset.y - offset.y) * Settings::LERP_FACTOR;
        DEBUG_PRINT("Smoothed Offset: (%.12f, %.12f)", offset.x, offset.y);
    }

    if (autoZoomEnabledParam && !pauseZoom) {
        if (std::isfinite(zoom) && zoom < 1e18f) {
            zoom += Settings::ZOOM_STEP_FACTOR * zoom;
            DEBUG_PRINT("Zoom Updated: %.12f", zoom);
        }
    }

    CHECK_CUDA_STEP(hipGraphicsUnmapResources(1, &cudaPboRes), "UnmapResources");
    DEBUG_PRINT("Frame render complete");
}

} // namespace CudaInterop
