#include "hip/hip_runtime.h"
// 🍝 Maus-Kommentar: CUDA-Interop für Mandelbrot-Renderer –
// verwaltet PBO-Mapping, Fraktal-Rendering, adaptive Komplexitätsbewertung & Auto-Zoom-Logik.

#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <stdexcept>
#include <algorithm>

#include "settings.hpp"
#include "core_kernel.h"
#include "memory_utils.hpp"
#include "progressive.hpp"
#include "common.hpp"

namespace CudaInterop {

static hipGraphicsResource_t cudaResource = nullptr;  // 🔗 CUDA ↔ OpenGL Interop-Handle
static bool pauseZoom = false;                         // ⏸️ Auto-Zoom pausiert?

// 🧼 Deregistriert PBO aus CUDA
void unregisterPBO() {
    if (cudaResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaResource));
        cudaResource = nullptr;
    }
}

// 🧼 Registriert OpenGL-PBO für CUDA-Zugriff
void registerPBO(GLuint pbo) {
    if (cudaResource) {
        unregisterPBO();
    }
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaResource, pbo, cudaGraphicsMapFlagsWriteDiscard));
}

// 🖼️ Rendert CUDA-Fraktal-Frame & analysiert Komplexität (Auto-Zoom)
void renderCudaFrame(uchar4* pbo,
                     int* d_iterations,
                     float* d_complexity,
                     float* d_stddev,
                     int width,
                     int height,
                     float zoom,
                     float2 offset,
                     int maxIterations,
                     const std::vector<float>& h_complexity,
                     float2& outNewOffset,
                     bool& shouldZoom,
                     int tileSize)
{
    if (!cudaResource) {
        std::fprintf(stderr, "[ERROR] CUDA resource not registered!\n");
        return;
    }

    // 📥 CUDA-Pointer auf OpenGL-PBO holen
    CUDA_CHECK(hipGraphicsMapResources(1, &cudaResource, 0));
    uchar4* devPtr;
    size_t size;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaResource));

    // 🎨 CUDA-Fraktal berechnen
    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations);

    // 📊 Komplexitätsanalyse je Tile
    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    int totalTiles = tilesX * tilesY;

    computeComplexity(d_iterations, d_stddev, width, height, tileSize);  // 🔬 GPU
    CUDA_CHECK(hipMemcpy((void*)h_complexity.data(), d_stddev, totalTiles * sizeof(float), hipMemcpyDeviceToHost));  // ⬇️ Host

    // 🔍 Scoring zur Auswahl des besten Tiles
    float bestScore = -1.0f;
    float2 bestTileOffset = {0.0f, 0.0f};
    shouldZoom = false;

    for (int tileY = 0; tileY < tilesY; ++tileY) {
        for (int tileX = 0; tileX < tilesX; ++tileX) {
            int tileIndex = tileY * tilesX + tileX;
            float gradient = h_complexity[tileIndex];

            if (gradient < Settings::dynamicVarianceThreshold(zoom)) continue;

            float pixelX = (tileX + 0.5f) * tileSize;
            float pixelY = (tileY + 0.5f) * tileSize;

            float2 tileOffset = {
                offset.x + (pixelX - width * 0.5f) / zoom,
                offset.y + (pixelY - height * 0.5f) / zoom
            };

            float tileDist = std::hypot(tileOffset.x - offset.x, tileOffset.y - offset.y);
            float distToCenter = std::hypot(tileOffset.x + 0.75f, tileOffset.y);
            float centralityBoost = 1.0f / (distToCenter + 0.1f);
            float score = gradient * centralityBoost / (tileDist + 1.0f);

            if (score > bestScore) {
                bestScore = score;
                bestTileOffset = tileOffset;
                shouldZoom = true;
            }
        }
    }

    if (shouldZoom) {
        outNewOffset = bestTileOffset;
    }

    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaResource, 0));
}

// 🔍 Getter für Auto-Zoom-Pause
bool getPauseZoom() {
    return pauseZoom;
}

// 📝 Setter für Auto-Zoom-Pause
void setPauseZoom(bool paused) {
    pauseZoom = paused;
}

// ⌨️ Callback für Tastendruck (SPACE/P toggelt Auto-Zoom)
void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action != GLFW_PRESS) return;

    switch (key) {
        case GLFW_KEY_SPACE:
        case GLFW_KEY_P:
            pauseZoom = !pauseZoom;
#if defined(DEBUG) || Settings::debugLogging
            std::printf("[INFO] Taste %s gedrückt – Auto-Zoom %s\n",
                        key == GLFW_KEY_SPACE ? "SPACE" : "P",
                        pauseZoom ? "PAUSIERT" : "AKTIV");
#endif
            break;
        default:
            break;
    }
}

}  // namespace CudaInterop
