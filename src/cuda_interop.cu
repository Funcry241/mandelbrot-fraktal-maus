#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐜 Schwarze Ameise: Klare Parametrisierung, deterministisches Logging, robustes Ressourcenhandling.
// 🦦 Otter: Explizite und einheitliche Übergabe aller Parameter. Fehler- und Kontextlogging überall. (Bezug zu Otter)
// 🦊 Schneefuchs: Keine impliziten Zugriffe, transparente Speicher- und Fehlerprüfung. (Bezug zu Schneefuchs)

#include "pch.hpp"
#include "luchs_log_host.hpp"
#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"
#include "renderer_state.hpp"
// #include "zoom_logic.hpp" // Schneefuchs: Zoom-Entscheidung liegt jetzt in frame_pipeline, hier nicht mehr nötig.
#include "luchs_cuda_log_buffer.hpp"
#include "hermelin_buffer.hpp"
#include "bear_CudaPBOResource.hpp"
#include <cuda_gl_interop.h>
#include <vector>
#include <stdexcept> // Schneefuchs: explicit for std::runtime_error (no reliance on pch)

#ifndef CUDA_ARCH
#include <chrono>
#endif

namespace CudaInterop {

static bear_CudaPBOResource* pboResource = nullptr;
static bool pauseZoom = false;
static bool luchsBabyInitDone = false;

void logCudaDeviceContext(const char* context) {
    int device = -1;
    hipError_t err = hipGetDevice(&device);
    LUCHS_LOG_HOST("[CTX] %s: hipGetDevice() = %d (%s)", context, device, hipGetErrorString(err));
}

void registerPBO(const Hermelin::GLBuffer& pbo) {
    if (pboResource) {
        LUCHS_LOG_HOST("[ERROR] registerPBO: already registered!");
        return;
    }

    GLint boundBefore = 0;
    glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &boundBefore);
    LUCHS_LOG_HOST("[CHECK] GL bind state BEFORE bind: %d", boundBefore);

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo.id());
    GLint boundAfter = 0;
    glGetIntegerv(GL_PIXEL_UNPACK_BUFFER_BINDING, &boundAfter);
    LUCHS_LOG_HOST("[CHECK] GL bind state AFTER bind: %d (expected: %u)", boundAfter, pbo.id());

    if (boundAfter != static_cast<GLint>(pbo.id())) {
        LUCHS_LOG_HOST("[FATAL] GL bind failed - buffer %u was not bound (GL reports: %d)", pbo.id(), boundAfter);
        throw std::runtime_error("glBindBuffer(GL_PIXEL_UNPACK_BUFFER) failed");
    }

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[CU-PBO] Preparing to register PBO ID %u", pbo.id());

    pboResource = new bear_CudaPBOResource(pbo.id());

    logCudaDeviceContext("after registerPBO");
}

void renderCudaFrame(
    Hermelin::CudaDeviceBuffer& d_iterations,
    Hermelin::CudaDeviceBuffer& d_entropy,
    Hermelin::CudaDeviceBuffer& d_contrast,
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    std::vector<float>& h_contrast,
    float2& newOffset,
    bool& shouldZoom,
    int tileSize,
    RendererState& state
) {
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[ENTER] renderCudaFrame(tileSize=%d)", tileSize);

    logCudaDeviceContext("renderCudaFrame ENTER");

    if (!pboResource)
        throw std::runtime_error("[FATAL] CUDA PBO not registered!");

#ifndef CUDA_ARCH
    const auto t0 = std::chrono::high_resolution_clock::now();
#endif

    const int totalPixels = width * height;
    const int tilesX = (width + tileSize - 1) / tileSize;
    const int tilesY = (height + tileSize - 1) / tileSize;
    const int numTiles = tilesX * tilesY;

    // --- Size sanity & allocation guards (Ameise) ---
    const size_t it_bytes       = static_cast<size_t>(totalPixels) * sizeof(int);
    const size_t entropy_bytes  = static_cast<size_t>(numTiles)    * sizeof(float);
    const size_t contrast_bytes = static_cast<size_t>(numTiles)    * sizeof(float);

    const size_t d_it_size       = d_iterations.size();
    const size_t d_entropy_size  = d_entropy.size();
    const size_t d_contrast_size = d_contrast.size();

    if (Settings::debugLogging) {
        LUCHS_LOG_HOST("[SANITY] w=%d h=%d pixels=%d tileSize=%d tiles=%d (%d x %d)",
                       width, height, totalPixels, tileSize, numTiles, tilesX, tilesY);
        LUCHS_LOG_HOST("[SANITY] alloc(it=%zu, entropy=%zu, contrast=%zu) need(it=%zu, entropy=%zu, contrast=%zu)",
                       d_it_size, d_entropy_size, d_contrast_size, it_bytes, entropy_bytes, contrast_bytes);
    }

    bool alloc_ok = true;
    if (d_it_size < it_bytes) {
        LUCHS_LOG_HOST("[FATAL] iterations buffer too small: have=%zu need=%zu", d_it_size, it_bytes);
        alloc_ok = false;
    }
    if (d_entropy_size < entropy_bytes) {
        LUCHS_LOG_HOST("[FATAL] entropy buffer too small: have=%zu need=%zu (tiles=%d)", d_entropy_size, entropy_bytes, numTiles);
        alloc_ok = false;
    }
    if (d_contrast_size < contrast_bytes) {
        LUCHS_LOG_HOST("[FATAL] contrast buffer too small: have=%zu need=%zu (tiles=%d)", d_contrast_size, contrast_bytes, numTiles);
        alloc_ok = false;
    }
    if (!alloc_ok) {
        throw std::runtime_error("CudaInterop::renderCudaFrame: device buffers undersized for current tile layout");
    }

    CUDA_CHECK(hipSetDevice(0));

    CUDA_CHECK(hipMemset(d_iterations.get(), 0, d_iterations.size()));
    LUCHS_LOG_HOST("[MEM] d_iterations memset: %d pixels -> %zu bytes", totalPixels, d_iterations.size());
    CUDA_CHECK(hipMemset(d_entropy.get(),   0, d_entropy.size()));
    CUDA_CHECK(hipMemset(d_contrast.get(),  0, d_contrast.size()));

    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[MAP] Using Baer to map CUDA-GL resource");

    CUDA_CHECK(hipDeviceSynchronize());

    size_t sizeBytes = 0;
    uchar4* devPtr = static_cast<uchar4*>(pboResource->mapAndLog(sizeBytes));

    if (!devPtr) {
        LUCHS_LOG_HOST("[FATAL] Kernel skipped: surface pointer is null");
        return;
    }

    // Sanity-Check auf PBO-Größe
    const size_t expected = static_cast<size_t>(width) * static_cast<size_t>(height) * sizeof(uchar4);
    if (sizeBytes < expected) {
        LUCHS_LOG_HOST("[FATAL] PBO size too small: got=%zu need=%zu (w=%d h=%d)", sizeBytes, expected, width, height);
        pboResource->unmap();
        throw std::runtime_error("PBO byte size mismatch");
    }

    if (!luchsBabyInitDone) {
        LuchsLogger::initCudaLogBuffer(0);
        luchsBabyInitDone = true;
    }

    if (Settings::debugLogging) {
        LUCHS_LOG_HOST(
            "[KERNEL] launch_mandelbrotHybrid(surface=%p, w=%d, h=%d, zoom=%.5f, offset=(%.5f,%.5f), iter=%d, tile=%d)",
            (void*)devPtr, width, height, zoom, offset.x, offset.y, maxIterations, tileSize
        );
    }
    launch_mandelbrotHybrid(
        devPtr,
        static_cast<int*>(d_iterations.get()),
        width, height, zoom, offset, maxIterations, tileSize
    );

    LuchsLogger::flushDeviceLogToHost();

    if (Settings::debugLogging) {
        int dbg_after[3] = {};
        CUDA_CHECK(hipMemcpy(dbg_after, d_iterations.get(), sizeof(dbg_after), hipMemcpyDeviceToHost));
        LUCHS_LOG_HOST("[KERNEL] iters sample: %d %d %d", dbg_after[0], dbg_after[1], dbg_after[2]);
    }

    ::computeCudaEntropyContrast(
        static_cast<const int*>(d_iterations.get()),
        static_cast<float*>(d_entropy.get()),
        static_cast<float*>(d_contrast.get()),
        width, height, tileSize, maxIterations
    );

    h_entropy.resize(numTiles);
    h_contrast.resize(numTiles);

    if (Settings::debugLogging) {
        LUCHS_LOG_HOST("[COPY] entropy D->H: dst=%p src=%p bytes=%zu",
                       (void*)h_entropy.data(), d_entropy.get(), entropy_bytes);
        LUCHS_LOG_HOST("[COPY] contrast D->H: dst=%p src=%p bytes=%zu",
                       (void*)h_contrast.data(), d_contrast.get(), contrast_bytes);
    }

    CUDA_CHECK(hipMemcpy(h_entropy.data(),  d_entropy.get(),   entropy_bytes,  hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_contrast.data(), d_contrast.get(),  contrast_bytes, hipMemcpyDeviceToHost));

    shouldZoom = false;
    newOffset  = offset;

    pboResource->unmap();

    if (Settings::debugLogging) {
        LUCHS_LOG_HOST("[UNMAP] PBO unmapped successfully");
        LUCHS_LOG_HOST("[KERNEL] renderCudaFrame finished");
    }

#ifndef CUDA_ARCH
    const auto t1 = std::chrono::high_resolution_clock::now();
    float totalMs = std::chrono::duration<float,std::milli>(t1 - t0).count();
    if (Settings::debugLogging)
        LUCHS_LOG_HOST("[PERF] renderCudaFrame() = %.2f ms", totalMs);
#endif
}

void setPauseZoom(bool pause) { pauseZoom = pause; }
bool getPauseZoom()           { return pauseZoom; }

bool precheckCudaRuntime() {
    int deviceCount = 0;
    hipError_t e1 = hipFree(0);
    hipError_t e2 = hipGetDeviceCount(&deviceCount);
    LUCHS_LOG_HOST("[CUDA] precheck err1=%d err2=%d count=%d", (int)e1, (int)e2, deviceCount);
    return e1 == hipSuccess && e2 == hipSuccess && deviceCount > 0;
}

bool verifyCudaGetErrorStringSafe() {
    hipError_t dummy = hipErrorInvalidValue;
    const char* msg = hipGetErrorString(dummy);
    if (msg) {
        LUCHS_LOG_HOST("[CHECK] hipGetErrorString(dummy) = \"%s\" -> Aufloesung gefahrlos", msg);
        return true;
    } else {
        LUCHS_LOG_HOST("[FATAL] hipGetErrorString returned null");
        return false;
    }
}

void unregisterPBO() {
    delete pboResource;
    pboResource = nullptr;
}

} // namespace CudaInterop
