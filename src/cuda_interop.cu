#include "hip/hip_runtime.h"
// Datei: src/cuda_interop.cu
// 🐭 Maus-Kommentar: CUDA/OpenGL-Interop, Auto-Zoom via Entropieanalyse, PBO-Mapping, Key-Handling

#include "pch.hpp" // 💡 Muss als erstes stehen!

#include "cuda_interop.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "common.hpp"

namespace CudaInterop {

static hipGraphicsResource_t cudaPboResource;
static bool pauseZoom = false;

void registerPBO(unsigned int pbo) {
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, hipGraphicsRegisterFlagsWriteDiscard));
}

void unregisterPBO() {
    if (cudaPboResource) {
        CUDA_CHECK(hipGraphicsUnregisterResource(cudaPboResource));
        cudaPboResource = nullptr;
    }
}

void renderCudaFrame(    
    int* d_iterations,
    float* d_entropy,    
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    std::vector<float>& h_entropy,
    float2& newOffset,
    bool& shouldZoom,
    int tileSize
) {
    // PBO an CUDA binden
    CUDA_CHECK(hipGraphicsMapResources(1, &cudaPboResource, 0));
    uchar4* devPtr = nullptr;
    size_t size = 0;
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, cudaPboResource));

    // CUDA-Kernel starten
    launch_mandelbrotHybrid(devPtr, d_iterations, width, height, zoom, offset, maxIterations);

    // Entropie berechnen
    computeTileEntropy(d_iterations, d_entropy, width, height, tileSize, maxIterations);

    // Analyse zurück an Host
    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    int numTiles = tilesX * tilesY;
    h_entropy.resize(numTiles);
    CUDA_CHECK(hipMemcpy(h_entropy.data(), d_entropy, numTiles * sizeof(float), hipMemcpyDeviceToHost));

    // Auto-Zoom nur wenn nicht pausiert
    if (!pauseZoom) {
        int bestIndex = -1;
        float bestScore = -1.0f;

        for (int i = 0; i < numTiles; ++i) {
            if (h_entropy[i] > bestScore) {
                bestScore = h_entropy[i];
                bestIndex = i;
            }
        }

        if (bestIndex >= 0) {
            int bx = bestIndex % tilesX;
            int by = bestIndex / tilesX;

            float2 tileCenter = {
                offset.x + (bx + 0.5f) * tileSize / width / zoom * 2.0f - 1.0f,
                offset.y + (by + 0.5f) * tileSize / height / zoom * 2.0f - 1.0f
            };

            newOffset = tileCenter;
            shouldZoom = true;
        } else {
            shouldZoom = false;
        }
    } else {
        shouldZoom = false;
    }

    // PBO unmap
    CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPboResource, 0));
}

void setPauseZoom(bool pause) {
    pauseZoom = pause;
}

bool getPauseZoom() {
    return pauseZoom;
}

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (action == GLFW_PRESS && (key == GLFW_KEY_P || key == GLFW_KEY_SPACE)) {
        pauseZoom = !pauseZoom;
        std::cout << "[Zoom] Auto-Zoom " << (pauseZoom ? "paused" : "resumed") << "\n";
    }
}

} // namespace CudaInterop
