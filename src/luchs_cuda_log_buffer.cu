#include "hip/hip_runtime.h"
// Datei: src/luchs_cuda_log_buffer.cu
// 🐭 Maus-Kommentar: Rückbau auf klare Nicht-Formatierung – robust, simpel, sicher.
// 🦦 Otter: Keine varargs mehr – Klartext-only im device-Code, kompatibel & portabel. (Bezug zu Otter)
// 🦊 Schneefuchs: hipMemcpyFromSymbolAsync statt ungültiger Symbol-Zeiger – deterministisch, korrekt. (Bezug zu Schneefuchs)

#include "luchs_cuda_log_buffer.hpp"
#include "luchs_log_host.hpp"
#include "settings.hpp"
#include <cstring>

namespace LuchsLogger {

    // =========================================================================
    // 🌌 Device-seitiger Logpuffer (1 MB) + Offset (nur hier definiert!)
    // =========================================================================

    __device__ char d_logBuffer[LOG_BUFFER_SIZE];
    __device__ int d_logOffset = 0;

    // Hostseitiger Zwischenspeicher
    static char h_logBuffer[LOG_BUFFER_SIZE] = {0};

    // =========================================================================
    // 🦦 Otter: Initialisierungsstatus und Stream speichern (Luchs Baby)
    // =========================================================================

    static bool s_isInitialized = false;
    static hipStream_t s_logStream = nullptr;

    // =========================================================================
    // 🚀 Device-Logfunktion – kein Format, nur Klartext (LUCHS_LOG_DEVICE)
    // =========================================================================

    __device__ void deviceLog(const char* file, int line, const char* msg) {
        int idx = atomicAdd(&d_logOffset, 0);  // Nur lesen
        if (idx >= LOG_BUFFER_SIZE - 256) return;

        int len = 0;

        const char* filenameOnly = file;
        for (int i = 0; file[i] != '\0'; ++i) {
            if (file[i] == '/' || file[i] == '\\')
                filenameOnly = &file[i + 1];
        }

        for (int i = 0; filenameOnly[i] && len + idx < LOG_BUFFER_SIZE - 2; ++i)
            d_logBuffer[idx + len++] = filenameOnly[i];

        if (len + 6 + idx < LOG_BUFFER_SIZE) {
            d_logBuffer[idx + len++] = ':';
            int l = line, div = 10000;
            bool started = false;
            for (; div > 0; div /= 10) {
                int digit = (l / div) % 10;
                if (digit != 0 || started || div == 1) {
                    d_logBuffer[idx + len++] = '0' + digit;
                    started = true;
                }
            }
            d_logBuffer[idx + len++] = ' ';
            d_logBuffer[idx + len++] = '|';
            d_logBuffer[idx + len++] = ' ';
        }

        for (int i = 0; msg[i] && len + idx < LOG_BUFFER_SIZE - 2; ++i)
            d_logBuffer[idx + len++] = msg[i];

        d_logBuffer[idx + len++] = '\n';
        d_logBuffer[idx + len] = 0;

        atomicAdd(&d_logOffset, len);
    }

    // =========================================================================
    // 🧹 Logbuffer zurücksetzen (via Kernel)
    // =========================================================================

    __global__ void resetLogKernel() {
        d_logOffset = 0;
        if (threadIdx.x == 0 && blockIdx.x == 0)
            d_logBuffer[0] = 0;
    }

    void resetDeviceLog() {
        if (!s_isInitialized) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby ERROR] resetDeviceLog called before init!");
            }
            return;
        }
        resetLogKernel<<<1,1>>>();
        CUDA_CHECK(hipStreamSynchronize(s_logStream));
    }

    // =========================================================================
    // 🦦 Luchs Baby: Initialisierung
    // =========================================================================

    void initCudaLogBuffer(hipStream_t stream) {
        if (s_isInitialized) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby INFO] initCudaLogBuffer already called.");
            }
            return;
        }
        s_logStream = stream;
        resetLogKernel<<<1,1>>>();
        CUDA_CHECK(hipStreamSynchronize(s_logStream));
        s_isInitialized = true;
        if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[LuchsBaby] LogBuffer initialized on stream %p", (void*)stream);
        }
    }

    void freeCudaLogBuffer() {
        if (!s_isInitialized) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby INFO] freeCudaLogBuffer called but not initialized.");
            }
            return;
        }
        s_isInitialized = false;
        s_logStream = nullptr;
        if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[LuchsBaby] LogBuffer freed");
        }
    }

    bool isCudaLogBufferInitialized() {
        return s_isInitialized;
    }

    // =========================================================================
    // 📤 Host: Device-Logbuffer auslesen und über LUCHS_LOG_HOST ausgeben
    // =========================================================================

    void flushDeviceLogToHost(hipStream_t stream) {
        if (!s_isInitialized) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby ERROR] flushDeviceLogToHost called before init!");
            }
            return;
        }

        if (!h_logBuffer) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby ERROR] flushDeviceLogToHost: h_logBuffer null!");
            }
            return;
        }

        if (stream == nullptr) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby] stream==nullptr, using default stream 0");
            }
            stream = 0;
        }

        if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[DEBUG] flushDeviceLogToHost: using hipMemcpyFromSymbolAsync (size=%zu)", LOG_BUFFER_SIZE);
        }

        hipError_t copyErr = hipMemcpyFromSymbolAsync(
            h_logBuffer,
            HIP_SYMBOL(d_logBuffer),
            LOG_BUFFER_SIZE,
            0,
            hipMemcpyDeviceToHost,
            stream
        );
        if (copyErr != hipSuccess) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[CUDA ERROR] hipMemcpyFromSymbolAsync failed: %s", hipGetErrorString(copyErr));
            }
            return;
        }

        hipError_t syncErr = hipStreamSynchronize(stream);
        if (syncErr != hipSuccess) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[CUDA ERROR] hipStreamSynchronize failed: %s", hipGetErrorString(syncErr));
            }
            return;
        }

        if constexpr (Settings::debugLogging) {
            char* ptr = h_logBuffer;
            while (*ptr) {
                char* lineEnd = strchr(ptr, '\n');
                if (!lineEnd) break;
                *lineEnd = 0;
                LUCHS_LOG_HOST("[CUDA] %s", ptr);
                ptr = lineEnd + 1;
            }
        }
    }

} // namespace LuchsLogger
