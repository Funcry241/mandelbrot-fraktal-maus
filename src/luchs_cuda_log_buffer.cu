#include "hip/hip_runtime.h"
///// Otter: Rueckbau auf Klartext-Device-Logging; keine Varargs; robust, simpel, sicher.
///// Schneefuchs: hipMemcpyFromSymbolAsync statt ungueltiger Symbol-Zeiger; deterministisch & korrekt.
///// Maus: ASCII-only; Host-orchestriert; Logs ausschliesslich via LUCHS_LOG_HOST.
///// Datei: src/luchs_cuda_log_buffer.cu

#include "luchs_cuda_log_buffer.hpp"
#include "luchs_log_host.hpp"
#include "settings.hpp"
#include <cstring>

namespace LuchsLogger {

    // =========================================================================
    // Device-seitiger Logpuffer (1 MB) + Offset (nur hier definiert)
    // =========================================================================

    __device__ char d_logBuffer[LOG_BUFFER_SIZE];
    __device__ int d_logOffset = 0;

    // Hostseitiger Zwischenspeicher
    static char h_logBuffer[LOG_BUFFER_SIZE] = {0};

    // =========================================================================
    // Initialisierungsstatus und Stream speichern
    // =========================================================================

    static bool s_isInitialized = false;
    static hipStream_t s_logStream = nullptr;

    // =========================================================================
    // Device-Logfunktion – kein Format, nur Klartext (LUCHS_LOG_DEVICE)
    // =========================================================================

    __device__ void deviceLog(const char* file, int line, const char* msg) {
        int idx = atomicAdd(&d_logOffset, 0);  // nur lesen (aktueller Offset)
        if (idx >= LOG_BUFFER_SIZE - 256) return;

        int len = 0;

        // Dateiname extrahieren (ohne Pfad)
        const char* filenameOnly = file;
        for (int i = 0; file[i] != '\0'; ++i) {
            if (file[i] == '/' || file[i] == '\\')
                filenameOnly = &file[i + 1];
        }

        // "file:line | " schreiben
        for (int i = 0; filenameOnly[i] && len + idx < LOG_BUFFER_SIZE - 2; ++i)
            d_logBuffer[idx + len++] = filenameOnly[i];

        if (len + 6 + idx < LOG_BUFFER_SIZE) {
            d_logBuffer[idx + len++] = ':';
            int l = line, div = 10000;
            bool started = false;
            for (; div > 0; div /= 10) {
                int digit = (l / div) % 10;
                if (digit != 0 || started || div == 1) {
                    d_logBuffer[idx + len++] = '0' + digit;
                    started = true;
                }
            }
            d_logBuffer[idx + len++] = ' ';
            d_logBuffer[idx + len++] = '|';
            d_logBuffer[idx + len++] = ' ';
        }

        // Nachricht
        for (int i = 0; msg[i] && len + idx < LOG_BUFFER_SIZE - 2; ++i)
            d_logBuffer[idx + len++] = msg[i];

        // Zeilenende + Nullterminator
        d_logBuffer[idx + len++] = '\n';
        d_logBuffer[idx + len]   = 0;

        // neuen Offset publizieren
        atomicAdd(&d_logOffset, len);
    }

    // =========================================================================
    // Logbuffer zuruecksetzen (via Kernel)
    // =========================================================================

    __global__ void resetLogKernel() {
        d_logOffset = 0;
        if (threadIdx.x == 0 && blockIdx.x == 0)
            d_logBuffer[0] = 0;
    }

    void resetDeviceLog() {
        if (!s_isInitialized) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby ERROR] resetDeviceLog called before init!");
            }
            return;
        }
        resetLogKernel<<<1,1>>>();
        CUDA_CHECK(hipStreamSynchronize(s_logStream));
    }

    // =========================================================================
    // Initialisierung / Freigabe
    // =========================================================================

    void initCudaLogBuffer(hipStream_t stream) {
        if (s_isInitialized) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby INFO] initCudaLogBuffer already called.");
            }
            return;
        }
        s_logStream = stream;
        resetLogKernel<<<1,1>>>();
        CUDA_CHECK(hipStreamSynchronize(s_logStream));
        s_isInitialized = true;
        if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[LuchsBaby] LogBuffer initialized on stream %p", (void*)stream);
        }
    }

    void freeCudaLogBuffer() {
        if (!s_isInitialized) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby INFO] freeCudaLogBuffer called but not initialized.");
            }
            return;
        }
        s_isInitialized = false;
        s_logStream = nullptr;
        if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[LuchsBaby] LogBuffer freed");
        }
    }

    bool isCudaLogBufferInitialized() {
        return s_isInitialized;
    }

    // =========================================================================
    // Host: Device-Logbuffer auslesen und ueber LUCHS_LOG_HOST ausgeben
    // =========================================================================

    void flushDeviceLogToHost(hipStream_t stream) {
        if (!s_isInitialized) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby ERROR] flushDeviceLogToHost called before init!");
            }
            return;
        }

        // Stream normalisieren
        if (stream == nullptr) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[LuchsBaby] stream==nullptr, using default stream 0");
            }
            stream = 0;
        }

        if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[DEBUG] flushDeviceLogToHost: using hipMemcpyFromSymbolAsync (size=%zu)", LOG_BUFFER_SIZE);
        }

        // Pufferinhalt kopieren (Symbol -> Host)
        hipError_t copyErr = hipMemcpyFromSymbolAsync(
            h_logBuffer,
            HIP_SYMBOL(d_logBuffer),
            LOG_BUFFER_SIZE,
            0,
            hipMemcpyDeviceToHost,
            stream
        );
        if (copyErr != hipSuccess) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[CUDA ERROR] hipMemcpyFromSymbolAsync failed: %s", hipGetErrorString(copyErr));
            }
            return;
        }

        // Warten, bis der Copy fertig ist
        hipError_t syncErr = hipStreamSynchronize(stream);
        if (syncErr != hipSuccess) {
            if constexpr (Settings::debugLogging) {
                LUCHS_LOG_HOST("[CUDA ERROR] hipStreamSynchronize failed: %s", hipGetErrorString(syncErr));
            }
            return;
        }

        // Zeilenweise ausgeben (nur im Debug-Modus, um Log-Spam zu vermeiden)
        if constexpr (Settings::debugLogging) {
            char* ptr = h_logBuffer;
            while (*ptr) {
                char* lineEnd = std::strchr(ptr, '\n');
                if (!lineEnd) break;
                *lineEnd = 0;
                LUCHS_LOG_HOST("[CUDA] %s", ptr);
                ptr = lineEnd + 1;
            }
        }
    }

} // namespace LuchsLogger
