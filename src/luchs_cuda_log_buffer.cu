#include "hip/hip_runtime.h"
// Datei: src/luchs_cuda_log_buffer.cu
// 🐭 Maus-Kommentar: Rückbau auf klare Nicht-Formatierung - robust, simpel, sicher.
// 🦦 Otter: Keine varargs mehr - Klartext-only im __device__-Code, kompatibel & portabel.
// 🦊 Schneefuchs: Präzise Begrenzung, keine Host-Abhängigkeit, garantiert lauffähig.

#include "luchs_cuda_log_buffer.hpp"
#include "luchs_log_host.hpp"
#include <cstring>

namespace LuchsLogger {

    // =========================================================================
    // 🌌 Device-seitiger Logpuffer (1 MB) + Offset (nur hier definiert!)
    // =========================================================================

    __device__ char d_logBuffer[LOG_BUFFER_SIZE];
    __device__ int d_logOffset = 0;

    // Hostseitiger Zwischenspeicher
    char h_logBuffer[LOG_BUFFER_SIZE] = {0};

    // =========================================================================
    // 🚀 Device-Logfunktion - kein Format, nur Klartext (LUCHS_LOG_DEVICE)
    // =========================================================================

    __device__ void deviceLog(const char* file, int line, const char* msg) {
        int idx = atomicAdd(&d_logOffset, 0);  // Nur lesen
        if (idx >= LOG_BUFFER_SIZE - 256) return;

        int len = 0;

        // Dateiname
        for (int i = 0; file[i] && len + idx < LOG_BUFFER_SIZE - 2; ++i)
            d_logBuffer[idx + len++] = file[i];

        // ":" + Zeile + "| "
        if (len + 6 + idx < LOG_BUFFER_SIZE) {
            d_logBuffer[idx + len++] = ':';
            int l = line, div = 10000;
            bool started = false;
            for (; div > 0; div /= 10) {
                int digit = (l / div) % 10;
                if (digit != 0 || started || div == 1) {
                    d_logBuffer[idx + len++] = '0' + digit;
                    started = true;
                }
            }
            d_logBuffer[idx + len++] = ' ';
            d_logBuffer[idx + len++] = '|';
            d_logBuffer[idx + len++] = ' ';
        }

        // Nachricht (klartext, keine Formatierung)
        for (int i = 0; msg[i] && len + idx < LOG_BUFFER_SIZE - 2; ++i)
            d_logBuffer[idx + len++] = msg[i];

        d_logBuffer[idx + len++] = '\n';
        d_logBuffer[idx + len] = 0;

        atomicAdd(&d_logOffset, len);
    }

    // =========================================================================
    // 🧹 Logbuffer zurücksetzen (via Kernel)
    // =========================================================================

    __global__ void resetLogKernel() {
        d_logOffset = 0;
        if (threadIdx.x == 0 && blockIdx.x == 0)
            d_logBuffer[0] = 0;
    }

    void resetDeviceLog() {
        resetLogKernel<<<1,1>>>();
        hipDeviceSynchronize();
    }

    // =========================================================================
    // 📤 Host: Device-Logbuffer auslesen und über LUCHS_LOG_HOST ausgeben
    // =========================================================================

    void flushDeviceLogToHost(hipStream_t stream) {
        hipMemcpyAsync(h_logBuffer, d_logBuffer, LOG_BUFFER_SIZE, hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        char* ptr = h_logBuffer;
        while (*ptr) {
            char* lineEnd = strchr(ptr, '\n');
            if (!lineEnd) break;
            *lineEnd = 0;

            LUCHS_LOG_HOST("[CUDA] %s", ptr);
            ptr = lineEnd + 1;
        }
    }

} // namespace LuchsLogger
