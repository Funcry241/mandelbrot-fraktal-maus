#include "hip/hip_runtime.h"
///// Otter: GPU heatmap metrics — compact kernel, hash-binned entropy, stddev contrast.
///// Schneefuchs: No GL; numeric rc logs; slab device buffers; deterministic behavior.
///// Maus: One kernel launch; immediate stream sync for same-frame use.

#include "pch.hpp"
#include "heatmap_metrics.hpp"
#include "luchs_log_host.hpp"
#include "luchs_cuda_log_buffer.hpp"
#include "settings.hpp"
#include "renderer_state.hpp"

#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>

// -------------------------------- kernel --------------------------------
__global__ void kernel_tile_metrics(const uint16_t* __restrict__ it,
                                    int w, int h,
                                    int tilePx, int tilesX, int tilesY,
                                    float* __restrict__ entropy,
                                    float* __restrict__ contrast)
{
    const int tx = blockIdx.x;
    const int ty = blockIdx.y;
    if (tx >= tilesX || ty >= tilesY) return;

    const int x0 = tx * tilePx;
    const int y0 = ty * tilePx;
    const int x1 = min(w, x0 + tilePx);
    const int y1 = min(h, y0 + tilePx);

    const int tileW = max(0, x1 - x0);
    const int tileH = max(0, y1 - y0);
    const int nPix  = tileW * tileH;
    const int outIx = ty * tilesX + tx;

    if (nPix <= 0) {
        if (entropy)  entropy[outIx]  = 0.0f;
        if (contrast) contrast[outIx] = 0.0f;
        return;
    }

    // Ein-Pass: Summe, Summe^2 und Hash-Histogramm
    double sum = 0.0;
    double sum2 = 0.0;

    constexpr int B = 32;
    int hist[B];
    #pragma unroll
    for (int i = 0; i < B; ++i) hist[i] = 0;

    for (int y = y0; y < y1; ++y) {
        const uint16_t* row = it + (size_t)y * (size_t)w + x0;
        for (int x = 0; x < tileW; ++x) {
            const int v = (int)row[x];
            sum  += (double)v;
            sum2 += (double)v * (double)v;

            const int b = (v ^ (v >> 5)) & (B - 1);
            hist[b] += 1;
        }
    }

    // Kontrast = Standardabweichung
    const double invN = 1.0 / (double)nPix;
    const double mean = sum * invN;
    double var = sum2 * invN - mean * mean;
    if (var < 0.0) var = 0.0;
    if (contrast) contrast[outIx] = (float)sqrt(var);

    // Entropie (hash-binned, 32 Buckets), log2
    float H = 0.0f;
    const float invNf  = 1.0f / (float)nPix;
    constexpr float invLn2 = 1.0f / 0.6931471805599453f;
    for (int i = 0; i < B; ++i) {
        const float p = (float)hist[i] * invNf;
        if (p > 0.0f) H -= p * (logf(p) * invLn2);
    }
    if (entropy) entropy[outIx] = H;
}

// --------------- device slab buffer for entropy+contrast ----------------
static float* s_dMetrics = nullptr;   // layout: [tiles] entropy | [tiles] contrast
static size_t s_tilesCap = 0;

static bool ensureDeviceBuffers(size_t tiles) {
    if (tiles <= s_tilesCap && s_dMetrics) return true;
    if (s_dMetrics) { hipFree(s_dMetrics); s_dMetrics = nullptr; }
    s_tilesCap = 0;

    const size_t bytes = 2 * tiles * sizeof(float);
    const hipError_t rc = hipMalloc((void**)&s_dMetrics, bytes);
    if (rc != hipSuccess) {
        LUCHS_LOG_HOST("[HM][ERR] hipMalloc metrics tiles=%zu rc=%d", tiles, (int)rc);
        LuchsLogger::flushDeviceLogToHost(0);
        return false;
    }
    s_tilesCap = tiles;
    return true;
}

// -------------------------------- public API --------------------------------
namespace HeatmapMetrics {

bool buildGPU(RendererState& state,
              int width, int height, int tilePx,
              hipStream_t stream) noexcept
{
    if (width <= 0 || height <= 0 || tilePx <= 0) return false;
    if (!state.d_iterations.get()) return false;

    const int px = std::max(1, tilePx);
    const int tilesX = (width  + px - 1) / px;
    const int tilesY = (height + px - 1) / px;
    const size_t tiles = (size_t)tilesX * (size_t)tilesY;

    if (!ensureDeviceBuffers(tiles)) return false;

    float* dEntropy  = s_dMetrics;
    float* dContrast = s_dMetrics + tiles;

    dim3 grid((unsigned)tilesX, (unsigned)tilesY, 1);
    dim3 block(1, 1, 1);

    kernel_tile_metrics<<<grid, block, 0, stream>>>(
        static_cast<const uint16_t*>(state.d_iterations.get()),
        width, height, px, tilesX, tilesY,
        dEntropy, dContrast
    );
    hipError_t rc = hipPeekAtLastError();
    if (rc != hipSuccess) {
        LUCHS_LOG_HOST("[HM][ERR] kernel launch rc=%d", (int)rc);
        LuchsLogger::flushDeviceLogToHost(0);
        return false;
    }

    state.h_entropy.resize(tiles);
    state.h_contrast.resize(tiles);

    rc = hipMemcpyAsync(state.h_entropy.data(),  dEntropy,  tiles * sizeof(float),
                         hipMemcpyDeviceToHost, stream);
    if (rc == hipSuccess)
        rc = hipMemcpyAsync(state.h_contrast.data(), dContrast, tiles * sizeof(float),
                             hipMemcpyDeviceToHost, stream);
    if (rc != hipSuccess) {
        LUCHS_LOG_HOST("[HM][ERR] memcpyAsync metrics->host rc=%d", (int)rc);
        LuchsLogger::flushDeviceLogToHost(0);
        return false;
    }

    rc = hipStreamSynchronize(stream);
    if (rc != hipSuccess) {
        LUCHS_LOG_HOST("[HM][ERR] streamSync metrics rc=%d", (int)rc);
        LuchsLogger::flushDeviceLogToHost(0);
        return false;
    }

    if constexpr (Settings::debugLogging) {
        LUCHS_LOG_HOST("[HM][GPU] ok tiles=%dx%d N=%zu tilePx=%d", tilesX, tilesY, tiles, px);
    }
    return true;
}

} // namespace HeatmapMetrics
