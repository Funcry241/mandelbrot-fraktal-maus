#include "hip/hip_runtime.h"
// Datei: src/core_kernel.cu
// Zeilen: 378
// 🐭 Maus-Kommentar: Capybara+Kiwi+MausZoom – Grid/Block-Logging, OOB-Guard, Iter-Check. Alle Parameter explizit, Block/Grid sichtbar im Log, Debuglogging kontrolliert alles!

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_math_constants.h>
#include <cmath>
#include "common.hpp"
#include "core_kernel.h"
#include "settings.hpp"

__device__ __forceinline__ uchar4 elegantColor(float t) {
    if (t < 0.0f) return make_uchar4(0, 0, 0, 255);
    t = fminf(fmaxf(t, 0.0f), 1.0f);
    float intensity = sqrtf(t);
    float r = 0.5f + 0.5f * __sinf(6.2831f * (intensity + 0.0f));
    float g = 0.5f + 0.5f * __sinf(6.2831f * (intensity + 0.33f));
    float b = 0.5f + 0.5f * __sinf(6.2831f * (intensity + 0.66f));
    return make_uchar4(r * 255, g * 255, b * 255, 255);
}

__device__ int mandelbrotIterations(float x0, float y0, int maxIter, float& finalX, float& finalY) {
    float x = 0.0f, y = 0.0f;
    int iter = 0;
    while (x * x + y * y <= 4.0f && iter < maxIter) {
        float xtemp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = xtemp;
        ++iter;
    }
    finalX = x;
    finalY = y;
    return iter;
}

__global__ void mandelbrotKernelAdaptive(uchar4* output, int* iterationsOut,
                                         int width, int height,
                                         float zoom, float2 offset,
                                         int maxIterations,
                                         int tileSize,
                                         const int* tileSupersampling) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x == 0 && y == 0) output[idx] = make_uchar4(255, 0, 0, 255); // Sollte oben links rot werden!


    // MausFix v3: Schreibe *immer* einen gültigen Wert (0) für OOB-Pixel, damit keine -1 entstehen!
    if (x >= width || y >= height) {
        if (idx < width * height && iterationsOut)
            iterationsOut[idx] = 0;
        if (output && idx < width * height)
            output[idx] = make_uchar4(0, 0, 0, 255);
        return;
    }

    int tileX = x / tileSize;
    int tileY = y / tileSize;
    int tilesX = (width + tileSize - 1) / tileSize;
    int tileIndex = tileY * tilesX + tileX;

    int S = tileSupersampling ? tileSupersampling[tileIndex] : 1;
    float totalT = 0.0f;
    int totalIter = 0;

    for (int i = 0; i < S; ++i) {
        for (int j = 0; j < S; ++j) {
            float dx = (i + 0.5f) / S;
            float dy = (j + 0.5f) / S;
            float jx = (x + dx - width * 0.5f) / zoom + offset.x;
            float jy = (y + dy - height * 0.5f) / zoom + offset.y;

            float zx, zy;
            int iter = mandelbrotIterations(jx, jy, maxIterations, zx, zy);
            totalIter += iter;

            float norm = zx * zx + zy * zy;
            float t = (iter + 1.0f - log2f(log2f(fmaxf(norm, 1e-8f)))) / maxIterations;
            t = fminf(fmaxf(t, 0.0f), 1.0f);
            totalT += t;
        }
    }

    float avgT = totalT / (S * S);
    int avgIter = totalIter / (S * S);

    output[idx] = elegantColor(avgT);
    iterationsOut[idx] = max(0, avgIter);
}

// Fix: Setze Entropie für *jede* Kachel, auch wenn Thread 0 keine Pixel verarbeitet (Kiwi)
__global__ void entropyKernel(const int* iterations, float* entropyOut,
                              int width, int height, int tileSize,
                              int maxIter) {
    int tileX = blockIdx.x;
    int tileY = blockIdx.y;
    int startX = tileX * tileSize;
    int startY = tileY * tileSize;

    __shared__ int histo[256];
    for (int i = threadIdx.x; i < 256; i += blockDim.x)
        histo[i] = 0;
    __syncthreads();

    int total = tileSize * tileSize;
    int threads = blockDim.x;
    int tid = threadIdx.x;
    int localCount = 0;

    for (int idx = tid; idx < total; idx += threads) {
        int dx = idx % tileSize;
        int dy = idx / tileSize;
        int x = startX + dx;
        int y = startY + dy;
        if (x >= width || y >= height) continue;
        int iter = iterations[y * width + x];
        iter = max(0, iter); // Fix: Keine negativen Werte
        int bin = min(iter * 256 / (maxIter + 1), 255);
        atomicAdd(&histo[bin], 1);
        localCount++;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        float entropy = 0.0f;
        int usedCount = 0;
        for (int i = 0; i < 256; ++i) {
            float p = (localCount > 0 ? histo[i] / (float)total : 0.0f);
            if (p > 0.0f)
                entropy -= p * log2f(p);
            usedCount += histo[i];
        }
        int tilesX = (width + tileSize - 1) / tileSize;
        int tileIndex = tileY * tilesX + tileX;
        entropyOut[tileIndex] = (usedCount > 0) ? entropy : 0.0f;
    }
}

__global__ void contrastKernel(const float* entropy, float* contrastOut,
                               int tilesX, int tilesY) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= tilesX || ty >= tilesY) return;

    int idx = ty * tilesX + tx;
    float center = entropy[idx];
    float sumDiff = 0.0f;
    int count = 0;
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            if (dx == 0 && dy == 0) continue;
            int nx = tx + dx;
            int ny = ty + dy;
            if (nx < 0 || ny < 0 || nx >= tilesX || ny >= tilesY) continue;
            int nIdx = ny * tilesX + nx;
            sumDiff += fabsf(entropy[nIdx] - center);
            count++;
        }
    }
    contrastOut[idx] = (count > 0) ? sumDiff / count : 0.0f;
}

void computeCudaEntropyContrast(
    const int* d_iterations,
    float* d_entropyOut,
    float* d_contrastOut,
    int width,
    int height,
    int tileSize,
    int maxIter
) {
    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    dim3 gridE(tilesX, tilesY);
    dim3 blockE(128);
    entropyKernel<<<gridE, blockE>>>(d_iterations, d_entropyOut, width, height, tileSize, maxIter);
    hipDeviceSynchronize();

    dim3 gridC((tilesX + 15) / 16, (tilesY + 15) / 16);
    dim3 blockC(16, 16);
    contrastKernel<<<gridC, blockC>>>(d_entropyOut, d_contrastOut, tilesX, tilesY);
    hipDeviceSynchronize();
}

void launch_mandelbrotHybrid(
    uchar4* output,
    int* d_iterations,
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    int tileSize,
    int* d_tileSupersampling,
    int supersampling
) {
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    // --- DEBUG: Log Kernel-Parameter (nur wenn debugLogging aktiv)
    if (Settings::debugLogging) {
        std::printf("[DEBUG] Mandelbrot-Kernel Call: width=%d, height=%d, maxIter=%d, zoom=%.2f, offset=(%.10f, %.10f), tileSize=%d, supersampling=%d, block=(%d,%d), grid=(%d,%d)\n",
            width, height, maxIterations, zoom, offset.x, offset.y, tileSize,
            (d_tileSupersampling ? -42 : 1), // Falls du S sichtbar loggen willst, sonst entfernen
            block.x, block.y, grid.x, grid.y
        );
    }

    mandelbrotKernelAdaptive<<<grid, block>>>(output, d_iterations,
                                              width, height,
                                              zoom, offset,
                                              maxIterations,
                                              tileSize,
                                              d_tileSupersampling);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::fprintf(stderr, "[CUDA ERROR] Kernel launch failed: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    // --- DEBUG: Iterations-Buffer nach Kernel prüfen (Settings::debugLogging)
    if (Settings::debugLogging) {
        int iters_dbg[10] = {0};
        hipMemcpy(iters_dbg, d_iterations, 10 * sizeof(int), hipMemcpyDeviceToHost);
        bool anyInvalid = false;
        for (int i = 0; i < 10; ++i) if (iters_dbg[i] < 0) anyInvalid = true;
        std::printf("[KERNEL] Iterations First10: ");
        for (int i = 0; i < 10; ++i) std::printf("%d ", iters_dbg[i]);
        if (anyInvalid)
            std::printf("[WARN] Found <0 value! Check buffer init or kernel OOB.\n");
        std::puts("");
    }
}
