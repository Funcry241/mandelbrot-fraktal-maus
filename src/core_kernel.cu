#include "hip/hip_runtime.h"
// Datei: src/core_kernel.cu
#include "core_kernel.h"
#include <cstdio>                  // für printf
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#define TILE_W 32
#define TILE_H 32
#define DYNAMIC_THRESHOLD 100.0f

// Farb-Mapping
__device__ __forceinline__ uchar4 colorMap(int iter, int maxIter) {
    if (iter == maxIter) return make_uchar4(0, 0, 0, 255);
    float t = float(iter) / maxIter;
    unsigned char r = unsigned char(9*(1-t)*t*t*t*255);
    unsigned char g = unsigned char(15*(1-t)*(1-t)*t*t*255);
    unsigned char b = unsigned char(8.5*(1-t)*(1-t)*(1-t)*t*255);
    return make_uchar4(r, g, b, 255);
}

// Nested Kernel: Verfeinerung einer Kachel mit doppelter Iterationszahl
__global__ void refineTile(uchar4* img, int width, int height,
                           float zoom, float2 offset,
                           int startX, int startY,
                           int tileW, int tileH,
                           int maxIter)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= tileW || ty >= tileH) return;
    int x = startX + tx, y = startY + ty;
    if (x >= width || y >= height) return;

    float cx = (x - width * 0.5f) / zoom + offset.x;
    float cy = (y - height * 0.5f) / zoom + offset.y;
    float zx = 0.0f, zy = 0.0f;
    int iter = 0;
    while (zx*zx + zy*zy < 4.0f && iter < maxIter) {
        float xt = zx*zx - zy*zy + cx;
        zy = 2.0f*zx*zy + cy;
        zx = xt;
        ++iter;
    }
    img[y * width + x] = colorMap(iter, maxIter);
}

// Haupt-Kernel: Tile-parallel mit adaptiver Rekursion
__global__ void mandelbrotHybrid(uchar4* img,
                                 int width, int height,
                                 float zoom, float2 offset,
                                 int maxIter)
{
    int tileX = blockIdx.x;
    int tileY = blockIdx.y;
    int startX = tileX * TILE_W;
    int startY = tileY * TILE_H;
    int endX = min(startX + TILE_W, width);
    int endY = min(startY + TILE_H, height);

    // Lokale Summe und Zählung
    float sumIter = 0.0f;
    int cntPix = 0;

    // Thread-strided Loop für Basiszeichnung
    for (int y = startY + threadIdx.y; y < endY; y += blockDim.y) {
        for (int x = startX + threadIdx.x; x < endX; x += blockDim.x) {
            float cx = (x - width * 0.5f) / zoom + offset.x;
            float cy = (y - height * 0.5f) / zoom + offset.y;
            float zx = 0.0f, zy = 0.0f;
            int iter = 0;
            while (zx*zx + zy*zy < 4.0f && iter < maxIter) {
                float xt = zx*zx - zy*zy + cx;
                zy = 2.0f*zx*zy + cy;
                zx = xt;
                ++iter;
            }
            sumIter += iter;
            ++cntPix;
            img[y * width + x] = colorMap(iter, maxIter);
        }
    }

    // Nur ein Thread pro Block startet Nested Kernel
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        float avgIter = sumIter / cntPix;
        if (avgIter > DYNAMIC_THRESHOLD) {
            int tileW = endX - startX;
            int tileH = endY - startY;
            dim3 bs(min(tileW, TILE_W), min(tileH, TILE_H));
            dim3 gs((tileW + bs.x - 1) / bs.x,
                    (tileH + bs.y - 1) / bs.y);
            refineTile<<<gs, bs>>>(img, width, height,
                                  zoom, offset,
                                  startX, startY,
                                  tileW, tileH,
                                  maxIter * 2);
            hipGetLastError(); // Fehlercheck
        }
    }
}

extern "C" void launch_mandelbrotHybrid(uchar4* img,
                                        int w, int h,
                                        float zoom, float2 offset,
                                        int maxIter)
{
    dim3 blockDim(TILE_W, TILE_H);
    dim3 gridDim ((w + TILE_W - 1) / TILE_W,
                  (h + TILE_H - 1) / TILE_H);
    mandelbrotHybrid<<<gridDim, blockDim>>>(img, w, h, zoom, offset, maxIter);
    hipGetLastError(); // Fehlercheck
}

// Neuer Complexity-Kernel mit Atomics, zählt nicht‐schwarze Pixel pro Tile
__global__ void computeComplexity(const uchar4* img,
                                  int width, int height,
                                  float* complexity)
{
    int tileX = blockIdx.x;
    int tileY = blockIdx.y;
    int idx   = tileY * gridDim.x + tileX;
    int startX = tileX * TILE_W;
    int startY = tileY * TILE_H;
    int endX   = min(startX + TILE_W, width);
    int endY   = min(startY + TILE_H, height);

    // Unique thread‐index innerhalb der Tile
    int x = startX + threadIdx.x;
    for (int y = startY + threadIdx.y; y < endY; y += blockDim.y) {
        if (x < endX) {
            uchar4 c = img[y * width + x];
            // schwarz?
            if (!(c.x == 0 && c.y == 0 && c.z == 0)) {
                atomicAdd(&complexity[idx], 1.0f);
            }
        }
    }
    // kein __syncthreads() nötig, jeder Thread addiert nur atomic
}
