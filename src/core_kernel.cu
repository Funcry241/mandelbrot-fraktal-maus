#include "hip/hip_runtime.h"
// =============================== core_kernel.cu ===============================
// 2-pass Mandelbrot (Warmup + Sliced Survivor Finish)
// Features:
// - Metric AA via distance estimator (no supersampling).
// - Consolidated post-fx: hue rotation, edge glow, orbit tint.
// - Circular wavy hotspot ripples (screen-space, soft, animated).
// - Angle-crest + orbit-trap gating; budget-aware sliced finish.
// All comments/strings ASCII-only.

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_math_constants.h>
#include <cmath>
#include <chrono>
#include <algorithm>
#include "common.hpp"
#include "luchs_device_format.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "luchs_log_device.hpp"
#include "luchs_log_host.hpp"
#include "otter_color.hpp"

namespace {
// ------------------------------- tuning --------------------------------------
constexpr int   WARP_CHUNK        = 64;
static_assert((WARP_CHUNK % 32) == 0, "WARP_CHUNK must be multiple of 32");

constexpr int   LOOP_CHECK_EVERY  = 16;
constexpr float LOOP_EPS2         = 1e-6f;
constexpr int   LOOP_REQ_HITS     = 1;

constexpr int   P1_LOOP_EVERY     = 48;
constexpr float P1_LOOP_EPS2      = 2e-7f;
constexpr int   P1_LOOP_REQ_HITS  = 1;

constexpr int   WARMUP_IT_BASE    = 1024;
constexpr int   FINISH_SLICE_IT   = 1024;
constexpr int   MAX_SLICES        = 64;

constexpr otter::Palette kPalette = otter::Palette::Glacier;
constexpr float kStripeF   = 3.0f;
constexpr float kStripeAmp = 0.10f;

#if defined(OTTER_FRAMEBUFFER_SRGB) && (OTTER_FRAMEBUFFER_SRGB)
  constexpr float kGamma = 1.0f;
#else
  constexpr float kGamma = 2.2f;
#endif

constexpr double FRAME_BUDGET_FALLBACK_MS = 16.6667;
constexpr double KERNEL_BUDGET_FRAC       = 0.62;
constexpr double MIN_BUDGET_MS            = 6.0;

constexpr float DROP_EMA_ALPHA      = 0.25f;
constexpr float DROP_UPPER_BACKOFF  = 0.30f;
constexpr float DROP_LOWER_ACCEL    = 0.005f;

constexpr float DEEP_EPS2           = 1e-10f;
constexpr int   DEEP_REQ_HITS       = 3;

constexpr float PI2 = 6.28318530717958647692f;

// --------------------------- device constants --------------------------------
struct EffectsParams {
    // AA scale (coverage from DE)
    float aaK;

    // hue rotation
    float huePhase;            // radians

    // edge glow
    float glowAmount;          // 0..1

    // orbit tint
    float orbitK;              // exp(-k * sqrt(minR2))
    float tintMix;             // 0..1
    float3 tint;               // RGB 0..1

    // angle-crest detector
    float crestF;              // frequency for cos(F * angle)
    float crestSharp;          // exponent to sharpen peaks

    // green hotspot pulses (intensity base/gating)
    float hotspotStrength;     // intensity
    float hotspotRate;         // pulses per unit phase
    float hotspotTau;          // decay in phase units
    float3 hotColor;           // RGB 0..1

    // screen-space sprite layout
    float dotCellPx;           // grid cell size in pixels
    float dotRadiusPx;         // reference radius in pixels

    // wave-ring controls (radial ripples)
    float waveLambdaPx;        // wavelength in pixels
    float waveWidthPx;         // gaussian envelope sigma (px)
    float waveSpeed;           // phase speed multiplier
    float waveGain;            // extra gain for the ring crest
};
__device__ __constant__ EffectsParams d_fx;

// adaptive warmup iterations
__device__ __constant__ int d_warmup_it = WARMUP_IT_BASE;

// ------------------------------- helpers -------------------------------------
__device__ __forceinline__ float2 pixelToComplex(
    float px, float py, int w, int h,
    float spanX, float spanY, float2 offset)
{
    return make_float2(
        (px / w - 0.5f) * spanX + offset.x,
        (py / h - 0.5f) * spanY + offset.y
    );
}

__device__ __forceinline__ bool insideMainCardioidOrBulb(float x, float y) {
    float xm = x - 0.25f;
    float q  = xm * xm + y * y;
    if (q * (q + xm) <= 0.25f * y * y) return true;
    float xp = x + 1.0f;
    if (xp * xp + y * y <= 0.0625f) return true;
    return false;
}

__device__ __forceinline__ float clamp01(float x) {
    return fminf(fmaxf(x, 0.0f), 1.0f);
}
__device__ __forceinline__ float smooth01(float x) {
    x = clamp01(x);
    return x * x * (3.0f - 2.0f * x);
}
__device__ __forceinline__ float fractf(float x) {
    return x - floorf(x);
}
__device__ __forceinline__ float hash2(float a, float b) {
    float n = a * 12.9898f + b * 78.233f;
    return fractf(sinf(n) * 43758.5453f);
}

// RGB<->YIQ helpers for hue rotation
__device__ __forceinline__ float3 rgb2yiq(float3 c) {
    float Y = 0.299f * c.x + 0.587f * c.y + 0.114f * c.z;
    float I = 0.596f * c.x - 0.274f * c.y - 0.322f * c.z;
    float Q = 0.211f * c.x - 0.523f * c.y + 0.312f * c.z;
    return make_float3(Y, I, Q);
}
__device__ __forceinline__ float3 yiq2rgb(float3 c) {
    float r = c.x + 0.956f * c.y + 0.621f * c.z;
    float g = c.x - 0.272f * c.y - 0.647f * c.z;
    float b = c.x - 1.106f * c.y + 1.703f * c.z;
    return make_float3(fminf(fmaxf(r, 0.0f), 1.0f),
                       fminf(fmaxf(g, 0.0f), 1.0f),
                       fminf(fmaxf(b, 0.0f), 1.0f));
}
__device__ __forceinline__ float3 hueRotateYIQ(float3 rgb, float phaseRad) {
    if (fabsf(phaseRad) < 1e-6f) return rgb;
    float3 yiq = rgb2yiq(rgb);
    float c = cosf(phaseRad), s = sinf(phaseRad);
    float I = yiq.y * c - yiq.z * s;
    float Q = yiq.y * s + yiq.z * c;
    return yiq2rgb(make_float3(yiq.x, I, Q));
}

// coverage from distance estimator
__device__ __forceinline__ float coverage_from_de(float r2, float dx, float dy, float pixR) {
    float r  = sqrtf(fmaxf(r2, 0.0f));
    float dd = sqrtf(dx*dx + dy*dy);
    dd = (dd > 1e-30f) ? dd : 1e-30f;
    float de = (r > 0.0f) ? (r * logf(r) / dd) : 0.0f;
    float cov = smooth01(de / (d_fx.aaK * fmaxf(pixR, 1e-30f)));
    return cov;
}

// angle-crest weight from escape z = (zx, zy)
__device__ __forceinline__ float crest_weight(float zx, float zy) {
    float ang = atan2f(zy, zx);              // -pi..pi
    float v   = 0.5f * (cosf(d_fx.crestF * ang) + 1.0f); // 0..1
    return powf(clamp01(v), fmaxf(1.0f, d_fx.crestSharp));
}

// wavy circular sprite (radial ripple), deterministic per cell; returns 0..1
__device__ __forceinline__ float dot_mask_wave_screen(
    int xPix, int yPix, int w, int h,
    float cellPx, float radiusPx,
    float lambdaPx, float sigmaPx, float speed, float phase)
{
    // cell indices
    float gx = float(xPix) / fmaxf(cellPx, 1.0f);
    float gy = float(yPix) / fmaxf(cellPx, 1.0f);
    int ix = int(floorf(gx));
    int iy = int(floorf(gy));

    // jittered center inside cell (stable)
    float jx = hash2(float(ix), float(iy)) - 0.5f; // [-0.5, 0.5)
    float jy = hash2(float(iy), float(ix)) - 0.5f;
    float cx = (float(ix) + 0.5f + 0.35f * jx) * cellPx;
    float cy = (float(iy) + 0.5f + 0.35f * jy) * cellPx;

    // pixel center
    float px = float(xPix) + 0.5f;
    float py = float(yPix) + 0.5f;

    float dx = px - cx;
    float dy = py - cy;
    float d  = sqrtf(dx*dx + dy*dy);

    // gaussian envelope around radiusPx (soft window)
    float sig = fmaxf(0.5f, sigmaPx);
    float g   = __expf(-((d - radiusPx)*(d - radiusPx)) / (2.0f * sig * sig));

    // radial sine wave centered at radiusPx
    float k   = PI2 / fmaxf(1.0f, lambdaPx);
    // slight per-cell phase offset for dephasing
    float phiJ = PI2 * hash2(float(ix)*3.1f, float(iy)*7.7f);
    float arg  = k * (d - radiusPx) - speed * phase + phiJ;

    float wave = 0.5f * (cosf(arg) + 1.0f); // 0..1
    // sharpen crest
    wave = powf(wave, 3.0f);

    return clamp01(g * wave);
}

// consolidated post-fx with wavy dots
__device__ __forceinline__ float3 apply_post_fx(
    float3 base, float coverage, float trapMinR2,
    float cr, float ci, float zx, float zy, float phase01,
    int xPix, int yPix, int w, int h)
{
    // edge weight: 0 in interior, 1 near thin edge
    float edgeW = 1.0f - clamp01(coverage);

    // hue rotation
    base = hueRotateYIQ(base, d_fx.huePhase);

    // edge glow toward white
    if (d_fx.glowAmount > 1e-6f) {
        float g = d_fx.glowAmount * edgeW;
        base.x = fminf(1.0f, base.x + g * (1.0f - base.x));
        base.y = fminf(1.0f, base.y + g * (1.0f - base.y));
        base.z = fminf(1.0f, base.z + g * (1.0f - base.z));
    }

    // orbit tint
    float trapW = __expf(-d_fx.orbitK * fmaxf(0.0f, sqrtf(fmaxf(trapMinR2, 0.0f))));
    if (d_fx.tintMix > 1e-6f) {
        float m = d_fx.tintMix * trapW;
        base.x = (1.0f - m) * base.x + m * d_fx.tint.x;
        base.y = (1.0f - m) * base.y + m * d_fx.tint.y;
        base.z = (1.0f - m) * base.z + m * d_fx.tint.z;
    }

    // hotspot gate: either orbit proximity OR angle crest
    float crestW = crest_weight(zx, zy);
    float gate   = fmaxf(trapW, crestW);

    // pulsation envelope (phase01 in 0..1)
    float s   = fractf(phase01 * d_fx.hotspotRate + hash2(cr, ci));
    float env = __expf(-s / fmaxf(1e-3f, d_fx.hotspotTau));

    // wavy circular sprite in screen space
    float dotM = dot_mask_wave_screen(
        xPix, yPix, w, h,
        d_fx.dotCellPx, d_fx.dotRadiusPx,
        d_fx.waveLambdaPx, d_fx.waveWidthPx,
        d_fx.waveSpeed, phase01 * PI2);

    // final hotspot (additive with gain, modulated by gates)
    if (d_fx.hotspotStrength > 1e-6f) {
        float hot = d_fx.hotspotStrength * d_fx.waveGain * gate * edgeW * env * dotM;
        base.x = fminf(1.0f, base.x + hot * d_fx.hotColor.x);
        base.y = fminf(1.0f, base.y + hot * d_fx.hotColor.y);
        base.z = fminf(1.0f, base.z + d_fx.hotColor.z * hot);
    }

    return base;
}

// --------------------------- iteration (chunked) -----------------------------
// Pass 1: warmup with light periodicity + dz/dc + orbit-trap
__device__ __forceinline__ int iterate_warmup_noLoop(
    float cr, float ci, int maxSteps,
    float& x, float& y, bool& interiorFlag,
    float& dx, float& dy, float& trapMinR2)
{
    x = 0.0f; y = 0.0f; dx = 0.0f; dy = 0.0f;
    trapMinR2 = HIP_INF_F;
    int it = 0;
    interiorFlag = false;

    const float trX = -0.745f, trY = 0.186f;

    float px = x, py = y;
    int pc = 0, close_hits = 0;

    unsigned mask = 0xFFFFFFFFu;
#if (__CUDA_ARCH__ >= 700)
    mask = __activemask();
#endif
    bool active = true;

#pragma unroll 1
    for (int k = 0; k < maxSteps; k += WARP_CHUNK) {
#pragma unroll 1
        for (int s = 0; s < WARP_CHUNK; ++s) {
            if (!active) { ++pc; continue; }

            float x2 = x * x, y2 = y * y;
            if (x2 + y2 > 4.0f) { active = false; ++pc; continue; }

            // orbit trap min r^2
            float tx = x - trX, ty = y - trY;
            float r2t = tx*tx + ty*ty;
            trapMinR2 = fminf(trapMinR2, r2t);

            // dz/dc update: d <- 2*z*d + 1
            float ndx = 2.0f * (x * dx - y * dy) + 1.0f;
            float ndy = 2.0f * (x * dy + y * dx);

            float xt = fmaf(x, x, -y2) + cr;
            y = fmaf(2.0f * x, y, ci);
            x = xt;
            dx = ndx; dy = ndy;

            ++it; ++pc;

            if (pc >= P1_LOOP_EVERY) {
                float ex = x - px, ey = y - py;
                float d2 = ex*ex + ey*ey;
                if (d2 < P1_LOOP_EPS2) {
                    if (++close_hits >= P1_LOOP_REQ_HITS) {
                        active = false;
                        interiorFlag = true;
                        it = maxSteps;
                    }
                } else {
                    close_hits = 0;
                }
                px = x; py = y; pc = 0;
            }

            if (it >= maxSteps) { active = false; break; }
        }
        unsigned anyActive = __ballot_sync(mask, active);
        if (anyActive == 0u) break;
    }
    return it;
}

// survivor payload
struct Survivor { float x, y, dx, dy, cr, ci, trapMinR2; int it, idx; };

// pass-2 slice iteration
struct SliceResult { int it; float x, y, dx, dy, trapMinR2; bool escaped; bool interior; float de; };

__device__ __forceinline__ SliceResult iterate_finish_slice(
    float cr, float ci, int start_it, int maxIter,
    float x, float y, float dx, float dy, float trapMinR2,
    int sliceSteps)
{
    if (insideMainCardioidOrBulb(cr, ci)) {
        return { maxIter, x, y, dx, dy, trapMinR2, false, true, 0.0f };
    }

    int it = start_it;
    float px = x, py = y;
    int   pc = 0, close_hits = 0, deep_hits = 0;

    const float trX = -0.745f, trY = 0.186f;

    unsigned mask = 0xFFFFFFFFu;
#if (__CUDA_ARCH__ >= 700)
    mask = __activemask();
#endif
    bool active = true, escaped = false, interior = false;
    float deOut = 0.0f;

#pragma unroll 1
    for (int k = 0; k < sliceSteps; k += WARP_CHUNK) {
#pragma unroll 1
        for (int s = 0; s < WARP_CHUNK; ++s) {
            if (!active) { ++pc; continue; }

            float x2 = x * x, y2 = y * y;
            if (x2 + y2 > 4.0f) {
                float r  = sqrtf(x2 + y2);
                float dd = sqrtf(dx*dx + dy*dy) + 1e-30f;
                deOut    = (r > 0.0f) ? (r * logf(r) / dd) : 0.0f;
                active   = false; escaped = true; ++pc; continue;
            }

            // orbit trap update
            float tx = x - trX, ty = y - trY;
            float r2t = tx*tx + ty*ty;
            trapMinR2 = fminf(trapMinR2, r2t);

            // dz/dc update
            float ndx = 2.0f * (x * dx - y * dy) + 1.0f;
            float ndy = 2.0f * (x * dy + y * dx);
            dx = ndx; dy = ndy;

            float xt = fmaf(x, x, -y2) + cr;
            y = fmaf(2.0f * x, y, ci);
            x = xt;
            ++it; ++pc;

            if (pc >= LOOP_CHECK_EVERY) {
                float ex = x - px, ey = y - py;
                float d2 = ex*ex + ey*ey;
                if (d2 < LOOP_EPS2) {
                    if (++close_hits >= LOOP_REQ_HITS) { active = false; interior = true; it = maxIter; }
                } else {
                    close_hits = 0;
                }
                if (d2 < DEEP_EPS2 && (x2 + y2) < 4.0f) {
                    if (++deep_hits >= DEEP_REQ_HITS) { active = false; interior = true; it = maxIter; }
                } else {
                    deep_hits = 0;
                }
                px = x; py = y; pc = 0;
            }
            if (it >= maxIter) { active = false; break; }
        }
        unsigned anyActive = __ballot_sync(mask, active);
        if (anyActive == 0u) break;
    }
    return { it, x, y, dx, dy, trapMinR2, escaped, interior, deOut };
}

// ----------------------- pass 1 kernel (warmup/compact) ----------------------
__global__ __launch_bounds__(256, 2)
void mandelbrotPass1Warmup(
    uchar4* __restrict__ out, int* __restrict__ iterOut,
    Survivor* __restrict__ surv, int* __restrict__ survCount,
    int w, int h, float zoom, float2 offset,
    int maxIter, float pixR)
{
    const int xPix = blockIdx.x * blockDim.x + threadIdx.x;
    const int yPix = blockIdx.y * blockDim.y + threadIdx.y;
    if (xPix >= w || yPix >= h) return;

    const int idx = yPix * w + xPix;

    const float scale = 1.0f / zoom;
    const float spanX = 3.5f * scale;
    const float spanY = spanX * (float)h / (float)w;
    const float2 c = pixelToComplex(xPix + 0.5f, yPix + 0.5f, w, h, spanX, spanY, offset);

    if (insideMainCardioidOrBulb(c.x, c.y)) {
        out[idx]     = make_uchar4(0,0,0,255);
        iterOut[idx] = maxIter;
        return;
    }

    const int warmupSteps = d_warmup_it;

    float zx=0.0f, zy=0.0f, dx=0.0f, dy=0.0f, trapMinR2=HIP_INF_F;
    bool interior = false;
    int itWarm = iterate_warmup_noLoop(c.x, c.y, warmupSteps, zx, zy, interior, dx, dy, trapMinR2);

    if (interior) {
        out[idx]     = make_uchar4(0,0,0,255);
        iterOut[idx] = maxIter;
        return;
    }

    const float r2 = zx*zx + zy*zy;
    const bool escaped = (itWarm < warmupSteps) && (r2 > 4.0f);

    float phase01 = fractf(d_fx.huePhase * (0.5f / HIP_PI_F) + 0.5f);

    if (escaped) {
        float cov = coverage_from_de(r2, dx, dy, pixR);
        float3 col = otter::shade(itWarm, maxIter, zx, zy, kPalette, kStripeF, kStripeAmp, kGamma);
        col = apply_post_fx(col, cov, trapMinR2, c.x, c.y, zx, zy, phase01, xPix, yPix, w, h);

        out[idx] = make_uchar4(
            (unsigned char)(255.0f * fminf(fmaxf(col.x, 0.0f), 1.0f)),
            (unsigned char)(255.0f * fminf(fmaxf(col.y, 0.0f), 1.0f)),
            (unsigned char)(255.0f * fminf(fmaxf(col.z, 0.0f), 1.0f)),
            255);
        iterOut[idx] = itWarm;
        return;
    }

    // survivors: black now to avoid ghosting
    out[idx] = make_uchar4(0,0,0,255);

    unsigned actMask = 0xFFFFFFFFu;
#if (__CUDA_ARCH__ >= 700)
    actMask = __activemask();
#endif
    const bool isSurvivor = true;
    const unsigned ballot = __ballot_sync(actMask, isSurvivor);
    const int      voteCount = __popc(ballot);

    const int linear_tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int lane       = (linear_tid & 31);
    const unsigned laneMask = ballot & ((1u << lane) - 1u);
    const int      prefix   = __popc(laneMask);

    int base = 0;
    const int leader = __ffs(ballot) - 1;
    if (lane == leader) base = atomicAdd(survCount, voteCount);
    base = __shfl_sync(ballot, base, leader);

    Survivor s; s.x = zx; s.y = zy; s.dx = dx; s.dy = dy;
    s.cr = c.x; s.ci = c.y; s.it = itWarm; s.idx = idx; s.trapMinR2 = trapMinR2;
    surv[base + prefix] = s;
}

// ------------------------- pass 2 kernel (sliced) ----------------------------
__global__ __launch_bounds__(128, 2)
void mandelbrotPass2Slice(
    uchar4* __restrict__ out, int* __restrict__ iterOut,
    const Survivor* __restrict__ survIn, int survInCount,
    Survivor* __restrict__ survOut, int* __restrict__ survOutCount,
    int maxIter, int sliceIt, float pixR,
    int w, int h) // pass w,h to reconstruct pixel coords
{
    const int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t >= survInCount) return;

    Survivor s = survIn[t];

    if (insideMainCardioidOrBulb(s.cr, s.ci)) {
        out[s.idx]     = make_uchar4(0,0,0,255);
        iterOut[s.idx] = maxIter;
        return;
    }

    SliceResult r = iterate_finish_slice(
        s.cr, s.ci, s.it, maxIter, s.x, s.y, s.dx, s.dy, s.trapMinR2, sliceIt);

    float phase01 = fractf(d_fx.huePhase * (0.5f / HIP_PI_F) + 0.5f);

    // reconstruct pixel coords from linear index
    int xPix = s.idx % w;
    int yPix = s.idx / w;

    if (r.escaped) {
        float cov = coverage_from_de(r.x*r.x + r.y*r.y, r.dx, r.dy, pixR);
        float3 col = otter::shade(r.it, maxIter, r.x, r.y, kPalette, kStripeF, kStripeAmp, kGamma);
        col = apply_post_fx(col, cov, r.trapMinR2, s.cr, s.ci, r.x, r.y, phase01, xPix, yPix, w, h);

        out[s.idx] = make_uchar4(
            (unsigned char)(255.0f * fminf(fmaxf(col.x, 0.0f), 1.0f)),
            (unsigned char)(255.0f * fminf(fmaxf(col.y, 0.0f), 1.0f)),
            (unsigned char)(255.0f * fminf(fmaxf(col.z, 0.0f), 1.0f)),
            255);
        iterOut[s.idx] = r.it;
        return;
    }

    if (r.it >= maxIter || r.interior) {
        out[s.idx]     = make_uchar4(0,0,0,255);
        iterOut[s.idx] = r.it;
        return;
    }

    // survivor continues: write black and compact
    out[s.idx] = make_uchar4(0,0,0,255);

    unsigned actMask = 0xFFFFFFFFu;
#if (__CUDA_ARCH__ >= 700)
    actMask = __activemask();
#endif
    const bool isSurvivor = true;
    const unsigned ballot = __ballot_sync(actMask, isSurvivor);
    const int      voteCount = __popc(ballot);

    const int lane = threadIdx.x & 31;
    const unsigned laneMask = ballot & ((1u << lane) - 1u);
    const int prefix = __popc(laneMask);

    int base = 0;
    const int leader = __ffs(ballot) - 1;
    if (lane == leader) base = atomicAdd(survOutCount, voteCount);
    base = __shfl_sync(ballot, base, leader);

    Survivor ns; ns.x = r.x; ns.y = r.y; ns.dx = r.dx; ns.dy = r.dy;
    ns.trapMinR2 = r.trapMinR2; ns.cr = s.cr; ns.ci = s.ci; ns.it = r.it; ns.idx = s.idx;
    survOut[base + prefix] = ns;
}

// ----------------- entropy & contrast (coarse metrics) -----------------------
__global__ void entropyKernel(
    const int* it, float* eOut,
    int w, int h, int tile, int maxIter)
{
    int tX = blockIdx.x, tY = blockIdx.y;
    int startX = tX * tile, startY = tY * tile;

    int tilesX = (w + tile - 1) / tile;
    int tilesY = (h + tile - 1) / tile;
    int tileIndex = tY * tilesX + tX;

    __shared__ int histo[256];
    for (int i = threadIdx.x; i < 256; i += blockDim.x) histo[i] = 0;
    __syncthreads();

    const int total = tile * tile;
    for (int idx = threadIdx.x; idx < total; idx += blockDim.x) {
        int dx = idx % tile, dy = idx / tile;
        int x = startX + dx, y = startY + dy;
        if (x >= w || y >= h) continue;
        int v = it[y * w + x];
        v = max(0, v);
        int bin = min(v * 256 / (maxIter + 1), 255);
        atomicAdd(&histo[bin], 1);
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        float entropy = 0.0f;
        for (int i = 0; i < 256; ++i) {
            float p = float(histo[i]) / float(total);
            if (p > 0.0f) entropy -= p * __log2f(p);
        }
        eOut[tileIndex] = entropy;
    }
}

__global__ void contrastKernel(
    const float* e, float* cOut,
    int tilesX, int tilesY)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= tilesX || ty >= tilesY) return;

    int idx = ty * tilesX + tx;
    float center = e[idx], sum = 0.0f;
    int cnt = 0;

    for (int dy = -1; dy <= 1; ++dy)
        for (int dx = -1; dx <= 1; ++dx) {
            if (dx == 0 && dy == 0) continue;
            int nx = tx + dx, ny = ty + dy;
            if (nx < 0 || ny < 0 || nx >= tilesX || ny >= tilesY) continue;
            int nIdx = ny * tilesX + nx;
            sum += fabsf(e[nIdx] - center);
            ++cnt;
        }

    cOut[idx] = (cnt > 0) ? (sum / cnt) : 0.0f;
}

// ---------------- host wrappers: metrics -------------------------------------
} // namespace

void computeCudaEntropyContrast(
    const int* d_it, float* d_e, float* d_c,
    int w, int h, int tile, int maxIter)
{
    int tilesX = (w + tile - 1) / tile;
    int tilesY = (h + tile - 1) / tile;

    hipMemset(d_e, 0, tilesX * tilesY * sizeof(float));

    if (Settings::performanceLogging || Settings::debugLogging) {
        hipEvent_t evStart, evMid, evEnd;
        hipEventCreate(&evStart); hipEventCreate(&evMid); hipEventCreate(&evEnd);

        hipEventRecord(evStart, 0);
        entropyKernel<<<dim3(tilesX, tilesY), 128>>>(d_it, d_e, w, h, tile, maxIter);
        hipEventRecord(evMid, 0);

        contrastKernel<<<dim3((tilesX + 15) / 16, (tilesY + 15) / 16), dim3(16,16)>>>(d_e, d_c, tilesX, tilesY);
        hipEventRecord(evEnd, 0);
        hipEventSynchronize(evEnd);

        float ms1=0.f, ms2=0.f;
        hipEventElapsedTime(&ms1, evStart, evMid);
        hipEventElapsedTime(&ms2, evMid, evEnd);

        if (Settings::performanceLogging) {
            LUCHS_LOG_HOST("[PERF] entropy=%.3f ms contrast=%.3f ms", ms1, ms2);
        } else {
            LUCHS_LOG_HOST("[TIME] Entropy %.3f ms | Contrast %.3f ms", ms1, ms2);
        }

        hipEventDestroy(evStart); hipEventDestroy(evMid); hipEventDestroy(evEnd);
    } else {
        entropyKernel<<<dim3(tilesX, tilesY), 128>>>(d_it, d_e, w, h, tile, maxIter);
        contrastKernel<<<dim3((tilesX + 15) / 16, (tilesY + 15) / 16), dim3(16,16)>>>(d_e, d_c, tilesX, tilesY);
    }
}

// ---------------- host wrapper: 2-pass sliced renderer -----------------------
namespace {
    using clk = std::chrono::high_resolution_clock;

    struct Survivor; // fwd (already defined above in TU)
    struct DevicePools {
        Survivor* A = nullptr;
        Survivor* B = nullptr;
        int*      cntA = nullptr;
        int*      cntB = nullptr;
        size_t    cap = 0;
    };
    DevicePools g_pools;
    double      g_prevSurvivorsPct = -1.0;

    void ensureSurvivorCapacity(size_t need) {
        if (need <= g_pools.cap) return;
        if (g_pools.A)    hipFree(g_pools.A);
        if (g_pools.B)    hipFree(g_pools.B);
        if (g_pools.cntA) hipFree(g_pools.cntA);
        if (g_pools.cntB) hipFree(g_pools.cntB);
        hipMalloc(&g_pools.A,    need * sizeof(Survivor));
        hipMalloc(&g_pools.B,    need * sizeof(Survivor));
        hipMalloc(&g_pools.cntA, sizeof(int));
        hipMalloc(&g_pools.cntB, sizeof(int));
        g_pools.cap = need;
    }

    int chooseWarmupIt(int maxIter) {
        int warm = WARMUP_IT_BASE;
        if (g_prevSurvivorsPct >= 90.0)      warm = std::min(maxIter, WARMUP_IT_BASE * 3);
        else if (g_prevSurvivorsPct >= 80.0) warm = std::min(maxIter, WARMUP_IT_BASE * 2);
        else if (g_prevSurvivorsPct >= 60.0) warm = std::min(maxIter, (WARMUP_IT_BASE * 3) / 2);
        warm = std::min(warm, std::max(64, maxIter / 3));
        return warm;
    }

    double frameBudgetMsFromSettings() {
        if (Settings::capFramerate && Settings::capTargetFps > 0) {
            return std::max(MIN_BUDGET_MS, 1000.0 / double(Settings::capTargetFps));
        }
        return std::max(MIN_BUDGET_MS, FRAME_BUDGET_FALLBACK_MS);
    }
} // namespace

void launch_mandelbrotHybrid(
    uchar4* out, int* d_it,
    int w, int h, float zoom, float2 offset,
    int maxIter, int /*tile*/)
{
    using namespace std::chrono;

    // default FX params (can be adjusted live before memcpy)
    EffectsParams fx{};
    fx.aaK             = 2.0f;
    fx.huePhase        = float(duration<double>(clk::now().time_since_epoch()).count() * 0.6f); // slow drift
    fx.glowAmount      = 0.22f;  // stronger subtle glow
    fx.orbitK          = 6.0f;
    fx.tintMix         = 0.25f;
    fx.tint            = make_float3(0.62f, 0.40f, 0.95f);   // soft violet
    fx.crestF          = 9.0f;   // number of bright crests around lobes
    fx.crestSharp      = 4.0f;   // sharpness of crests
    fx.hotspotStrength = 0.85f;  // boosted hotspot intensity
    fx.hotspotRate     = 1.2f;
    fx.hotspotTau      = 0.35f;
    fx.hotColor        = make_float3(0.05f, 1.0f, 0.10f);    // green
    fx.dotCellPx       = 10.0f;  // denser grid
    fx.dotRadiusPx     = 5.5f;   // slightly larger
    fx.waveLambdaPx    = 5.0f;   // ring spacing
    fx.waveWidthPx     = 8.0f;   // gaussian sigma around radius
    fx.waveSpeed       = 1.25f;  // ripple motion speed
    fx.waveGain        = 1.8f;   // stronger crest
    hipMemcpyToSymbol(HIP_SYMBOL(d_fx), &fx, sizeof(EffectsParams), 0, hipMemcpyHostToDevice);

    // pixel footprint radius in c-plane (half pixel diagonal)
    const float scale = 1.0f / zoom;
    const float spanX = 3.5f * scale;
    const float spanY = spanX * (float)h / (float)w;
    const float pixR  = 0.5f * sqrtf((spanX / w)*(spanX / w) + (spanY / h)*(spanY / h));

    // kernels grid/block
    dim3 block(32, 8);
    dim3 grid((w + block.x - 1)/block.x, (h + block.y - 1)/block.y);

    ensureSurvivorCapacity(size_t(w) * size_t(h));

    // adaptive warmup
    const int warmupIt = chooseWarmupIt(maxIter);
    hipMemcpyToSymbol(HIP_SYMBOL(d_warmup_it), &warmupIt, sizeof(int), 0, hipMemcpyHostToDevice);

    if (Settings::performanceLogging) {
        LUCHS_LOG_HOST("[PERF] warmup_it=%d prev_survivors=%.2f%%", warmupIt, g_prevSurvivorsPct);
    }

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(mandelbrotPass1Warmup), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(mandelbrotPass2Slice),  hipFuncCachePreferL1);

    // frame budget pacing (host-side only)
    const double frameBudgetMs  = frameBudgetMsFromSettings();
    const double kernelBudgetMs = frameBudgetMs * KERNEL_BUDGET_FRAC;
    const auto   hostStart      = clk::now();

    // pass 1
    hipMemset(g_pools.cntA, 0, sizeof(int));
    mandelbrotPass1Warmup<<<grid, block>>>(out, d_it, g_pools.A, g_pools.cntA, w, h, zoom, offset, maxIter, pixR);

    int h_survA = 0;
    hipMemcpy(&h_survA, g_pools.cntA, sizeof(int), hipMemcpyDeviceToHost); // waits for P1
    const double survPct = (double)h_survA * 100.0 / (double(w) * double(h));
    if (Settings::performanceLogging) {
        LUCHS_LOG_HOST("[PERF] survivors=%d (%.2f%% of %d)", h_survA, survPct, w*h);
    }
    g_prevSurvivorsPct = survPct;

    double p1Ms = duration<double, std::milli>(clk::now() - hostStart).count();
    if (h_survA <= 0) {
        if (Settings::performanceLogging) {
            LUCHS_LOG_HOST("[PERF] mandelbrot (hybrid-sliced): total=%.3f ms", p1Ms);
        } else if (Settings::debugLogging) {
            LUCHS_LOG_HOST("[TIME] Mandelbrot Sliced | Total %.3f ms", p1Ms);
        }
        return;
    }
    if (p1Ms > kernelBudgetMs && Settings::performanceLogging) {
        LUCHS_LOG_HOST("[PERF] budget_hit after P1: p1=%.3f ms budget=%.3f ms -> defer P2", p1Ms, kernelBudgetMs);
    }

    // pass 2 (sliced, budget-aware)
    int threads = 128;
    int slice   = 0;
    int sliceIt = FINISH_SLICE_IT;

    Survivor* curBuf = g_pools.A;
    Survivor* nxtBuf = g_pools.B;
    int*      curCnt = g_pools.cntA;
    int*      nxtCnt = g_pools.cntB;
    int       h_cur  = h_survA;

    float emaDrop = 0.2f;

    while (h_cur > 0 && slice < MAX_SLICES) {
        double elapsedMs = duration<double, std::milli>(clk::now() - hostStart).count();
        if (elapsedMs >= kernelBudgetMs) {
            if (Settings::performanceLogging) {
                LUCHS_LOG_HOST("[PERF] budget_exhausted before slice %d: elapsed=%.3f ms budget=%.3f ms",
                               slice, elapsedMs, kernelBudgetMs);
            }
            break;
        }

        hipMemset(nxtCnt, 0, sizeof(int));
        int blocks = (h_cur + threads - 1) / threads;

        mandelbrotPass2Slice<<<blocks, threads>>>(
            out, d_it, curBuf, h_cur, nxtBuf, nxtCnt, maxIter, sliceIt, pixR, w, h);

        int h_next = 0;
        hipMemcpy(&h_next, nxtCnt, sizeof(int), hipMemcpyDeviceToHost);

        elapsedMs = duration<double, std::milli>(clk::now() - hostStart).count();
        if (Settings::performanceLogging || Settings::debugLogging) {
            LUCHS_LOG_HOST("[PERF] slice=%d steps=%d survivors_in=%d survivors_out=%d elapsed=%.3f ms (budget=%.3f)",
                           slice, sliceIt, h_cur, h_next, elapsedMs, kernelBudgetMs);
        }
        if (elapsedMs >= kernelBudgetMs) {
            if (Settings::performanceLogging) {
                LUCHS_LOG_HOST("[PERF] budget_stop at slice %d", slice);
            }
            std::swap(curBuf, nxtBuf); std::swap(curCnt, nxtCnt); h_cur = h_next; ++slice;
            break;
        }

        const int drop = h_cur - h_next;
        const float dropPct = (h_cur > 0) ? float(drop) / float(h_cur) : 1.0f;
        emaDrop = (1.0f - DROP_EMA_ALPHA) * emaDrop + DROP_EMA_ALPHA * dropPct;

        // adapt slice length
        if (emaDrop < DROP_LOWER_ACCEL && sliceIt < (maxIter / 2)) {
            sliceIt = std::min(sliceIt * 2, maxIter / 2);
            if (Settings::performanceLogging) {
                LUCHS_LOG_HOST("[PERF] adapt_slice_it=%d (emaDrop=%.4f)", sliceIt, emaDrop);
            }
        } else if (emaDrop > DROP_UPPER_BACKOFF && sliceIt > FINISH_SLICE_IT) {
            sliceIt = std::max(sliceIt / 2, FINISH_SLICE_IT);
            if (Settings::performanceLogging) {
                LUCHS_LOG_HOST("[PERF] backoff_slice_it=%d (emaDrop=%.4f)", sliceIt, emaDrop);
            }
        }

        std::swap(curBuf, nxtBuf);
        std::swap(curCnt, nxtCnt);
        h_cur = h_next;
        ++slice;
    }

    if (Settings::performanceLogging || Settings::debugLogging) {
        double totalMs = duration<double, std::milli>(clk::now() - hostStart).count();
        if (Settings::performanceLogging) {
            LUCHS_LOG_HOST("[PERF] mandelbrot (hybrid-sliced): total=%.3f ms (budget=%.3f ms)",
                           totalMs, kernelBudgetMs);
        } else {
            LUCHS_LOG_HOST("[TIME] Mandelbrot Sliced | Total %.3f ms", totalMs);
        }
    }
}
