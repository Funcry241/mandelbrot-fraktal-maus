#include "hip/hip_runtime.h"
#pragma once // ensure header inclusion guard if needed

// Datei: src/core_kernel.cu
// Maus-Kommentar: Hybrid-Mandelbrot-Kernel und Complexity-Zähler für adaptive Zoomsteuerung

#include "core_kernel.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>     // uchar4, float2, …
#include <vector_functions.h> // make_uchar4, make_float2, …

#define DYNAMIC_THRESHOLD 100.0f  // durchschnittliche Iterationen pro Pixel

// Farb-Mapping
__device__ __forceinline__ uchar4 colorMap(int iter, int maxIter) {
    if (iter == maxIter) return make_uchar4(0, 0, 0, 255);
    float t = float(iter) / maxIter;
    unsigned char r = unsigned char(9*(1-t)*t*t*t*255);
    unsigned char g = unsigned char(15*(1-t)*(1-t)*t*t*255);
    unsigned char b = unsigned char(8.5*(1-t)*(1-t)*(1-t)*t*255);
    return make_uchar4(r, g, b, 255);
}

// Complexity-Kernel: zählt nicht-schwarze Pixel pro Tile
__global__ void computeComplexity(const uchar4* img,
                                  int width, int height,
                                  float* complexity)
{
    int tileX = blockIdx.x;
    int tileY = blockIdx.y;
    int tileId = tileY * gridDim.x + tileX;

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = tileX * blockDim.x + tx;
    int y = tileY * blockDim.y + ty;
    if (x >= width || y >= height) return;

    int idx = y * width + x;
    uchar4 pixel = img[idx];
    // Schwarz (0,0,0) wird nicht gezählt
    if (pixel.x || pixel.y || pixel.z) {
        atomicAdd(&complexity[tileId], 1.0f);
    }
}

// Nested Kernel: Verfeinerung einer Kachel mit doppelter Iterationszahl
__global__ void refineTile(uchar4* img, int width, int height,
                           float zoom, float2 offset,
                           int startX, int startY,
                           int tileW, int tileH,
                           int maxIter)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= tileW || ty >= tileH) return;
    int x = startX + tx;
    int y = startY + ty;
    if (x >= width || y >= height) return;

    float cx = (x - width * 0.5f) / zoom + offset.x;
    float cy = (y - height * 0.5f) / zoom + offset.y;
    float zx = 0.0f, zy = 0.0f;
    int iter = 0;
    while (zx*zx + zy*zy < 4.0f && iter < maxIter) {
        float xt = zx*zx - zy*zy + cx;
        zy = 2.0f*zx*zy + cy;
        zx = xt;
        ++iter;
    }
    img[y * width + x] = colorMap(iter, maxIter);
}

// Haupt-Kernel: Tile-parallel mit adaptiver Rekursion
__global__ void mandelbrotHybrid(uchar4* img,
                                 int width, int height,
                                 float zoom, float2 offset,
                                 int maxIter)
{
    int tileX = blockIdx.x;
    int tileY = blockIdx.y;
    int startX = tileX * TILE_W;
    int startY = tileY * TILE_H;
    int endX = min(startX + TILE_W, width);
    int endY = min(startY + TILE_H, height);

    // Lokale Summe und Zählung
    float sumIter = 0.0f;
    int cntPix = 0;

    // Thread-strided Loop für Basiszeichnung
    for (int y = startY + threadIdx.y; y < endY; y += blockDim.y) {
        for (int x = startX + threadIdx.x; x < endX; x += blockDim.x) {
            float cx = (x - width * 0.5f) / zoom + offset.x;
            float cy = (y - height * 0.5f) / zoom + offset.y;
            float zx = 0.0f, zy = 0.0f;
            int iter = 0;
            while (zx*zx + zy*zy < 4.0f && iter < maxIter) {
                float xt = zx*zx - zy*zy + cx;
                zy = 2.0f*zx*zy + cy;
                zx = xt;
                ++iter;
            }
            sumIter += iter;
            ++cntPix;
            img[y * width + x] = colorMap(iter, maxIter);
        }
    }

    // Nur ein Thread pro Block startet Nested Kernel
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        float avgIter = sumIter / cntPix;
        if (avgIter > DYNAMIC_THRESHOLD) {
            int tileW = endX - startX;
            int tileH = endY - startY;
            dim3 bs(min(tileW, TILE_W), min(tileH, TILE_H));
            dim3 gs((tileW + bs.x - 1) / bs.x,
                    (tileH + bs.y - 1) / bs.y);
            refineTile<<<gs, bs>>>(img, width, height,
                                  zoom, offset,
                                  startX, startY,
                                  tileW, tileH,
                                  maxIter * 2);
        }
    }
}

extern "C" void launch_mandelbrotHybrid(uchar4* img,
                                         int w, int h,
                                         float zoom, float2 offset,
                                         int maxIter)
{
    dim3 blockDim(TILE_W, TILE_H);
    dim3 gridDim((w + TILE_W - 1) / TILE_W,
                 (h + TILE_H - 1) / TILE_H);
    mandelbrotHybrid<<<gridDim, blockDim>>>(img, w, h, zoom, offset, maxIter);
}
