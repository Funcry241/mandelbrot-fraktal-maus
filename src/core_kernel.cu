#include "hip/hip_runtime.h"
///// Otter: Minimal GPU E/C kernels; early guards; events only when logging enabled (no C4702).
///// Schneefuchs: Predictable occupancy (__launch_bounds__), ASCII logs, bounds-checked sizes; /WX-safe.
///// Maus: Rendering/shading removed; clear host wrapper API computeCudaEntropyContrast.

#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include "core_kernel.h"
#include "settings.hpp"
#include "luchs_log_host.hpp"

// --------------------------------- helpers -----------------------------------
static __device__ __forceinline__ int clamp_int_0_255(int v) {
    v = (v < 0) ? 0 : v;
    return (v > 255) ? 255 : v;
}

// ------------------------------- entropy kernel ------------------------------
// Schneefuchs: launch-bounds for predictable occupancy with 128 threads.
__global__ __launch_bounds__(128)
void entropyKernel(
    const int* __restrict__ it,
    float* __restrict__ eOut,
    int w, int h, int tile, int maxIter)
{
    const int tX = blockIdx.x;
    const int tY = blockIdx.y;

    const int tilesX = (w + tile - 1) / tile;
    const int tilesY = (h + tile - 1) / tile;
    if (tX >= tilesX || tY >= tilesY) return;

    const int startX = tX * tile;
    const int startY = tY * tile;
    const int tileIndex = tY * tilesX + tX;

    __shared__ int histo[256];
    for (int i = threadIdx.x; i < 256; i += blockDim.x) {
        histo[i] = 0;
    }
    __syncthreads();

    // Otter: precomputed scale avoids a division in the hot path.
    const float scale = 256.0f / float(maxIter + 1);

    const int totalCells = tile * tile;
    for (int idx = threadIdx.x; idx < totalCells; idx += blockDim.x) {
        const int dx = idx % tile;
        const int dy = idx / tile;
        const int x  = startX + dx;
        const int y  = startY + dy;
        if (x >= w || y >= h) continue;

        int v = it[y * w + x];
        v = (v < 0) ? 0 : v;
        int bin = __float2int_rz(float(v) * scale);
        bin = clamp_int_0_255(bin);
        atomicAdd(&histo[bin], 1);
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        // Schneefuchs: compute exact sample count from the histogram.
        int count = 0;
        for (int i = 0; i < 256; ++i) count += histo[i];

        float entropy = 0.0f;
        if (count > 0) {
            const float invCount = 1.0f / float(count);
            for (int i = 0; i < 256; ++i) {
                const float p = float(histo[i]) * invCount;
                if (p > 0.0f) entropy -= p * __log2f(p);
            }
        }
        eOut[tileIndex] = entropy;
    }
}

// ------------------------------- contrast kernel -----------------------------
// Launch with 16x16 (256 thr) blocks; predictable occupancy.
__global__ __launch_bounds__(256)
void contrastKernel(
    const float* __restrict__ e,
    float* __restrict__ cOut,
    int tilesX, int tilesY)
{
    const int tx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= tilesX || ty >= tilesY) return;

    const int idx = ty * tilesX + tx;
    const float center = e[idx];
    float sum = 0.0f;
    int cnt = 0;

    // 8-neighborhood (without center)
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            if (dx == 0 && dy == 0) continue;
            const int nx = tx + dx;
            const int ny = ty + dy;
            if (nx < 0 || ny < 0 || nx >= tilesX || ny >= tilesY) continue;
            const int nIdx = ny * tilesX + nx;
            sum += fabsf(e[nIdx] - center);
            ++cnt;
        }
    }

    cOut[idx] = (cnt > 0) ? (sum / cnt) : 0.0f;
}

// --------------------------- host wrapper: E/C only ---------------------------
void computeCudaEntropyContrast(
    const int* d_it, float* d_e, float* d_c,
    int w, int h, int tile, int maxIter)
{
    // Early guards: robust zeroing for invalid sizes.
    if (w <= 0 || h <= 0 || tile <= 0 || maxIter < 0) {
        const int tilesX0 = (tile > 0) ? (w + tile - 1) / tile : 0;
        const int tilesY0 = (tile > 0) ? (h + tile - 1) / tile : 0;
        const size_t tilesTotal0 = size_t(tilesX0) * size_t(tilesY0);
        if (d_e && tilesTotal0) CUDA_CHECK(hipMemset(d_e, 0, tilesTotal0 * sizeof(float)));
        if (d_c && tilesTotal0) CUDA_CHECK(hipMemset(d_c, 0, tilesTotal0 * sizeof(float)));
        return;
    }

    const int tilesX = (w + tile - 1) / tile;
    const int tilesY = (h + tile - 1) / tile;
    const size_t tilesTotal = size_t(tilesX) * size_t(tilesY);
    if (tilesTotal == 0) {
        return;
    }

    // Clear entropy buffer (contrast reads neighbors; entropy kernel overwrites all valid tiles).
    CUDA_CHECK(hipMemset(d_e, 0, tilesTotal * sizeof(float)));

    // Launch config
    constexpr int EN_BLOCK_THREADS = 128;
    const dim3 enGrid(tilesX, tilesY);
    const dim3 enBlock(EN_BLOCK_THREADS);

    const dim3 ctBlock(16, 16);
    const dim3 ctGrid(
        (tilesX + ctBlock.x - 1) / ctBlock.x,
        (tilesY + ctBlock.y - 1) / ctBlock.y
    );

    // Events only when logging is enabled; avoids overhead and unreachable-code warnings.
    if constexpr (Settings::performanceLogging || Settings::debugLogging) {
        hipEvent_t evStart{}, evMid{}, evEnd{};
        CUDA_CHECK(hipEventCreate(&evStart));
        CUDA_CHECK(hipEventCreate(&evMid));
        CUDA_CHECK(hipEventCreate(&evEnd));

        CUDA_CHECK(hipEventRecord(evStart, 0));
        entropyKernel<<<enGrid, enBlock>>>(d_it, d_e, w, h, tile, maxIter);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipEventRecord(evMid, 0));

        contrastKernel<<<ctGrid, ctBlock>>>(d_e, d_c, tilesX, tilesY);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipEventRecord(evEnd, 0));
        CUDA_CHECK(hipEventSynchronize(evEnd));

        float ms1 = 0.0f, ms2 = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&ms1, evStart, evMid));
        CUDA_CHECK(hipEventElapsedTime(&ms2, evMid, evEnd));

        if constexpr (Settings::performanceLogging) {
            LUCHS_LOG_HOST("[PERF] entropy=%.2f ms contrast=%.2f ms", ms1, ms2);
        } else if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[TIME] entropy=%.2f ms | contrast=%.2f ms", ms1, ms2);
        }

        CUDA_CHECK(hipEventDestroy(evStart));
        CUDA_CHECK(hipEventDestroy(evMid));
        CUDA_CHECK(hipEventDestroy(evEnd));
    } else {
        entropyKernel<<<enGrid, enBlock>>>(d_it, d_e, w, h, tile, maxIter);
        CUDA_CHECK(hipGetLastError());
        contrastKernel<<<ctGrid, ctBlock>>>(d_e, d_c, tilesX, tilesY);
        CUDA_CHECK(hipGetLastError());
    }
}
