#include "hip/hip_runtime.h"
// 🐭 Maus-Kommentar: CUDA-Kernel für Mandelbrot-Fraktal und Entropieanalyse pro Tile
// - launch_mandelbrotHybrid: rendert Fraktalbild + Iterationen
// - computeTileEntropy: misst Entropie je Tile zur Bewertung der Bildstruktur (für Auto-Zoom)

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_math_constants.h>
#include "settings.hpp"
#include "core_kernel.h"

__device__ __forceinline__ uchar4 elegantColor(float t) {
    t = fmodf(t, 1.0f);  // t ∈ [0, 1)
    float s = sinf(3.14159f * t);  // sanfter Verlauf

    float r = 0.8f * s;
    float g = 0.5f + 0.4f * cosf(2.0f * 3.14159f * t);
    float b = 0.6f + 0.3f * sinf(4.0f * 3.14159f * t);

    return make_uchar4(r * 255, g * 255, b * 255, 255);
}

__device__ int mandelbrotIterations(float x0, float y0, int maxIter) {
    float x = 0.0f, y = 0.0f;
    int iter = 0;
    while (x * x + y * y <= 4.0f && iter < maxIter) {
        float xtemp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = xtemp;
        ++iter;
    }
    return iter;
}

__global__ void mandelbrotKernel(uchar4* output, int* iterationsOut,
                                 int width, int height,
                                 float zoom, float2 offset,
                                 int maxIterations) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float jx = (x - width  / 2.0f) / zoom + offset.x;
    float jy = (y - height / 2.0f) / zoom + offset.y;

    int iter = mandelbrotIterations(jx, jy, maxIterations);
    iterationsOut[y * width + x] = iter;

    float t = iter / (float)maxIterations;
    output[y * width + x] = elegantColor(t);
}

__global__ void entropyKernel(const int* iterations, float* entropyOut,
                              int width, int height, int tileSize,
                              int maxIter) {
    int tileX = blockIdx.x;
    int tileY = blockIdx.y;

    int startX = tileX * tileSize;
    int startY = tileY * tileSize;

    __shared__ int histo[256];
    for (int i = threadIdx.x; i < 256; i += blockDim.x)
        histo[i] = 0;
    __syncthreads();

    int localCount = 0;

    int tid = threadIdx.x;
    int threads = blockDim.x;
    int total = tileSize * tileSize;

    for (int idx = tid; idx < total; idx += threads) {
        int dx = idx % tileSize;
        int dy = idx / tileSize;
        int x = startX + dx;
        int y = startY + dy;

        if (x >= width || y >= height) continue;

        int iter = iterations[y * width + x];
        int bin = min(iter * 256 / (maxIter + 1), 255);
        atomicAdd(&histo[bin], 1);
        localCount++;
    }
    __syncthreads();

    __shared__ int totalCount;
    if (threadIdx.x == 0) totalCount = 0;
    __syncthreads();

    atomicAdd(&totalCount, localCount);
    __syncthreads();

    if (threadIdx.x == 0 && totalCount > 0) {
        float entropy = 0.0f;
        for (int i = 0; i < 256; ++i) {
            float p = histo[i] / (float)totalCount;
            if (p > 0.0f)
                entropy -= p * log2f(p);
        }

        int tileIndex = tileY * gridDim.x + tileX;
        entropyOut[tileIndex] = entropy;

#ifdef DEBUG
        if (entropy > 3.0f) {
            printf("[ENTROPY] tile (%d,%d) idx %d -> H=%.4f\n",
                   tileX, tileY, tileIndex, entropy);
        }
#endif
    }
}

extern "C" void launch_mandelbrotHybrid(uchar4* output, int* d_iterations,
                                        int width, int height,
                                        float zoom, float2 offset,
                                        int maxIterations) {
    dim3 block(Settings::TILE_W, Settings::TILE_H);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    mandelbrotKernel<<<grid, block>>>(output, d_iterations,
                                      width, height,
                                      zoom, offset,
                                      maxIterations);
    hipDeviceSynchronize();
}

extern "C" void computeTileEntropy(const int* d_iterations,
                                   float* d_entropyOut,
                                   int width, int height,
                                   int tileSize,
                                   int maxIter) {
    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    dim3 grid(tilesX, tilesY);
    dim3 block(128);  // erhöhte Parallelität pro Tile

    entropyKernel<<<grid, block>>>(d_iterations, d_entropyOut,
                                   width, height,
                                   tileSize, maxIter);
    hipDeviceSynchronize();
}
