#include "hip/hip_runtime.h"
// Datei: src/core_kernel.cu
// Zeilen: 394
// 🐭 Maus-Kommentar: Capybara+Kiwi+MausZoom – Device-Debug für Fraktal-Koord, robust gegen OOB, Kernels immer CUDA-Error-checked. Otter liebt ASCII-Logs und float-Klarheit!

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_math_constants.h>
#include <cmath>
#include "common.hpp"
#include "core_kernel.h"
#include "settings.hpp"

__device__ __forceinline__ uchar4 elegantColor(float t) {
    if (t < 0.0f) return make_uchar4(0, 0, 0, 255);
    t = fminf(fmaxf(t, 0.0f), 1.0f);
    float intensity = sqrtf(t);
    float r = 0.5f + 0.5f * __sinf(6.2831f * (intensity + 0.0f));
    float g = 0.5f + 0.5f * __sinf(6.2831f * (intensity + 0.33f));
    float b = 0.5f + 0.5f * __sinf(6.2831f * (intensity + 0.66f));
    return make_uchar4(r * 255, g * 255, b * 255, 255);
}

__device__ int mandelbrotIterations(float x0, float y0, int maxIter, float& finalX, float& finalY) {
    float x = 0.0f, y = 0.0f;
    int iter = 0;
    while (x * x + y * y <= 4.0f && iter < maxIter) {
        float xtemp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = xtemp;
        ++iter;
    }
    finalX = x;
    finalY = y;
    return iter;
}

__global__ void mandelbrotKernelAdaptive(uchar4* output, int* iterationsOut,
                                         int width, int height,
                                         float zoom, float2 offset,
                                         int maxIterations,
                                         int tileSize,
                                         int* tileSupersampling) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    // Debug-Injection: Schreibe Testwert und Device-Debug für Pixel 0,0 & 1,0/0,1/1,1
#if 1
    if (x < 2 && y < 2) {
        float dx = 0.5f, dy = 0.5f;
        float jx = (x + dx - width * 0.5f) / zoom + offset.x;
        float jy = (y + dy - height * 0.5f) / zoom + offset.y;
        float zx, zy;
        int iter = mandelbrotIterations(jx, jy, maxIterations, zx, zy);
        printf("[DEVICE] Pixel(%d,%d): jx=%.8f jy=%.8f iter=%d\n", x, y, jx, jy, iter);

        if (x == 0 && y == 0) {
            iterationsOut[idx] = 1234; // Testwert, sollte im [KERNEL] Iterations First10: sichtbar sein!
            output[idx] = make_uchar4(255, 0, 0, 255); // Rot für links oben
            // Kein return, restlicher Mandelbrot-Code läuft normal weiter
        }
    }
#endif

    // OOB-Guard: Immer gültigen Wert schreiben
    if (x >= width || y >= height) {
        if (idx < width * height && iterationsOut) iterationsOut[idx] = 0;
        if (output && idx < width * height) output[idx] = make_uchar4(0, 0, 0, 255);
        return;
    }

    int tileX = x / tileSize;
    int tileY = y / tileSize;
    int tilesX = (width + tileSize - 1) / tileSize;
    int tileIndex = tileY * tilesX + tileX;
    int S = (tileSupersampling ? tileSupersampling[tileIndex] : 1);
    float totalT = 0.0f;
    int totalIter = 0;

    for (int i = 0; i < S; ++i) {
        float dx = (i + 0.5f) / S;
        for (int j = 0; j < S; ++j) {
            float dy = (j + 0.5f) / S;
            float jx = (x + dx - width * 0.5f) / zoom + offset.x;
            float jy = (y + dy - height * 0.5f) / zoom + offset.y;
            float zx, zy;
            int iter = mandelbrotIterations(jx, jy, maxIterations, zx, zy);
            totalIter += iter;
            float norm = zx * zx + zy * zy;
            float t = (iter + 1.0f - log2f(log2f(fmaxf(norm, 1e-8f)))) / maxIterations;
            t = fminf(fmaxf(t, 0.0f), 1.0f);
            totalT += t;
        }
    }

    float invS2 = 1.0f / (S * S);
    output[idx] = elegantColor(totalT * invS2);
    iterationsOut[idx] = max(0, (int)(totalIter * invS2));
}

__global__ void entropyKernel(const int* iterations, float* entropyOut,
                              int width, int height, int tileSize,
                              int maxIter) {
    int tileX = blockIdx.x;
    int tileY = blockIdx.y;
    int startX = tileX * tileSize;
    int startY = tileY * tileSize;

    __shared__ int histo[256];
    for (int i = threadIdx.x; i < 256; i += blockDim.x)
        histo[i] = 0;
    __syncthreads();

    int total = tileSize * tileSize;
    int threads = blockDim.x;
    int tid = threadIdx.x;
    int localCount = 0;

    for (int idx = tid; idx < total; idx += threads) {
        int dx = idx % tileSize;
        int dy = idx / tileSize;
        int x = startX + dx;
        int y = startY + dy;
        if (x >= width || y >= height) continue;
        int iter = iterations[y * width + x];
        iter = max(0, iter);
        int bin = min(iter * 256 / (maxIter + 1), 255);
        atomicAdd(&histo[bin], 1);
        localCount++;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        float entropy = 0.0f;
        int usedCount = 0;
        for (int i = 0; i < 256; ++i) {
            float p = (localCount > 0 ? histo[i] / (float)total : 0.0f);
            if (p > 0.0f) entropy -= p * log2f(p);
            usedCount += histo[i];
        }
        int tilesX = (width + tileSize - 1) / tileSize;
        int tileIndex = tileY * tilesX + tileX;
        entropyOut[tileIndex] = (usedCount > 0) ? entropy : 0.0f;
    }
}

__global__ void contrastKernel(const float* entropy, float* contrastOut,
                               int tilesX, int tilesY) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= tilesX || ty >= tilesY) return;
    int idx = ty * tilesX + tx;
    float center = entropy[idx];
    float sumDiff = 0.0f;
    int count = 0;
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            if (dx == 0 && dy == 0) continue;
            int nx = tx + dx;
            int ny = ty + dy;
            if (nx < 0 || ny < 0 || nx >= tilesX || ny >= tilesY) continue;
            int nIdx = ny * tilesX + nx;
            sumDiff += fabsf(entropy[nIdx] - center);
            count++;
        }
    }
    contrastOut[idx] = (count > 0) ? sumDiff / count : 0.0f;
}

void computeCudaEntropyContrast(
    const int* d_iterations,
    float* d_entropyOut,
    float* d_contrastOut,
    int width,
    int height,
    int tileSize,
    int maxIter
) {
    int tilesX = (width + tileSize - 1) / tileSize;
    int tilesY = (height + tileSize - 1) / tileSize;
    dim3 gridE(tilesX, tilesY);
    dim3 blockE(128);
    entropyKernel<<<gridE, blockE>>>(d_iterations, d_entropyOut, width, height, tileSize, maxIter);
    hipError_t errE = hipGetLastError();
    if (errE != hipSuccess) {
        printf("[CUDA ERROR] entropyKernel: %s\n", hipGetErrorString(errE));
    }
    hipDeviceSynchronize();

    dim3 gridC((tilesX + 15) / 16, (tilesY + 15) / 16);
    dim3 blockC(16, 16);
    contrastKernel<<<gridC, blockC>>>(d_entropyOut, d_contrastOut, tilesX, tilesY);
    hipError_t errC = hipGetLastError();
    if (errC != hipSuccess) {
        printf("[CUDA ERROR] contrastKernel: %s\n", hipGetErrorString(errC));
    }
    hipDeviceSynchronize();
}

void launch_mandelbrotHybrid(
    uchar4* output,
    int* d_iterations,
    int width,
    int height,
    float zoom,
    float2 offset,
    int maxIterations,
    int tileSize,
    int* d_tileSupersampling,
    int supersampling
) {
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    if (Settings::debugLogging) {
        std::printf("[DEBUG] Mandelbrot-Kernel Call: width=%d, height=%d, maxIter=%d, zoom=%.2f, offset=(%.10f, %.10f), tileSize=%d, supersampling=%d, block=(%d,%d), grid=(%d,%d)\n",
            width, height, maxIterations, zoom, offset.x, offset.y, tileSize,
            (d_tileSupersampling ? -42 : 1),
            block.x, block.y, grid.x, grid.y
        );
    }

    mandelbrotKernelAdaptive<<<grid, block>>>(output, d_iterations,
                                              width, height,
                                              zoom, offset,
                                              maxIterations,
                                              tileSize,
                                              d_tileSupersampling);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::fprintf(stderr, "[CUDA ERROR] Kernel launch failed: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    if (Settings::debugLogging) {
        int iters_dbg[10] = {0};
        hipMemcpy(iters_dbg, d_iterations, 10 * sizeof(int), hipMemcpyDeviceToHost);
        bool anyInvalid = false;
        for (int i = 0; i < 10; ++i) if (iters_dbg[i] < 0) anyInvalid = true;
        std::printf("[KERNEL] Iterations First10: ");
        for (int i = 0; i < 10; ++i) std::printf("%d ", iters_dbg[i]);
        if (anyInvalid)
            std::printf("[WARN] Found <0 value! Check buffer init or kernel OOB.\n");
        std::puts("");
    }
}
