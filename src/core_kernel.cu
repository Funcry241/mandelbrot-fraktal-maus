#include "hip/hip_runtime.h"
///// Otter: MAUS header normalized; ASCII-only; no functional changes.
///// Schneefuchs: Header format per rules #60–62; path normalized.
///// Maus: Keep this as the only top header block; exact four lines.
///// Datei: src/core_kernel.cu
#include "pch.hpp"

#include <hip/hip_runtime.h>
#include <>
#include <cmath>

#include "core_kernel.h"
#include "settings.hpp"
#include "luchs_log_host.hpp"
#include "common.hpp"

// --------------------------------- helpers -----------------------------------
static __device__ __forceinline__ int clamp_int_0_255(int v) {
    v = (v < 0) ? 0 : v;
    return (v > 255) ? 255 : v;
}

namespace {
    // Keep block size in one place so __launch_bounds__ and host launch stay in sync.
    constexpr int EN_BLOCK_THREADS = 256;   // 256 == EN_BINS → einfache, volle Parallelität
    constexpr int EN_BINS          = 256;
    constexpr int WARP_SIZE        = 32;
    constexpr int EN_WARPS         = EN_BLOCK_THREADS / WARP_SIZE;
    static_assert(EN_WARPS * WARP_SIZE == EN_BLOCK_THREADS, "block size must be multiple of warp size");
}

// ------------------------------- entropy kernel ------------------------------
// Warp-private histograms to reduce atomic contention (EN_WARPS × 256 bins in shared mem).
__global__ __launch_bounds__(EN_BLOCK_THREADS, 2)
void entropyKernel(
    const uint16_t* __restrict__ it,
    float* __restrict__ eOut,
    int w, int h, int tile, int maxIter)
{
    const int tX = blockIdx.x;
    const int tY = blockIdx.y;

    const int tilesX = (w + tile - 1) / tile;
    const int tilesY = (h + tile - 1) / tile;
    if (tX >= tilesX || tY >= tilesY) return;

    const int startX = tX * tile;
    const int startY = tY * tile;
    const int tileIndex = tY * tilesX + tX;

    __shared__ int histo[EN_WARPS][EN_BINS];

    const int lane = threadIdx.x & (WARP_SIZE - 1);
    const int warp = threadIdx.x >> 5;

    // Zero warp-local histograms
    for (int i = lane; i < EN_BINS; i += WARP_SIZE) {
        histo[warp][i] = 0;
    }
    __syncthreads();

    // Precomputed scale avoids division in the hot path.
    const float scale = 256.0f / float(maxIter + 1);

    const int totalCells = tile * tile;
    for (int idx = threadIdx.x; idx < totalCells; idx += blockDim.x) {
        const int dx = idx % tile;
        const int dy = idx / tile;
        const int x  = startX + dx;
        const int y  = startY + dy;
        if (x >= w || y >= h) continue;

        // read-only cached fetch
        int v = (int)__ldg(&it[y * w + x]);
        v = (v < 0) ? 0 : v;
        int bin = __float2int_rz(float(v) * scale);
        bin = clamp_int_0_255(bin);
        atomicAdd(&histo[warp][bin], 1);
    }
    __syncthreads();

    // Reduce warp-local histograms into histo[0][*]  (STRIDED → deckt alle 256 Bins ab)
    for (int b = threadIdx.x; b < EN_BINS; b += blockDim.x) {
        int sum = 0;
        #pragma unroll
        for (int widx = 0; widx < EN_WARPS; ++widx) sum += histo[widx][b];
        histo[0][b] = sum;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        // Exact sample count from the merged histogram
        int count = 0;
        #pragma unroll
        for (int i = 0; i < EN_BINS; ++i) count += histo[0][i];

        float entropy = 0.0f;
        if (count > 0) {
            const float invCount = 1.0f / float(count);
            #pragma unroll
            for (int i = 0; i < EN_BINS; ++i) {
                const float p = float(histo[0][i]) * invCount;
                if (p > 0.0f) entropy -= p * __log2f(p);
            }
        }
        eOut[tileIndex] = entropy;
    }
}

// ------------------------------- contrast kernel -----------------------------
// Launch with 16x16 (256 thr) blocks; predictable occupancy.
__global__ __launch_bounds__(256, 2)
void contrastKernel(
    const float* __restrict__ e,
    float* __restrict__ cOut,
    int tilesX, int tilesY)
{
    const int tx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= tilesX || ty >= tilesY) return;

    const int idx = ty * tilesX + tx;
    const float center = e[idx];
    float sum = 0.0f;
    int cnt = 0;

    // 8-neighborhood (without center)
    #pragma unroll
    for (int dy = -1; dy <= 1; ++dy) {
        #pragma unroll
        for (int dx = -1; dx <= 1; ++dx) {
            if (dx == 0 && dy == 0) continue;
            const int nx = tx + dx;
            const int ny = ty + dy;
            if (nx < 0 || ny < 0 || nx >= tilesX || ny >= tilesY) continue;
            const int nIdx = ny * tilesX + nx;
            sum += fabsf(e[nIdx] - center);
            ++cnt;
        }
    }

    cOut[idx] = (cnt > 0) ? (sum / cnt) : 0.0f;
}

// --------------------------- host wrapper: E/C only ---------------------------
void computeCudaEntropyContrast(
    const uint16_t* d_it, float* d_e, float* d_c,
    int w, int h, int tile, int maxIter)
{
    // Early guards: robust zeroing for invalid sizes.
    if (w <= 0 || h <= 0 || tile <= 0 || maxIter < 0) {
        const int tilesX0 = (tile > 0) ? (w + tile - 1) / tile : 0;
        const int tilesY0 = (tile > 0) ? (h + tile - 1) / tile : 0;
        const size_t tilesTotal0 = size_t(tilesX0) * size_t(tilesY0);
        if (d_e && tilesTotal0) CUDA_CHECK(hipMemset(d_e, 0, tilesTotal0 * sizeof(float)));
        if (d_c && tilesTotal0) CUDA_CHECK(hipMemset(d_c, 0, tilesTotal0 * sizeof(float)));
        return;
    }

    const int tilesX = (w + tile - 1) / tile;
    const int tilesY = (h + tile - 1) / tile;
    const size_t tilesTotal = size_t(tilesX) * size_t(tilesY);
    if (tilesTotal == 0) {
        return;
    }

    // Clear entropy buffer (contrast reads neighbors; entropy kernel overwrites all valid tiles).
    CUDA_CHECK(hipMemset(d_e, 0, tilesTotal * sizeof(float)));

    // Launch config
    const dim3 enGrid(tilesX, tilesY);
    const dim3 enBlock(EN_BLOCK_THREADS);

    const dim3 ctBlock(16, 16);
    const dim3 ctGrid(
        (tilesX + ctBlock.x - 1) / ctBlock.x,
        (tilesY + ctBlock.y - 1) / ctBlock.y
    );

    // Events only when logging is enabled; avoids overhead and unreachable-code warnings.
    if constexpr (Settings::performanceLogging || Settings::debugLogging) {
        hipEvent_t evStart{}, evMid{}, evEnd{};
        CUDA_CHECK(hipEventCreate(&evStart));
        CUDA_CHECK(hipEventCreate(&evMid));
        CUDA_CHECK(hipEventCreate(&evEnd));

        CUDA_CHECK(hipEventRecord(evStart, 0));
        entropyKernel<<<enGrid, enBlock>>>(d_it, d_e, w, h, tile, maxIter);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipEventRecord(evMid, 0));

        contrastKernel<<<ctGrid, ctBlock>>>(d_e, d_c, tilesX, tilesY);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipEventRecord(evEnd, 0));
        CUDA_CHECK(hipEventSynchronize(evEnd));

        float ms1 = 0.0f, ms2 = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&ms1, evStart, evMid));
        CUDA_CHECK(hipEventElapsedTime(&ms2, evMid, evEnd));

        if constexpr (Settings::performanceLogging) {
            LUCHS_LOG_HOST("[PERF] entropy=%.2f ms contrast=%.2f ms", ms1, ms2);
        } else if constexpr (Settings::debugLogging) {
            LUCHS_LOG_HOST("[TIME] entropy=%.2f ms | contrast=%.2f ms", ms1, ms2);
        }

        CUDA_CHECK(hipEventDestroy(evStart));
        CUDA_CHECK(hipEventDestroy(evMid));
        CUDA_CHECK(hipEventDestroy(evEnd));
    } else {
        entropyKernel<<<enGrid, enBlock>>>(d_it, d_e, w, h, tile, maxIter);
        CUDA_CHECK(hipGetLastError());
        contrastKernel<<<ctGrid, ctBlock>>>(d_e, d_c, tilesX, tilesY);
        CUDA_CHECK(hipGetLastError());
    }
}
