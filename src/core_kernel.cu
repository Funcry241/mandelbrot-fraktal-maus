#include "hip/hip_runtime.h"
// MAUS:
// 🐭 Maus: Feature „Schwarze Schnauze“ – Early-Out für Innenpunkte (Cardioid/Bulb).
// 🦦 Otter: Spart Iterationen in schwarzen Bereichen, ohne Bildänderung. (Bezug zu Otter)
// 🦊 Schneefuchs: Mathematisch exakt; nur Workload-Reduktion, Logs ASCII. (Bezug zu Schneefuchs)
// 🐑 Schneefuchs: Warp-synchrones Escape & FMA – weniger Divergenz, weniger Instruktionen. (Bezug zu Schneefuchs)

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_math_constants.h>
#include <cmath>
#include <chrono> // Otter: Host-Timing
#include "common.hpp"
#include "core_kernel.h"
#include "settings.hpp"
#include "luchs_log_device.hpp"
#include "luchs_log_host.hpp"

#ifdef __CUDA_ARCH__
__device__ int sprintf(char* str, const char* format, ...);
#endif

// --- Helpers ----------------------------------------------------------------

__device__ __forceinline__ float fract(float x) {
    return x - floorf(x);
}

__device__ __forceinline__ uchar4 elegantColor(float t) {
    t = sqrtf(fminf(fmaxf(t, 0.0f), 1.0f));
    float rf = 0.5f + 0.5f * __sinf(6.2831f * (t + 0.0f));
    float gf = 0.5f + 0.5f * __sinf(6.2831f * (t + 0.33f));
    float bf = 0.5f + 0.5f * __sinf(6.2831f * (t + 0.66f));
    unsigned char r = static_cast<unsigned char>(rf * 255.0f);
    unsigned char g = static_cast<unsigned char>(gf * 255.0f);
    unsigned char b = static_cast<unsigned char>(bf * 255.0f);
    return make_uchar4(r, g, b, 255);
}

__device__ __forceinline__ float2 pixelToComplex(
    float px, float py, int w, int h,
    float spanX, float spanY, float2 offset)
{
    return make_float2(
        (px / w - 0.5f) * spanX + offset.x,
        (py / h - 0.5f) * spanY + offset.y
    );
}

// --- Mandelbrot (baseline) ---------------------------------------------------

__device__ __forceinline__ int mandelbrotIterations_scalar(
    float x0, float y0, int maxIter,
    float& fx, float& fy)
{
    float x = 0.0f, y = 0.0f;
    int i = 0;
#pragma unroll 1
    while (x * x + y * y <= 4.0f && i < maxIter) {
        float xx = x * x;
        float yy = y * y;
        float xy = x * y;
        float xt = xx - yy + x0;
        y = 2.0f * xy + y0;
        x = xt;
        ++i;
    }
    fx = x;
    fy = y;
    return i;
}

// 🐑 Schneefuchs: Warp-synchronisierte Iteration mit Ballot – reduziert Divergenz.
__device__ __forceinline__ int mandelbrotIterations_warp(
    float cr, float ci, int maxIter, float& xr, float& xi)
{
    float x = 0.0f, y = 0.0f;
    int it = 0;

    unsigned mask = 0xFFFFFFFFu;
#if (__CUDA_ARCH__ >= 700)
    mask = __activemask();
#endif

    bool active = true;

#pragma unroll 1
    for (int k = 0; k < maxIter; ++k) {
        // Check escape radius BEFORE heavy math for escaped threads.
        float x2 = x * x;
        float y2 = y * y;
        if (active && (x2 + y2 <= 4.0f)) {
            // z = z^2 + c  with FMA to reduce ops and improve precision.
            // xt = x*x - y*y + cr  == fmaf(x, x, -y2) + cr
            float xt = fmaf(x, x, -y2) + cr;                  // x^2 - y^2 + cr
            y = fmaf(2.0f * x, y, ci);                        // 2*x*y + ci
            x = xt;
            ++it;
        } else {
            active = false;
        }

        // Warp votes: break when all threads are inactive (escaped or finished).
        unsigned anyActive = __ballot_sync(mask, active);
        if (anyActive == 0u) break;
    }

    xr = x; xi = y;
    return it;
}

// --- Farbe / Mapping ---------------------------------------------------------

__device__ __forceinline__ float3 hsvToRgb(float h, float s, float v) {
    float r, g, b;
    int i = int(h * 6.0f);
    float f = h * 6.0f - i;
    float p = v * (1.0f - s);
    float q = v * (1.0f - f * s);
    float t = v * (1.0f - (1.0f - f) * s);
    switch (i % 6) {
        case 0: r = v, g = t, b = p; break;
        case 1: r = q, g = v, b = p; break;
        case 2: r = p, g = v, b = t; break;
        case 3: r = p, g = q, b = v; break;
        case 4: r = t, g = p, b = v; break;
        default: r = v, g = p, b = q; break;
    }
    return make_float3(r, g, b);
}

__device__ float pseudoRandomWarze(float x, float y) {
    float r = sqrtf(x * x + y * y);
    float angle = atan2f(y, x);
    return 0.5f + 0.5f * __sinf(r * 6.0f + angle * 4.0f);
}

// Continuous Escape-Time (CEC) + Stripe
__device__ __forceinline__
void computeCEC(float zx, float zy, int it, int maxIt, float& nu, float& stripe)
{
    float norm = zx * zx + zy * zy;
    if (it >= maxIt) {
        nu = 1.0f;
        stripe = 0.0f;
        return;
    }
    // Schneefuchs: __log2f ist schnelle Approx.; fmaxf schützt Bereich.
    float mu = (float)it + 1.0f - __log2f(__log2f(fmaxf(norm, 1.000001f)));
    nu = fminf(fmaxf(mu / (float)maxIt, 0.0f), 1.0f);
    float frac = fract(mu);
    stripe = powf(0.5f + 0.5f * __sinf(6.2831853f * frac), 0.75f);
}

__device__ __forceinline__
float3 colorFractalDetailed(float2 c, float zx, float zy, int it, int maxIt)
{
    if (it >= maxIt) {
        return make_float3(0.0f, 0.0f, 0.0f);
    }
    float nu, stripe;
    computeCEC(zx, zy, it, maxIt, nu, stripe);

    float angle = atan2f(c.y, c.x);
    // 0.15915494f = 1/(2*pi)
    float hue   = fract(nu * 0.25f + angle * 0.08f * 0.15915494f);
    float val   = 0.3f + 0.7f * stripe;
    float sat   = 0.9f;
    return hsvToRgb(hue, sat, val);
}

// --- „Schwarze Schnauze“: Innenraum-Shortcut --------------------------------
// Otter: Early-Out für Punkte sicher in der Menge – spart komplette Iteration.
// Schneefuchs: Zwei exakte Tests (Hauptcardioide, period-2 Bulb).
__device__ __forceinline__ bool insideMainCardioidOrBulb(float x, float y) {
    // Hauptcardioide
    float xm = x - 0.25f;
    float q  = xm * xm + y * y;
    if (q * (q + xm) <= 0.25f * y * y) return true;

    // period-2 Bulb um (-1,0) mit r=0.25
    float xp = x + 1.0f;
    if (xp * xp + y * y <= 0.0625f) return true;

    return false;
}

// --- Kernel ------------------------------------------------------------------

// 🐑 Schneefuchs: __restrict__-Aliase helfen dem Compiler ohne API-Änderung.
__global__ void mandelbrotKernel(
    uchar4* out, int* iterOut,
    int w, int h, float zoom, float2 offset, int maxIter)
{
    const bool doLog = Settings::debugLogging;

    uchar4* __restrict__ outR   = out;
    int*    __restrict__ iterR  = iterOut;

    const int x   = blockIdx.x * blockDim.x + threadIdx.x;
    const int y   = blockIdx.y * blockDim.y + threadIdx.y;
    const int idx = y * w + x;

    if (x >= w || y >= h || idx >= w * h) return;
    if (!outR || !iterR || w <= 0 || h <= 0) return;

    const float scale = 1.0f / zoom;
    const float spanX = 3.5f * scale;
    const float spanY = spanX * (float)h / (float)w;

    const float2 c = pixelToComplex(x + 0.5f, y + 0.5f, w, h, spanX, spanY, offset);

    // 🐽 Schwarze Schnauze: Early-Out für Innenpunkte (schwarz, it=maxIter)
    if (insideMainCardioidOrBulb(c.x, c.y)) {
        outR[idx]   = make_uchar4(0, 0, 0, 255);
        iterR[idx]  = maxIter;
        if (doLog && threadIdx.x == 0 && threadIdx.y == 0) {
            char msg[96]; int n = 0;
            n += sprintf(msg + n, "[NOSE] early_inside x=%d y=%d", x, y);
            LUCHS_LOG_DEVICE(msg);
        }
        return;
    }

    float zx, zy;
    // 🐑 Schneefuchs: Warp-synchronisierte Iterationen (weniger Divergenz).
    int it = mandelbrotIterations_warp(c.x, c.y, maxIter, zx, zy);
    // Fallback (optional, deaktiviert): // int it = mandelbrotIterations_scalar(c.x, c.y, maxIter, zx, zy);

    const float3 rgb = colorFractalDetailed(c, zx, zy, it, maxIter);
    outR[idx] = make_uchar4(
        (unsigned char)(rgb.x * 255.0f),
        (unsigned char)(rgb.y * 255.0f),
        (unsigned char)(rgb.z * 255.0f),
        255
    );
    iterR[idx] = it;

    if (doLog && threadIdx.x == 0 && threadIdx.y == 0) {
        float norm = zx * zx + zy * zy;
        float t = (it < maxIter)
                    ? (((float)it + 1.0f - __log2f(__log2f(fmaxf(norm, 1.000001f)))) / (float)maxIter)
                    : 1.0f;
        float tClamped = fminf(fmaxf(t, 0.0f), 1.0f);
        char msg[192]; int n = 0;
        n += sprintf(msg + n, "[KERNEL] x=%d y=%d it=%d tClamped=%.4f norm=%.4f ", x, y, it, tClamped, norm);
        LUCHS_LOG_DEVICE(msg);
    }
}

// ---- ENTROPY-KERNEL ----
__global__ void entropyKernel(
    const int* it, float* eOut,
    int w, int h, int tile, int maxIter)
{
    const bool doLog = Settings::debugLogging;
    int tX = blockIdx.x, tY = blockIdx.y;
    int startX = tX * tile, startY = tY * tile;

    int tilesX = (w + tile - 1) / tile;
    int tilesY = (h + tile - 1) / tile;
    int tileIndex = tY * tilesX + tX;

    if (doLog && threadIdx.x == 0) {
        char msg[256];
        sprintf(msg,
            "[ENTROPY-DEBUG] tX=%d tY=%d tile=%d w=%d h=%d tilesX=%d tilesY=%d tileIndex=%d",
            tX, tY, tile, w, h, tilesX, tilesY, tileIndex);
        LUCHS_LOG_DEVICE(msg);
    }

    __shared__ int histo[256];
    for (int i = threadIdx.x; i < 256; i += blockDim.x) histo[i] = 0;
    __syncthreads();

    const int total = tile * tile;
    for (int idx = threadIdx.x; idx < total; idx += blockDim.x) {
        int dx = idx % tile, dy = idx / tile;
        int x = startX + dx, y = startY + dy;
        if (x >= w || y >= h) continue;
        int v = it[y * w + x];
        v = max(0, v);
        int bin = min(v * 256 / (maxIter + 1), 255);
        atomicAdd(&histo[bin], 1);
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        float entropy = 0.0f;
        for (int i = 0; i < 256; ++i) {
            float p = float(histo[i]) / float(total);
            if (p > 0.0f) entropy -= p * __log2f(p);
        }
        eOut[tileIndex] = entropy;

        if (doLog) {
            char msg[128];
            sprintf(msg, "[ENTROPY] tile=(%d,%d) entropy=%.5f", tX, tY, entropy);
            LUCHS_LOG_DEVICE(msg);
        }
    }
}

// ---- CONTRAST-KERNEL ----
__global__ void contrastKernel(
    const float* e, float* cOut,
    int tilesX, int tilesY)
{
    const bool doLog = Settings::debugLogging;
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= tilesX || ty >= tilesY) return;

    int idx = ty * tilesX + tx;
    float center = e[idx], sum = 0.0f;
    int cnt = 0;

    for (int dy = -1; dy <= 1; ++dy)
        for (int dx = -1; dx <= 1; ++dx) {
            if (dx == 0 && dy == 0) continue;
            int nx = tx + dx, ny = ty + dy;
            if (nx < 0 || ny < 0 || nx >= tilesX || ny >= tilesY) continue;
            int nIdx = ny * tilesX + nx;
            sum += fabsf(e[nIdx] - center);
            ++cnt;
        }

    float contrast = (cnt > 0) ? sum / cnt : 0.0f;
    cOut[idx] = contrast;

    if (doLog && threadIdx.x == 0 && threadIdx.y == 0) {
        char msg[128];
        int n = 0;
        n += sprintf(msg + n, "[CONTRAST] tile=(%d,%d) contrast=%.5f", tx, ty, contrast);
        LUCHS_LOG_DEVICE(msg);
    }
}

// --- Host-Wrapper ------------------------------------------------------------

void computeCudaEntropyContrast(
    const int* d_it, float* d_e, float* d_c,
    int w, int h, int tile, int maxIter)
{
    using clk = std::chrono::high_resolution_clock;
    auto start = clk::now();

    int tilesX = (w + tile - 1) / tile;
    int tilesY = (h + tile - 1) / tile;

    hipMemset(d_e, 0, tilesX * tilesY * sizeof(float));

    entropyKernel<<<dim3(tilesX, tilesY), 128>>>(d_it, d_e, w, h, tile, maxIter);
    hipDeviceSynchronize();

    auto mid = clk::now();

    contrastKernel<<<dim3((tilesX + 15) / 16, (tilesY + 15) / 16), dim3(16,16)>>>(d_e, d_c, tilesX, tilesY);
    hipDeviceSynchronize();

    auto end = clk::now();

    if (Settings::performanceLogging) {
        double entropyMs = std::chrono::duration<double, std::milli>(mid - start).count();
        double contrastMs = std::chrono::duration<double, std::milli>(end - mid).count();
        LUCHS_LOG_HOST("[PERF] entropy=%.3f ms contrast=%.3f ms", entropyMs, contrastMs);
    } else if (Settings::debugLogging) {
        double entropyMs = std::chrono::duration<double, std::milli>(mid - start).count();
        double contrastMs = std::chrono::duration<double, std::milli>(end - mid).count();
        LUCHS_LOG_HOST("[TIME] Entropy %.3f ms | Contrast %.3f ms", entropyMs, contrastMs);
    }
}

void launch_mandelbrotHybrid(
    uchar4* out, int* d_it,
    int w, int h, float zoom, float2 offset,
    int maxIter, int /*tile*/)
{
    using clk = std::chrono::high_resolution_clock;
    auto t0 = clk::now();

    // Otter: 32x8 bei performanceLogging – gute Occupancy/Coalescing.
    dim3 block = Settings::performanceLogging ? dim3(32, 8) : dim3(16, 16);
    dim3 grid((w + block.x - 1)/block.x, (h + block.y - 1)/block.y);

    auto t_launchStart = clk::now();
    mandelbrotKernel<<<grid, block>>>(out, d_it, w, h, zoom, offset, maxIter);
    auto t_launchEnd = clk::now();

    auto t_syncStart = clk::now();
    hipDeviceSynchronize();
    auto t_syncEnd = clk::now();

    auto t1 = clk::now();

    if (Settings::performanceLogging) {
        double launchMs = std::chrono::duration<double, std::milli>(t_launchEnd - t_launchStart).count();
        double syncMs   = std::chrono::duration<double, std::milli>(t_syncEnd - t_syncStart).count();
        double totalMs  = std::chrono::duration<double, std::milli>(t1 - t0).count();
        LUCHS_LOG_HOST("[PERF] mandelbrot: launch=%.3f ms sync=%.3f ms total=%.3f ms",
                       launchMs, syncMs, totalMs);
    } else if (Settings::debugLogging) {
        double launchMs = std::chrono::duration<double, std::milli>(t_launchEnd - t_launchStart).count();
        double syncMs   = std::chrono::duration<double, std::milli>(t_syncEnd - t_syncStart).count();
        double totalMs  = std::chrono::duration<double, std::milli>(t1 - t0).count();
        LUCHS_LOG_HOST("[TIME] Mandelbrot | Launch %.3f ms | Sync %.3f ms | Total %.3f ms",
                       launchMs, syncMs, totalMs);
    }
}
