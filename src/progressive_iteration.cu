#include "hip/hip_runtime.h"
///// Otter: Progressive Iteration & Resume (impl) + optional local Nacktmull-Settings (default off)
///// Schneefuchs: Messbare Pfade; ASCII-Logs; kein versteckter Funktionswechsel.
///// Maus: Coalesced SoA, fixed block, chunked inner loop.
///// Datei: src/progressive_iteration.cu

#include "progressive_iteration.cuh"
#include "luchs_log_host.hpp"
#include "luchs_log_device.hpp"
#include "settings.hpp"              // Host toggles (performanceLogging/debugLogging)

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <cmath>
#include <algorithm>
#include <stdexcept>
#include <cstdint>

// -----------------------------------------------------------------------------
// Local, headerless Nacktmull settings (functional, but default OFF)
// This replaces the removed settings_nacktmull.hpp for this TU only.
// -----------------------------------------------------------------------------
namespace NacktmullSettings {
struct ProgressivePolicy {
    bool     enabled;                   // false → neutral (no influence)
    uint32_t sliceMin;                  // min addIter per step when enabled (>=1)
    double   sliceMaxPct;               // clamp addIter to <= sliceMaxPct * maxIterCap
    bool     deviceDebugLog;            // extra device logs from kernel
    double   stopThresholdSurvivorsPct; // suggest halt if survivors% < threshold (in PERCENT)
};
inline constexpr ProgressivePolicy Progressive_Default{
    /*enabled                    */ false,
    /*sliceMin                   */ 8u,
    /*sliceMaxPct                */ 0.50,   // 50% of maxIterCap
    /*deviceDebugLog             */ false,
    /*stopThresholdSurvivorsPct  */ 0.50    // 0.5%
};
} // namespace NacktmullSettings

namespace prog {

// -------------------------- Device-side tiny formatter ------------------------
// ASCII-only helpers for device logs (no snprintf in device code).
static __device__ __forceinline__ int dev_append_lit(char* dst, int pos, int cap, const char* lit) {
    while (*lit && pos < cap - 1) dst[pos++] = *lit++;
    return pos;
}
static __device__ __forceinline__ int dev_append_uint(char* dst, int pos, int cap, unsigned int v) {
    char tmp[16]; int n = 0;
    do { tmp[n++] = char('0' + (v % 10)); v /= 10; } while (v && n < (int)sizeof(tmp));
    for (int i = n - 1; i >= 0 && pos < cap - 1; --i) dst[pos++] = tmp[i];
    return pos;
}
static __device__ __forceinline__ int dev_append_int(char* dst, int pos, int cap, int v) {
    if (v < 0) { if (pos < cap - 1) dst[pos++] = '-'; unsigned int uv = (unsigned int)(-v); return dev_append_uint(dst, pos, cap, uv); }
    return dev_append_uint(dst, pos, cap, (unsigned int)v);
}

// ------------------------------- Device kernels -------------------------------

__global__ __launch_bounds__(256)
void k_reset_state(float2* __restrict__ z,
                   uint32_t* __restrict__ it,
                   uint8_t* __restrict__ flags,
                   uint32_t* __restrict__ esc,
                   int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    z[i] = make_float2(0.f, 0.f);
    it[i] = 0u;
    flags[i] = 0u;
    esc[i] = 0u;
}

__global__ __launch_bounds__(256)
void k_progressive_step(float2* __restrict__ z,
                        uint32_t* __restrict__ it,
                        uint8_t* __restrict__ flags,
                        uint32_t* __restrict__ esc,
                        uint32_t* __restrict__ activeCount,
                        int width, int height,
                        float x0, float y0, float dx, float dy,
                        uint32_t addIter, uint32_t maxIterCap, float bailout2,
                        int debugDevice)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    const int i = y * width + x;

    float2 zi = z[i];
    uint32_t iters = it[i];
    uint8_t  fl = flags[i];

    // finished (escaped or capped)
    if (fl & 0x03u) return;

    const float cr = x0 + dx * (float)x;
    const float ci = y0 + dy * (float)y;

    // chunked inner loop
    uint32_t local = 0u;
    while (local < addIter && iters < maxIterCap) {
        // z = z^2 + c (use 1 FMA for 2xy)
        const float x2 = zi.x * zi.x;
        const float y2 = zi.y * zi.y;
        const float xy = zi.x * zi.y;
        zi = make_float2((x2 - y2) + cr, fmaf(2.f, xy, ci));

        ++iters;
        ++local;

        // escape?
        const float r2 = zi.x * zi.x + zi.y * zi.y;
        if (r2 > bailout2) {
            fl |= 0x01u;          // escaped
            esc[i] = iters;
            break;
        }
    }

    if (iters >= maxIterCap) {
        fl |= 0x02u;              // reached cap
    }

    z[i] = zi;
    it[i] = iters;
    flags[i] = fl;

    // survivor counting + optional tiny device log
    if ((fl & 0x03u) == 0u) {
        atomicAdd(activeCount, 1u);
        if (debugDevice) {
            char msg[128];
            int p = 0;
            p = dev_append_lit (msg, p, (int)sizeof(msg), "[DEV] survivor x=");
            p = dev_append_int (msg, p, (int)sizeof(msg), x);
            p = dev_append_lit (msg, p, (int)sizeof(msg), " y=");
            p = dev_append_int (msg, p, (int)sizeof(msg), y);
            p = dev_append_lit (msg, p, (int)sizeof(msg), " it=");
            p = dev_append_uint(msg, p, (int)sizeof(msg), iters);
            p = dev_append_lit (msg, p, (int)sizeof(msg), " add=");
            p = dev_append_uint(msg, p, (int)sizeof(msg), local);
            msg[(p < (int)sizeof(msg) ? p : (int)sizeof(msg) - 1)] = '\0';
            LUCHS_LOG_DEVICE(msg);
        }
    }
}

// ------------------------------- Host helpers --------------------------------

static inline dim3 chooseBlock() { return dim3(32, 8, 1); }
static inline dim3 chooseGrid(int w, int h, dim3 b) {
    return dim3((w + (int)b.x - 1) / (int)b.x, (h + (int)b.y - 1) / (int)b.y, 1);
}

struct PixelMap { float x0, y0, dx, dy; };
static inline PixelMap makePixelMap(const ViewportParams& vp)
{
    const double dy = vp.scale / (double)vp.height;
    const double dx = dy;
    const double x0 = vp.centerX - dx * (double)vp.width  * 0.5;
    const double y0 = vp.centerY - dy * (double)vp.height * 0.5;
    return PixelMap{ (float)x0, (float)y0, (float)dx, (float)dy };
}

// ------------------------------ RAII methods ---------------------------------

CudaProgressiveState::~CudaProgressiveState()
{
    if (d_z_)            hipFree(d_z_);
    if (d_it_)           hipFree(d_it_);
    if (d_flags_)        hipFree(d_flags_);
    if (d_escapeIter_)   hipFree(d_escapeIter_);
    if (d_activeCount_)  hipFree(d_activeCount_);
}

void CudaProgressiveState::ensure(int width, int height)
{
    if (width == width_ && height == height_) return;

    if (d_z_)            CUDA_CHECK(hipFree(d_z_));
    if (d_it_)           CUDA_CHECK(hipFree(d_it_));
    if (d_flags_)        CUDA_CHECK(hipFree(d_flags_));
    if (d_escapeIter_)   CUDA_CHECK(hipFree(d_escapeIter_));
    if (d_activeCount_)  CUDA_CHECK(hipFree(d_activeCount_));

    width_ = width;
    height_ = height;
    const size_t n = (size_t)width_ * (size_t)height_;

    CUDA_CHECK(hipMalloc(&d_z_,           n * sizeof(float2)));
    CUDA_CHECK(hipMalloc(&d_it_,          n * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_flags_,       n * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_escapeIter_,  n * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_activeCount_, sizeof(uint32_t)));

    reset(0);

    LUCHS_LOG_HOST("[PROG] ensure buffers width=%d height=%d bytes=%.1fMB",
                   width_, height_,
                   (n*(sizeof(float2)+sizeof(uint32_t)+sizeof(uint8_t)+sizeof(uint32_t)) + sizeof(uint32_t)) / (1024.0*1024.0));
}

void CudaProgressiveState::reset(hipStream_t stream)
{
    const int n = width_ * height_;
    if (n <= 0) return;
    dim3 block(256);
    dim3 grid((n + (int)block.x - 1) / (int)block.x);
    k_reset_state<<<grid, block, 0, stream>>>(d_z_, d_it_, d_flags_, d_escapeIter_, n);
    CUDA_CHECK(hipGetLastError());
    const uint32_t zero = 0u;
    CUDA_CHECK(hipMemcpyAsync(d_activeCount_, &zero, sizeof(uint32_t), hipMemcpyHostToDevice, stream));
}

void CudaProgressiveState::maybeResetOnChange(const ViewportParams& vp, bool enableReset, hipStream_t stream)
{
    if (!enableReset) { lastCx_ = vp.centerX; lastCy_ = vp.centerY; lastScale_ = vp.scale; return; }

    if (vp.width != width_ || vp.height != height_ ||
        vp.centerX != lastCx_ || vp.centerY != lastCy_ || vp.scale != lastScale_) {
        reset(stream);
        LUCHS_LOG_HOST("[PROG] reset-on-change cx=%.17g cy=%.17g scale=%.17g w=%d h=%d",
                       vp.centerX, vp.centerY, vp.scale, vp.width, vp.height);
    }
    lastCx_ = vp.centerX; lastCy_ = vp.centerY; lastScale_ = vp.scale;
}

ProgressiveMetrics CudaProgressiveState::step(const ViewportParams& vp, const ProgressiveConfig& cfg, hipStream_t stream)
{
    ensure(vp.width, vp.height);
    maybeResetOnChange(vp, cfg.resetOnChange, stream);

    const uint32_t zero = 0u;
    CUDA_CHECK(hipMemcpyAsync(d_activeCount_, &zero, sizeof(uint32_t), hipMemcpyHostToDevice, stream));

    const auto map = makePixelMap(vp);
    const dim3 block = chooseBlock();
    const dim3 grid  = chooseGrid(width_, height_, block);

    // --- Optional progressive policy (default OFF → neutral) ------------------
    constexpr auto P = NacktmullSettings::Progressive_Default;
    const uint32_t sliceMin = P.enabled ? std::max<uint32_t>(1u, P.sliceMin) : 1u;
    const uint32_t sliceMax = P.enabled ? std::max<uint32_t>(1u, (uint32_t)std::floor(P.sliceMaxPct * (double)cfg.maxIterCap))
                                        : cfg.maxIterCap;
    const uint32_t effAddIter = std::min<uint32_t>(std::max<uint32_t>(cfg.chunkIter, sliceMin), sliceMax);
    const int devDbg = (cfg.debugDevice || (P.enabled && P.deviceDebugLog)) ? 1 : 0;
    // --------------------------------------------------------------------------

    float ms = 0.0f;

    if constexpr (Settings::performanceLogging || Settings::debugLogging) {
        hipEvent_t evStart{}, evStop{};
        CUDA_CHECK(hipEventCreate(&evStart));
        CUDA_CHECK(hipEventCreate(&evStop));
        CUDA_CHECK(hipEventRecord(evStart, stream));

        k_progressive_step<<<grid, block, 0, stream>>>(
            d_z_, d_it_, d_flags_, d_escapeIter_, d_activeCount_,
            width_, height_,
            map.x0, map.y0, map.dx, map.dy,
            effAddIter, cfg.maxIterCap, cfg.bailout2,
            devDbg
        );
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipEventRecord(evStop, stream));
        CUDA_CHECK(hipEventSynchronize(evStop));
        CUDA_CHECK(hipEventElapsedTime(&ms, evStart, evStop));
        CUDA_CHECK(hipEventDestroy(evStart));
        CUDA_CHECK(hipEventDestroy(evStop));
    } else {
        k_progressive_step<<<grid, block, 0, stream>>>(
            d_z_, d_it_, d_flags_, d_escapeIter_, d_activeCount_,
            width_, height_,
            map.x0, map.y0, map.dx, map.dy,
            effAddIter, cfg.maxIterCap, cfg.bailout2,
            devDbg
        );
        CUDA_CHECK(hipGetLastError());
    }

    uint32_t still = 0u;
    CUDA_CHECK(hipMemcpy(&still, d_activeCount_, sizeof(uint32_t), hipMemcpyDeviceToHost));

    ProgressiveMetrics m;
    m.kernel_ms = ms;
    m.stillActive = still;
    m.addIterApplied = effAddIter;

    const double totalPx = (double)width_ * (double)height_;
    const double survivorsPct = (totalPx > 0.0) ? (100.0 * (double)still / totalPx) : 0.0;

    if constexpr (Settings::debugLogging) {
        if (P.enabled) {
            LUCHS_LOG_HOST("[PROG] cfg clamp addIter=%u->%u sliceMin=%u sliceMax=%u surv=%.3f%%",
                           cfg.chunkIter, effAddIter, sliceMin, sliceMax, (float)survivorsPct);
        }
    }

    // Optional: Schwellenhinweis (kein API-Wechsel, nur Log-Hinweis)
    if (P.enabled && survivorsPct < P.stopThresholdSurvivorsPct) {
        LUCHS_LOG_HOST("[PROG] HALT-SUGGEST thresh=%.3f%% survivors=%.3f%% w=%d h=%d",
                       (float)P.stopThresholdSurvivorsPct, (float)survivorsPct, width_, height_);
    }

    LUCHS_LOG_HOST("[PROG] step done addIter=%u maxCap=%u ms=%.3f survivors=%u w=%d h=%d",
                   m.addIterApplied, cfg.maxIterCap, m.kernel_ms, m.stillActive, width_, height_);

    return m;
}

} // namespace prog
