#include "hip/hip_runtime.h"
// File: src/main.cu
#include "core_kernel.h"
#ifdef _WIN32
#include <windows.h>
#endif

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#include <iostream>
#include <vector>
#include <cstdlib>

// CUDA-Error-Check
void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

// Komplexitäts-Kernel
__global__ void computeComplexity(const uchar4* img,
                                  int width, int height,
                                  float* complexity);

// Vertex- und Fragment-Shader (wie gehabt)...

static const char* vertexShaderSrc = R"glsl(
#version 330 core
layout(location = 0) in vec2 aPos;
layout(location = 1) in vec2 aTexCoord;
out vec2 TexCoord;
void main() {
    gl_Position = vec4(aPos, 0.0, 1.0);
    TexCoord = aTexCoord;
}
)glsl";

static const char* fragmentShaderSrc = R"glsl(
#version 330 core
in vec2 TexCoord;
out vec4 FragColor;
uniform sampler2D uTex;
void main() {
    FragColor = texture(uTex, TexCoord);
}
)glsl";

int main() {
    const int width  = 1024;
    const int height = 768;
    size_t imgBytes  = width * height * sizeof(uchar4);

    float zoom    = 300.0f;
    float2 offset = make_float2(0.0f, 0.0f);
    int   maxIter = 500;

    // GLFW + OpenGL init (Core Profile)...
    if (!glfwInit()) std::exit(EXIT_FAILURE);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    GLFWwindow* window = glfwCreateWindow(width, height, "Auto-Zoom Mandelbrot", nullptr, nullptr);
    if (!window) std::exit(EXIT_FAILURE);
    glfwMakeContextCurrent(window);
    if (glewInit() != GLEW_OK) { std::cerr<<"GLEW init failed\n"; return EXIT_FAILURE; }

    // 1) PBO + CUDA-GL Interop
    GLuint pbo;
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, imgBytes, nullptr, GL_DYNAMIC_DRAW);

    hipGraphicsResource* cudaPbo = nullptr;
    checkCuda(hipGraphicsGLRegisterBuffer(&cudaPbo, pbo, cudaGraphicsMapFlagsWriteDiscard),
              "hipGraphicsGLRegisterBuffer");

    // 2) Texture
    GLuint tex;
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height,
                 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    // 3) Shader-Programm und Quad (VAO/VBO) wie zuvor...
    auto compileShader = [&](GLenum type, const char* src) {
        GLuint s = glCreateShader(type);
        glShaderSource(s, 1, &src, nullptr);
        glCompileShader(s);
        GLint ok; glGetShaderiv(s, GL_COMPILE_STATUS, &ok);
        if (!ok) { char buf[512]; glGetShaderInfoLog(s,512,nullptr,buf);
            std::cerr<<"Shader-Error:\n"<<buf; std::exit(EXIT_FAILURE);
        }
        return s;
    };
    GLuint vs = compileShader(GL_VERTEX_SHADER, vertexShaderSrc);
    GLuint fs = compileShader(GL_FRAGMENT_SHADER, fragmentShaderSrc);
    GLuint program = glCreateProgram();
    glAttachShader(program, vs);
    glAttachShader(program, fs);
    glLinkProgram(program);
    { GLint ok; glGetProgramiv(program,GL_LINK_STATUS,&ok);
      if (!ok){ char buf[512]; glGetProgramInfoLog(program,512,nullptr,buf);
          std::cerr<<"Link-Error:\n"<<buf; std::exit(EXIT_FAILURE);
      }
    }
    glDeleteShader(vs); glDeleteShader(fs);

    float quadVerts[] = {
        -1,-1, 0,0,
         1,-1, 1,0,
        -1, 1, 0,1,
         1, 1, 1,1,
    };
    GLuint VAO, VBO;
    glGenVertexArrays(1,&VAO);
    glGenBuffers(1,&VBO);
    glBindVertexArray(VAO);
      glBindBuffer(GL_ARRAY_BUFFER,VBO);
      glBufferData(GL_ARRAY_BUFFER,sizeof(quadVerts),quadVerts,GL_STATIC_DRAW);
      glEnableVertexAttribArray(0);
      glVertexAttribPointer(0,2,GL_FLOAT,GL_FALSE,4*sizeof(float),(void*)0);
      glEnableVertexAttribArray(1);
      glVertexAttribPointer(1,2,GL_FLOAT,GL_FALSE,4*sizeof(float),(void*)(2*sizeof(float)));
    glBindVertexArray(0);

    // 4) Complexity-Buffer
    int tilesX     = (width  + TILE_W - 1) / TILE_W;
    int tilesY     = (height + TILE_H - 1) / TILE_H;
    int totalTiles = tilesX*tilesY;
    float* d_complexity=nullptr;
    checkCuda(hipMalloc(&d_complexity, totalTiles*sizeof(float)), "hipMalloc complexity");
    std::vector<float> h_complexity(totalTiles);

    // Haupt-Loop
    while (!glfwWindowShouldClose(window)) {
        // a) Reset Complexity
        checkCuda(hipMemset(d_complexity, 0, totalTiles*sizeof(float)), "memset complexity");

        // b) Map PBO
        uchar4* d_img=nullptr; size_t sz=0;
        checkCuda(hipGraphicsMapResources(1,&cudaPbo,0),"MapResources");
        checkCuda(hipGraphicsResourceGetMappedPointer((void**)&d_img,&sz,cudaPbo),
                  "GetMappedPointer");

        // c) Aufruf Hybrid-Kernel
        dim3 blockDim(TILE_W,TILE_H), gridDim(tilesX,tilesY);
        mandelbrotHybrid<<<gridDim,blockDim>>>(d_img,width,height,zoom,offset,maxIter);
        checkCuda(hipDeviceSynchronize(),"mandelbrotHybrid");

        // d) Complexity-Kernel
        computeComplexity<<<gridDim,blockDim>>>(d_img,width,height,d_complexity);
        checkCuda(hipDeviceSynchronize(),"computeComplexity");

        // e) Unmap PBO
        checkCuda(hipGraphicsUnmapResources(1,&cudaPbo,0),"UnmapResources");

        // ————— NEU —————
        // f) PBO → Texture
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
        glBindTexture(GL_TEXTURE_2D, tex);
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height,
                        GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

        // g) Best Tile finden & Zoom-Update
        checkCuda(hipMemcpy(h_complexity.data(),d_complexity,
                             totalTiles*sizeof(float),
                             hipMemcpyDeviceToHost),"Memcpy complexity");
        int bestIdx=0; float bestScore=-1.0f;
        for(int i=0;i<totalTiles;++i){
            if(h_complexity[i]>bestScore){
                bestScore=h_complexity[i];
                bestIdx=i;
            }
        }
        int bestX = bestIdx%tilesX, bestY=bestIdx/tilesX;
        offset.x += ((bestX+0.5f)*TILE_W - width*0.5f)/zoom;
        offset.y += ((bestY+0.5f)*TILE_H - height*0.5f)/zoom;
        zoom *= 1.2f;

        // h) Rendern via Shader
        glClearColor(0,0,0,1);
        glClear(GL_COLOR_BUFFER_BIT);
        glUseProgram(program);
        glBindVertexArray(VAO);
        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, tex);
        glUniform1i(glGetUniformLocation(program,"uTex"),0);
        glDrawArrays(GL_TRIANGLE_STRIP,0,4);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Cleanup
    hipFree(d_complexity);
    hipGraphicsUnregisterResource(cudaPbo);
    glDeleteBuffers(1,&pbo);
    glDeleteTextures(1,&tex);
    glDeleteProgram(program);
    glDeleteBuffers(1,&VBO);
    glDeleteVertexArrays(1,&VAO);
    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
