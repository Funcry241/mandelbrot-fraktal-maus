#include "hip/hip_runtime.h"
// Datei: src/main.cu
// Maus-Kommentar:
// 1) Debug-Modus: Einfache Gradient-Ausgabe, um PBO→Texture→Quad-Pipeline zu prüfen.
// 2) Sobald der Gradient klappt, können wir wieder zu Mandelbrot™ zurückschalten.
//    Dazu am Anfang DEBUG_GRADIENT auf 0 setzen.
//
// Der untenstehende Code kennt zwei Modi:
//   - DEBUG_GRADIENT == 1 → testKernel() füllt das Bild mit einem X/Y-Gradient.
//   - DEBUG_GRADIENT == 0 → launch_mandelbrotHybrid() füllt das Bild wie gewohnt.

#include "core_kernel.h"

#ifdef _WIN32
#include <windows.h>
#endif

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#include <iostream>
#include <vector>
#include <cmath>
#include <limits>

// -------------------------------------------------------------
// Fehlerprüfung Makros
#define CUDA_CHECK(call)                                                   \
    do {                                                                   \
        hipError_t err = call;                                            \
        if (err != hipSuccess) {                                          \
            std::cerr << "CUDA Fehler in " << __FILE__ << ":" << __LINE__ \
                      << " -> " << hipGetErrorString(err) << std::endl;   \
            std::exit(EXIT_FAILURE);                                       \
        }                                                                  \
    } while(0)

#define GL_CHECK()                                                         \
    do {                                                                   \
        GLenum err = glGetError();                                         \
        if (err != GL_NO_ERROR) {                                          \
            std::cerr << "OpenGL Fehler in " << __FILE__ << ":"           \
                      << __LINE__ << " -> 0x" << std::hex << err << std::endl; \
            std::exit(EXIT_FAILURE);                                       \
        }                                                                  \
    } while(0)

// -------------------------------------------------------------
// Umschalter für Debug-Gradient
#define DEBUG_GRADIENT 1

// -------------------------------------------------------------
// Shader-Quellen
static const char* vertexShaderSrc = R"GLSL(
#version 430 core
layout(location=0) in vec2 aPos;
layout(location=1) in vec2 aTex;
out vec2 vTex;
void main(){
    vTex = aTex;
    gl_Position = vec4(aPos, 0.0, 1.0);
}
)GLSL";

static const char* fragmentShaderSrc = R"GLSL(
#version 430 core
in vec2 vTex;
out vec4 FragColor;
uniform sampler2D uTex;
void main(){
    FragColor = texture(uTex, vTex);
}
)GLSL";

// -------------------------------------------------------------
// Shader-Helpers
GLuint compileShader(GLenum type, const char* src) {
    GLuint s = glCreateShader(type);
    glShaderSource(s, 1, &src, nullptr);
    glCompileShader(s);
    GLint ok; glGetShaderiv(s, GL_COMPILE_STATUS, &ok);
    if (!ok) {
        char buf[512]; glGetShaderInfoLog(s, 512, nullptr, buf);
        std::cerr << "Shader-Compile-Error: " << buf << std::endl;
        std::exit(EXIT_FAILURE);
    }
    return s;
}

GLuint createProgram() {
    GLuint v = compileShader(GL_VERTEX_SHADER, vertexShaderSrc);
    GLuint f = compileShader(GL_FRAGMENT_SHADER, fragmentShaderSrc);
    GLuint p = glCreateProgram();
    glAttachShader(p, v);
    glAttachShader(p, f);
    glLinkProgram(p);
    GLint ok; glGetProgramiv(p, GL_LINK_STATUS, &ok);
    if (!ok) {
        char buf[512]; glGetProgramInfoLog(p, 512, nullptr, buf);
        std::cerr << "Program-Link-Error: " << buf << std::endl;
        std::exit(EXIT_FAILURE);
    }
    glDeleteShader(v);
    glDeleteShader(f);
    return p;
}

// -------------------------------------------------------------
// Prototyp für Complexity-Kernel (wird im echten Mandelbrot-Modus gebraucht)
__global__ void computeComplexity(const uchar4* img,
                                  int width, int height,
                                  float* complexity);

// -------------------------------------------------------------
// Debug / Test-Kernel: füllt IMG mit einem einfachen X/Y-Gradient
#if DEBUG_GRADIENT
__global__ void testKernel(uchar4* img, int width, int height) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if (tx >= width || ty >= height) return;
    // einfacher Farbverlauf: R = (tx % 256), G = (ty % 256), B = 0
    unsigned char r = static_cast<unsigned char>(tx & 0xFF);
    unsigned char g = static_cast<unsigned char>(ty & 0xFF);
    img[ty * width + tx] = make_uchar4(r, g, 0, 255);
}
#endif

// -------------------------------------------------------------
// Echt-Kernel-Prototyp für Mandelbrot (im Nicht-Debug-Modus)
#if !DEBUG_GRADIENT
extern "C" void launch_mandelbrotHybrid(uchar4* img,
                                        int w, int h,
                                        float zoom, float2 offset,
                                        int maxIter);
#endif

// -------------------------------------------------------------
int main() {
    std::cout << "=== Programm gestartet ===\n";

    // --- Bild-Settings ---
    const int W = 1024, H = 768;
    size_t imgBytes = size_t(W) * H * sizeof(uchar4);

    // --- Mandelbrot-Parameter (im echten Modus) ---
    float zoom = 300.0f;
    float2 offset = make_float2(0.0f, 0.0f);
    int maxIter = 500;

    // --- GLFW + GL Context ---
    if (!glfwInit()) {
        std::cerr << "GLFW-Init fehlgeschlagen!\n";
        std::exit(EXIT_FAILURE);
    }
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* win = glfwCreateWindow(W, H, "OtterDream Mandelbrot", nullptr, nullptr);
    if (!win) {
        std::cerr << "Fenster-Erstellung fehlgeschlagen!\n";
        std::exit(EXIT_FAILURE);
    }
    glfwMakeContextCurrent(win);

    if (glewInit() != GLEW_OK) {
        std::cerr << "GLEW-Init fehlgeschlagen\n";
        std::exit(EXIT_FAILURE);
    }
    std::cout << "GLFW + GLEW init OK\n";

    // --- PBO + CUDA-GL Interop ---
    GLuint pbo;
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, imgBytes, nullptr, GL_DYNAMIC_DRAW);
    GL_CHECK();

    hipGraphicsResource* cudaPbo;
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(
        &cudaPbo, pbo, cudaGraphicsMapFlagsWriteDiscard
    ));

    // --- Texture Setup ---
    GLuint tex;
    glGenTextures(1, &tex);
    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, W, H, 0,
                 GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    GL_CHECK();

    // --- Shader + Quad Setup ---
    GLuint program = createProgram();

    // Sampler-Uniform "uTex" auf Texture Unit 0 setzen
    glUseProgram(program);
    GLint loc = glGetUniformLocation(program, "uTex");
    if (loc >= 0) {
        glUniform1i(loc, 0);
    } else {
        std::cerr << "Warnung: Uniform 'uTex' nicht gefunden!\n";
    }

    GLuint VAO, VBO, EBO;
    float quad[] = {
        // Pos    // Tex
        -1.0f,-1.0f,   0.0f,0.0f,
         1.0f,-1.0f,   1.0f,0.0f,
         1.0f, 1.0f,   1.0f,1.0f,
        -1.0f, 1.0f,   0.0f,1.0f
    };
    unsigned idx[] = {0,1,2, 2,3,0};

    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    glBindVertexArray(VAO);
      glBindBuffer(GL_ARRAY_BUFFER, VBO);
      glBufferData(GL_ARRAY_BUFFER, sizeof(quad), quad, GL_STATIC_DRAW);

      glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
      glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(idx), idx, GL_STATIC_DRAW);

      // Position (location=0)
      glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
      glEnableVertexAttribArray(0);
      // TexCoord (location=1)
      glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float),
                            (void*)(2 * sizeof(float)));
      glEnableVertexAttribArray(1);
    glBindVertexArray(0);
    GL_CHECK();

    // --- Complexity Buffer (wird im echten Modus benutzt) ---
    int tilesX = (W + TILE_W - 1) / TILE_W;
    int tilesY = (H + TILE_H - 1) / TILE_H;
    int totalTiles = tilesX * tilesY;

    float* d_complexity = nullptr;
    if (!DEBUG_GRADIENT) {
        CUDA_CHECK(hipMalloc(&d_complexity,
                              totalTiles * sizeof(float)));
    }
    std::vector<float> h_complexity(totalTiles);

    std::cout << "Setup abgeschlossen, betrete Haupt-Loop\n";

    int frame = 0;
    while (!glfwWindowShouldClose(win)) {
        // 1) PBO → CUDA-Device (mappen)
        uchar4* d_img = nullptr;
        size_t sz_ptr = 0;
        CUDA_CHECK(hipGraphicsMapResources(1, &cudaPbo, 0));
        CUDA_CHECK(hipGraphicsResourceGetMappedPointer(
                       (void**)&d_img, &sz_ptr, cudaPbo));

#if DEBUG_GRADIENT
        // *** Debug-Modus: einfacher Gradient ***
        dim3 bd_g(16, 16);
        dim3 gd_g((W + bd_g.x - 1) / bd_g.x,
                  (H + bd_g.y - 1) / bd_g.y);
        testKernel<<<gd_g, bd_g>>>(d_img, W, H);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
#else
        // *** Echte Mandelbrot-Ausgabe ***
        launch_mandelbrotHybrid(d_img, W, H, zoom, offset, maxIter);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Complexity-Kernel
        CUDA_CHECK(hipMemset(d_complexity, 0,
                              totalTiles * sizeof(float)));
        dim3 bd(TILE_W, TILE_H);
        dim3 gd(tilesX, tilesY);
        computeComplexity<<<gd, bd>>>(d_img, W, H, d_complexity);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
#endif

        // 2) PBO unmappen
        CUDA_CHECK(hipGraphicsUnmapResources(1, &cudaPbo, 0));

        // 3) Upload PBO → Texture
        glActiveTexture(GL_TEXTURE0);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
        glBindTexture(GL_TEXTURE_2D, tex);
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, W, H,
                        GL_RGBA, GL_UNSIGNED_BYTE, 0);
        GL_CHECK();

        // 4) Rendern mit Vollbild-Quad + Shader
        glViewport(0, 0, W, H);
        glClear(GL_COLOR_BUFFER_BIT);
        glUseProgram(program);
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, nullptr);
        GL_CHECK();

        glfwSwapBuffers(win);
        glfwPollEvents();

        // 5) Debug-Ausgabe (NUR im echten Modus sinnvoll)
#if DEBUG_GRADIENT
        if (frame == 0) {
            std::cout << "(DEBUG_GRADIENT-Modus: Farbverlauf angezeigt)\n";
        }
#else
        // read back complexity & finde beste Kachel
        CUDA_CHECK(hipMemcpy(h_complexity.data(), d_complexity,
                              totalTiles * sizeof(float),
                              hipMemcpyDeviceToHost));

        int bestIdx = 0;
        float bestScore = -1.0f;
        for (int i = 0; i < totalTiles; ++i) {
            if (h_complexity[i] > bestScore) {
                bestScore = h_complexity[i];
                bestIdx = i;
            }
        }

        std::cout << "Frame " << frame
                  << ": zoom=" << zoom
                  << " offset=(" << offset.x << "," << offset.y << ")"
                  << " bestScore=" << bestScore
                  << std::endl;
#endif

        frame++;

#if !DEBUG_GRADIENT
        // 6) Zoom + Offset NACH dem Rendern (nur im echten Modus)
        if (bestScore > 0.0f) {
            int bx = bestIdx % tilesX;
            int by = bestIdx / tilesX;
            offset.x += ((bx + 0.5f)*TILE_W - W*0.5f)/zoom;
            offset.y += ((by + 0.5f)*TILE_H - H*0.5f)/zoom;
        }
        zoom *= 1.01f;
#endif
    }

    // --- Cleanup ---
    if (!DEBUG_GRADIENT) {
        CUDA_CHECK(hipFree(d_complexity));
    }
    CUDA_CHECK(hipGraphicsUnregisterResource(cudaPbo));
    glDeleteBuffers(1, &pbo);
    glDeleteTextures(1, &tex);
    glDeleteProgram(program);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    glDeleteVertexArrays(1, &VAO);
    glfwDestroyWindow(win);
    glfwTerminate();

    return 0;
}
